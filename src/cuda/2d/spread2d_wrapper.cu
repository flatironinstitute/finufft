#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>

#include "spreadinterp2d.cuh"

using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cufinufft_spread2d(int nf1, int nf2, cuda_complex<T> *d_fw, int M, T *d_kx, T *d_ky, cuda_complex<T> *d_c,
                       cufinufft_plan_template<T> d_plan)
/*
    This c function is written for only doing 2D spreading. See
    test/spread2d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    hipEventRecord(start);
    ier = ALLOCGPUMEM2D_PLAN(d_plan);
    ier = ALLOCGPUMEM2D_NUPTS(d_plan);

    if (d_plan->opts.gpu_method == 1) {
        ier = CUSPREAD2D_NUPTSDRIVEN_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_nuptsdriven_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    if (d_plan->opts.gpu_method == 2) {
        ier = CUSPREAD2D_SUBPROB_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

#ifdef TIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Obtain Spread Prop\t %.3g ms\n", milliseconds);
#endif

    hipEventRecord(start);
    ier = CUSPREAD2D(d_plan, 1);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Spread (%d)\t\t %5.3f ms\n", d_plan->opts.gpu_method, milliseconds);
#endif

    hipEventRecord(start);
    FREEGPUMEMORY2D(d_plan);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
    return ier;
}

template <typename T>
int cuspread2d(cufinufft_plan_template<T> d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int M = d_plan->M;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        hipEventRecord(start);
        ier = cuspread2d_nuptsdriven<T>(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread2d_gpu_nuptsdriven" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        hipEventRecord(start);
        ier = cuspread2d_subprob<T>(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread2d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1,2,3" << std::endl;
        return 2;
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "[time  ]"
              << " Spread " << milliseconds << " ms" << std::endl;
#endif
    return ier;
}

template <typename T>
int cuspread2d_nuptsdriven_prop(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (d_plan->opts.gpu_sort) {

        int bin_size_x = d_plan->opts.gpu_binsizex;
        int bin_size_y = d_plan->opts.gpu_binsizey;
        if (bin_size_x < 0 || bin_size_y < 0) {
            std::cout << "error: invalid binsize (binsizex, binsizey) = (";
            std::cout << bin_size_x << "," << bin_size_y << ")" << std::endl;
            return 1;
        }

        int numbins[2];
        numbins[0] = ceil((T)nf1 / bin_size_x);
        numbins[1] = ceil((T)nf2 / bin_size_y);

#ifdef DEBUG
        std::cout << "[debug ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
                  << d_plan->opts.gpu_binsizey << "]" << std::endl;
        std::cout << "[debug ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

        T *d_kx = d_plan->kx;
        T *d_ky = d_plan->ky;
#ifdef DEBUG
        T *h_kx;
        T *h_ky;
        h_kx = (T *)malloc(M * sizeof(T));
        h_ky = (T *)malloc(M * sizeof(T));

        checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(T), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_ky, d_ky, M * sizeof(T), hipMemcpyDeviceToHost));
        for (int i = M - 10; i < M; i++) {
            std::cout << "[debug ] ";
            std::cout << "(" << setw(3) << h_kx[i] << "," << setw(3) << h_ky[i] << ")" << std::endl;
        }
#endif
        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;

        hipEventRecord(start);
        checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int)));
        CalcBinSize_noghost_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
                                                                numbins[1], d_binsize, d_kx, d_ky, d_sortidx, pirange);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binsize; // For debug
        h_binsize = (int *)malloc(numbins[0] * numbins[1] * sizeof(int));
        checkCudaErrors(
            hipMemcpy(h_binsize, d_binsize, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] bin size:" << std::endl;
        for (int j = 0; j < numbins[1]; j++) {
            std::cout << "[debug ] ";
            for (int i = 0; i < numbins[0]; i++) {
                if (i != 0)
                    std::cout << " ";
                std::cout << " bin[" << setw(1) << i << "," << setw(1) << j << "]=" << h_binsize[i + j * numbins[0]];
            }
            std::cout << std::endl;
        }
        free(h_binsize);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;

        int *h_sortidx;
        h_sortidx = (int *)malloc(M * sizeof(int));

        checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));

        for (int i = 0; i < M; i++) {
            if (h_sortidx[i] < 0) {
                std::cout << "[debug ] ";
                std::cout << "point[" << setw(3) << i << "]=" << setw(3) << h_sortidx[i] << std::endl;
                std::cout << "[debug ] ";
                printf("(%10.10f, %10.10f) ", RESCALE(h_kx[i], nf1, pirange), RESCALE(h_ky[i], nf1, pirange));
                printf("(%10.10f, %10.10f) ", RESCALE(h_kx[i], nf1, pirange) / 32, RESCALE(h_ky[i], nf1, pirange) / 32);
                printf("(%f, %f)\n", floor(RESCALE(h_kx[i], nf1, pirange) / 32),
                       floor(RESCALE(h_ky[i], nf1, pirange) / 32));
            }
        }
#endif
        hipEventRecord(start);
        int n = numbins[0] * numbins[1];
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binstartpts;
        h_binstartpts = (int *)malloc((numbins[0] * numbins[1]) * sizeof(int));
        checkCudaErrors(
            hipMemcpy(h_binstartpts, d_binstartpts, (numbins[0] * numbins[1]) * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
        for (int j = 0; j < numbins[1]; j++) {
            std::cout << "[debug ] ";
            for (int i = 0; i < numbins[0]; i++) {
                if (i != 0)
                    std::cout << " ";
                std::cout << " bin[" << setw(1) << i << "," << setw(1) << j
                          << "]=" << h_binstartpts[i + j * numbins[0]];
            }
            std::cout << std::endl;
        }
        free(h_binstartpts);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;
#endif
        hipEventRecord(start);
        CalcInvertofGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                                      d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts,
                                                                      pirange, nf1, nf2);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_2d \t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_idxnupts;
        h_idxnupts = (int *)malloc(M * sizeof(int));
        checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 0; i < 10; i++) {
            std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
        }
        free(h_idxnupts);
#endif
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        hipEventRecord(start);
        TrivialGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, d_idxnupts);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel TrivialGlobalSortIDx_2d \t\t%.3g ms\n", milliseconds);
#endif
    }
    return 0;
}

template <typename T>
int cuspread2d_nuptsdriven(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;
    hipEventRecord(start);
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M,
                                                                      ns, nf1, nf2, sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns,
                                                               nf1, nf2, es_c, es_beta, d_idxnupts, pirange);
        }
    }

#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_2d_NUptsdriven (%d)\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

template <typename T>
int cuspread2d_subprob_prop(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    if (bin_size_x < 0 || bin_size_y < 0) {
        std::cout << "error: invalid binsize (binsizex, binsizey) = (";
        std::cout << bin_size_x << "," << bin_size_y << ")" << std::endl;
        return 1;
    }
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
#ifdef DEBUG
    std::cout << "[debug  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[debug  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;

#ifdef DEBUG
    T *h_kx;
    T *h_ky;
    h_kx = (T *)malloc(M * sizeof(T));
    h_ky = (T *)malloc(M * sizeof(T));

    checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_ky, d_ky, M * sizeof(T), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ]";
        std::cout << "(" << setw(3) << h_kx[i] << "," << setw(3) << h_ky[i] << ")" << std::endl;
    }
#endif
    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;

    int pirange = d_plan->spopts.pirange;

    hipEventRecord(start);
    checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int)));
    CalcBinSize_noghost_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                            d_binsize, d_kx, d_ky, d_sortidx, pirange);
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binsize; // For debug
    h_binsize = (int *)malloc(numbins[0] * numbins[1] * sizeof(int));
    checkCudaErrors(hipMemcpy(h_binsize, d_binsize, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] bin size:" << std::endl;
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << " bin[" << setw(3) << i << "," << setw(3) << j << "]=" << h_binsize[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_binsize);
    std::cout << "[debug ] ----------------------------------------------------" << std::endl;
#endif
#ifdef DEBUG
    int *h_sortidx;
    h_sortidx = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ]";
    for (int i = 0; i < M; i++) {
        std::cout << "[debug] point[" << setw(3) << i << "]=" << setw(3) << h_sortidx[i] << std::endl;
    }

#endif

    hipEventRecord(start);
    int n = numbins[0] * numbins[1];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binstartpts;
    h_binstartpts = (int *)malloc((numbins[0] * numbins[1]) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_binstartpts, d_binstartpts, (numbins[0] * numbins[1]) * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "bin[" << setw(3) << i << "," << setw(3) << j << "] = " << setw(2)
                      << h_binstartpts[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_binstartpts);
    std::cout << "[debug ] ---------------------------------------------------" << std::endl;
#endif
    hipEventRecord(start);
    CalcInvertofGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                                  d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts,
                                                                  pirange, nf1, nf2);
#ifdef DEBUG
    int *h_idxnupts;
    h_idxnupts = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
    }
    free(h_idxnupts);
#endif
    hipEventRecord(start);
    CalcSubProb_2d<<<(M + 1024 - 1) / 1024, 1024>>>(d_binsize, d_numsubprob, maxsubprobsize, numbins[0] * numbins[1]);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcSubProb_2d\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_numsubprob;
    h_numsubprob = (int *)malloc(n * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_numsubprob, d_numsubprob, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << setw(3) << i << "," << setw(3) << j << "] = " << setw(2)
                      << h_numsubprob[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_numsubprob);
#endif
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Scan Subprob array\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
    printf("[debug ] Subproblem start points\n");
    int *h_subprobstartpts;
    h_subprobstartpts = (int *)malloc((n + 1) * sizeof(int));
    checkCudaErrors(hipMemcpy(h_subprobstartpts, d_subprobstartpts, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << setw(3) << i << "," << setw(3) << j << "] = " << setw(2)
                      << h_subprobstartpts[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
    free(h_subprobstartpts);
#endif
    hipEventRecord(start);
    int totalnumsubprob;
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    MapBintoSubProb_2d<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024>>>(d_subprob_to_bin, d_subprobstartpts,
                                                                              d_numsubprob, numbins[0] * numbins[1]);
    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != NULL);
    d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
    printf("[debug ] Map Subproblem to Bins\n");
    int *h_subprob_to_bin;
    h_subprob_to_bin = (int *)malloc((totalnumsubprob) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_subprob_to_bin, d_subprob_to_bin, (totalnumsubprob) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < totalnumsubprob; j++) {
        std::cout << "[debug ] ";
        std::cout << "nsub[" << j << "] = " << setw(2) << h_subprob_to_bin[j];
        std::cout << std::endl;
    }
    free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
    return 0;
}

template <typename T>
int cuspread2d_subprob(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
#ifdef INFO
    std::cout << "[info  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[info  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    int pirange = d_plan->spopts.pirange;

    T sigma = d_plan->opts.upsampfac;
    hipEventRecord(start);

    size_t sharedplanorysize =
        (bin_size_x + 2 * (int)ceil(ns / 2.0)) * (bin_size_y + 2 * (int)ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_Subprob_Horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, sigma, d_binstartpts, d_binsize,
                bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins[0],
                numbins[1], d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_2d_Subprob (%d)\t\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}
template int cuspread2d<float>(cufinufft_plan_template<float> d_plan, int blksize);
template int cuspread2d<double>(cufinufft_plan_template<double> d_plan, int blksize);
template int cuspread2d_subprob_prop<float>(int nf1, int nf2, int M, cufinufft_plan_template<float> d_plan);
template int cuspread2d_subprob_prop<double>(int nf1, int nf2, int M, cufinufft_plan_template<double> d_plan);
template int cuspread2d_nuptsdriven_prop<float>(int nf1, int nf2, int M, cufinufft_plan_template<float> d_plan);
template int cuspread2d_nuptsdriven_prop<double>(int nf1, int nf2, int M, cufinufft_plan_template<double> d_plan);

} // namespace spreadinterp
} // namespace cufinufft
