#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>

#include "spreadinterp2d.cuh"

using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cuspread2d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int M = d_plan->M;

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuspread2d_nuptsdriven<T>(nf1, nf2, M, d_plan, blksize);
    } break;
    case 2: {
        ier = cuspread2d_subprob<T>(nf1, nf2, M, d_plan, blksize);
    } break;
    default:
        std::cerr << "[cuspread2d] error: incorrect method, should be 1 or 2\n";
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

    return ier;
}

template <typename T>
int cuspread2d_nuptsdriven_prop(int nf1, int nf2, int M, cufinufft_plan_t<T> *d_plan) {
    auto &stream = d_plan->stream;

    if (d_plan->opts.gpu_sort) {
        int bin_size_x = d_plan->opts.gpu_binsizex;
        int bin_size_y = d_plan->opts.gpu_binsizey;
        if (bin_size_x < 0 || bin_size_y < 0) {
            std::cerr << "[cuspread2d_nuptsdriven_prop] error: invalid binsize (binsizex, binsizey) = (";
            std::cerr << bin_size_x << "," << bin_size_y << ")" << std::endl;
            return FINUFFT_ERR_BINSIZE_NOTVALID;
        }

        int numbins[2];
        numbins[0] = ceil((T)nf1 / bin_size_x);
        numbins[1] = ceil((T)nf2 / bin_size_y);

        T *d_kx = d_plan->kx;
        T *d_ky = d_plan->ky;

        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;
        int ier;
        if ((ier = checkCudaErrors(hipMemsetAsync(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int), stream))))
            return ier;

        calc_bin_size_noghost_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
            M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], numbins[1], d_binsize, d_kx, d_ky, d_sortidx, pirange);
        RETURN_IF_CUDA_ERROR

        int n = numbins[0] * numbins[1];
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

        calc_inverse_of_global_sort_index_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
            M, bin_size_x, bin_size_y, numbins[0], numbins[1], d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts,
            pirange, nf1, nf2);
        RETURN_IF_CUDA_ERROR
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        trivial_global_sort_index_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(M, d_idxnupts);
        RETURN_IF_CUDA_ERROR
    }

    return 0;
}

template <typename T>
int cuspread2d_nuptsdriven(int nf1, int nf2, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;
    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            spread_2d_nupts_driven<T, 1>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2,
                                                         es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            spread_2d_nupts_driven<T, 0>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2,
                                                         es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template <typename T>
int cuspread2d_subprob_prop(int nf1, int nf2, int M, cufinufft_plan_t<T> *d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{
    auto &stream = d_plan->stream;

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    if (bin_size_x < 0 || bin_size_y < 0) {
        std::cerr << "[cuspread2d_subprob_prop] error: invalid binsize (binsizex, binsizey) = (";
        std::cerr << bin_size_x << "," << bin_size_y << ")" << std::endl;
        return FINUFFT_ERR_BINSIZE_NOTVALID;
    }
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;

    int pirange = d_plan->spopts.pirange;
    int ier;
    if ((ier = checkCudaErrors(hipMemsetAsync(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int), stream))))
        return ier;

    calc_bin_size_noghost_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], numbins[1], d_binsize, d_kx, d_ky, d_sortidx, pirange);
    RETURN_IF_CUDA_ERROR

    int n = numbins[0] * numbins[1];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

    calc_inverse_of_global_sort_index_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, bin_size_y, numbins[0], numbins[1], d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts, pirange,
        nf1, nf2);
    RETURN_IF_CUDA_ERROR
    calc_subprob_2d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(d_binsize, d_numsubprob, maxsubprobsize,
                                                                numbins[0] * numbins[1]);
    RETURN_IF_CUDA_ERROR

    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

    if ((ier = checkCudaErrors(hipMemsetAsync(d_subprobstartpts, 0, sizeof(int), stream))))
        return ier;

    int totalnumsubprob;
    if ((ier = checkCudaErrors(
             hipMemcpyAsync(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost, stream))))
        return ier;
    hipStreamSynchronize(stream);
    if ((ier = checkCudaErrors(hipMallocAsync(&d_subprob_to_bin, totalnumsubprob * sizeof(int), stream))))
        return ier;
    map_b_into_subprob_2d<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024, 0, stream>>>(
        d_subprob_to_bin, d_subprobstartpts, d_numsubprob, numbins[0] * numbins[1]);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[%s] Error: %s\n", __func__, hipGetErrorString(err));
        hipFree(d_subprob_to_bin);
        return FINUFFT_ERR_CUDA_FAILURE;
    }

    assert(d_subprob_to_bin != NULL);
    hipFreeAsync(d_plan->subprob_to_bin, stream);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    d_plan->totalnumsubprob = totalnumsubprob;

    return 0;
}

template <typename T>
int cuspread2d_subprob(int nf1, int nf2, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    int pirange = d_plan->spopts.pirange;

    T sigma = d_plan->opts.upsampfac;

    size_t sharedplanorysize =
        (bin_size_x + 2 * (int)ceil(ns / 2.0)) * (bin_size_y + 2 * (int)ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cerr << "[cuspread2d_subprob] error: not enough shared memory\n";
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            spread_2d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            spread_2d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template int cuspread2d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuspread2d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cuspread2d_subprob_prop<float>(int nf1, int nf2, int M, cufinufft_plan_t<float> *d_plan);
template int cuspread2d_subprob_prop<double>(int nf1, int nf2, int M, cufinufft_plan_t<double> *d_plan);
template int cuspread2d_nuptsdriven_prop<float>(int nf1, int nf2, int M, cufinufft_plan_t<float> *d_plan);
template int cuspread2d_nuptsdriven_prop<double>(int nf1, int nf2, int M, cufinufft_plan_t<double> *d_plan);

} // namespace spreadinterp
} // namespace cufinufft
