#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/extrema.h>

#include <cufinufft/defs.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/utils.h>
using namespace cufinufft::utils;

namespace cufinufft {
namespace spreadinterp {
/* ------------------------ 2d Spreading Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */

__global__ void spread_2d_nupts_driven(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
                                       int nf1, int nf2, CUFINUFFT_FLT es_c, CUFINUFFT_FLT es_beta, int *idxnupts,
                                       int pirange) {
    int xstart, ystart, xend, yend;
    int xx, yy, ix, iy;
    int outidx;
    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUFINUFFT_FLT kervalue1, kervalue2;
    CUCPX cnow;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
        x_rescaled = RESCALE(x[idxnupts[i]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[i]], nf2, pirange);
        cnow = c[idxnupts[i]];

        xstart = ceil(x_rescaled - ns / 2.0);
        ystart = ceil(y_rescaled - ns / 2.0);
        xend = floor(x_rescaled + ns / 2.0);
        yend = floor(y_rescaled + ns / 2.0);

        CUFINUFFT_FLT x1 = (CUFINUFFT_FLT)xstart - x_rescaled;
        CUFINUFFT_FLT y1 = (CUFINUFFT_FLT)ystart - y_rescaled;
        eval_kernel_vec(ker1, x1, ns, es_c, es_beta);
        eval_kernel_vec(ker2, y1, ns, es_c, es_beta);
        for (yy = ystart; yy <= yend; yy++) {
            for (xx = xstart; xx <= xend; xx++) {
                ix = xx < 0 ? xx + nf1 : (xx > nf1 - 1 ? xx - nf1 : xx);
                iy = yy < 0 ? yy + nf2 : (yy > nf2 - 1 ? yy - nf2 : yy);
                outidx = ix + iy * nf1;
                kervalue1 = ker1[xx - xstart];
                kervalue2 = ker2[yy - ystart];
                atomicAdd(&fw[outidx].x, cnow.x * kervalue1 * kervalue2);
                atomicAdd(&fw[outidx].y, cnow.y * kervalue1 * kervalue2);
            }
        }
    }
}

__global__ void spread_2d_nupts_driven_horner(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M,
                                              const int ns, int nf1, int nf2, CUFINUFFT_FLT sigma, int *idxnupts,
                                              int pirange) {
    int xx, yy, ix, iy;
    int outidx;
    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];
    CUFINUFFT_FLT ker1val, ker2val;

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUCPX cnow;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
        x_rescaled = RESCALE(x[idxnupts[i]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[i]], nf2, pirange);
        cnow = c[idxnupts[i]];
        int xstart = ceil(x_rescaled - ns / 2.0);
        int ystart = ceil(y_rescaled - ns / 2.0);
        int xend = floor(x_rescaled + ns / 2.0);
        int yend = floor(y_rescaled + ns / 2.0);

        CUFINUFFT_FLT x1 = (CUFINUFFT_FLT)xstart - x_rescaled;
        CUFINUFFT_FLT y1 = (CUFINUFFT_FLT)ystart - y_rescaled;
        eval_kernel_vec_horner(ker1, x1, ns, sigma);
        eval_kernel_vec_horner(ker2, y1, ns, sigma);
        for (yy = ystart; yy <= yend; yy++) {
            for (xx = xstart; xx <= xend; xx++) {
                ix = xx < 0 ? xx + nf1 : (xx > nf1 - 1 ? xx - nf1 : xx);
                iy = yy < 0 ? yy + nf2 : (yy > nf2 - 1 ? yy - nf2 : yy);
                outidx = ix + iy * nf1;
                ker1val = ker1[xx - xstart];
                ker2val = ker2[yy - ystart];
                CUFINUFFT_FLT kervalue = ker1val * ker2val;
                atomicAdd(&fw[outidx].x, cnow.x * kervalue);
                atomicAdd(&fw[outidx].y, cnow.y * kervalue);
            }
        }
    }
}

/* Kernels for SubProb Method */
// SubProb properties
__global__ void calc_bin_size_noghost_2d(int M, int nf1, int nf2, int bin_size_x, int bin_size_y, int nbinx, int nbiny,
                                         int *bin_size, CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, int *sortidx, int pirange) {
    int binidx, binx, biny;
    int oldidx;
    CUFINUFFT_FLT x_rescaled, y_rescaled;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < M; i += gridDim.x * blockDim.x) {
        x_rescaled = RESCALE(x[i], nf1, pirange);
        y_rescaled = RESCALE(y[i], nf2, pirange);
        binx = floor(x_rescaled / bin_size_x);
        binx = binx >= nbinx ? binx - 1 : binx;
        binx = binx < 0 ? 0 : binx;
        biny = floor(y_rescaled / bin_size_y);
        biny = biny >= nbiny ? biny - 1 : biny;
        biny = biny < 0 ? 0 : biny;
        binidx = binx + biny * nbinx;
        oldidx = atomicAdd(&bin_size[binidx], 1);
        sortidx[i] = oldidx;
        if (binx >= nbinx || biny >= nbiny) {
            sortidx[i] = -biny;
        }
    }
}

__global__ void calc_inverse_of_global_sort_index_2d(int M, int bin_size_x, int bin_size_y, int nbinx, int nbiny,
                                                     int *bin_startpts, int *sortidx, CUFINUFFT_FLT *x,
                                                     CUFINUFFT_FLT *y, int *index, int pirange, int nf1, int nf2) {
    int binx, biny;
    int binidx;
    CUFINUFFT_FLT x_rescaled, y_rescaled;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < M; i += gridDim.x * blockDim.x) {
        x_rescaled = RESCALE(x[i], nf1, pirange);
        y_rescaled = RESCALE(y[i], nf2, pirange);
        binx = floor(x_rescaled / bin_size_x);
        binx = binx >= nbinx ? binx - 1 : binx;
        binx = binx < 0 ? 0 : binx;
        biny = floor(y_rescaled / bin_size_y);
        biny = biny >= nbiny ? biny - 1 : biny;
        biny = biny < 0 ? 0 : biny;
        binidx = binx + biny * nbinx;

        index[bin_startpts[binidx] + sortidx[i]] = i;
    }
}

__global__ void spread_2d_subprob(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns, int nf1,
                                  int nf2, CUFINUFFT_FLT es_c, CUFINUFFT_FLT es_beta, CUFINUFFT_FLT sigma,
                                  int *binstartpts, int *bin_size, int bin_size_x, int bin_size_y, int *subprob_to_bin,
                                  int *subprobstartpts, int *numsubprob, int maxsubprobsize, int nbinx, int nbiny,
                                  int *idxnupts, int pirange) {
    extern __shared__ CUCPX fwshared[];

    int xstart, ystart, xend, yend;
    int subpidx = blockIdx.x;
    int bidx = subprob_to_bin[subpidx];
    int binsubp_idx = subpidx - subprobstartpts[bidx];
    int ix, iy;
    int outidx;
    int ptstart = binstartpts[bidx] + binsubp_idx * maxsubprobsize;
    int nupts = min(maxsubprobsize, bin_size[bidx] - binsubp_idx * maxsubprobsize);

    int xoffset = (bidx % nbinx) * bin_size_x;
    int yoffset = (bidx / nbinx) * bin_size_y;

    int N = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0));
    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];

    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        fwshared[i].x = 0.0;
        fwshared[i].y = 0.0;
    }
    __syncthreads();

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUCPX cnow;
    for (int i = threadIdx.x; i < nupts; i += blockDim.x) {
        int idx = ptstart + i;
        x_rescaled = RESCALE(x[idxnupts[idx]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[idx]], nf2, pirange);
        cnow = c[idxnupts[idx]];

        xstart = ceil(x_rescaled - ns / 2.0) - xoffset;
        ystart = ceil(y_rescaled - ns / 2.0) - yoffset;
        xend = floor(x_rescaled + ns / 2.0) - xoffset;
        yend = floor(y_rescaled + ns / 2.0) - yoffset;

        CUFINUFFT_FLT x1 = (CUFINUFFT_FLT)xstart + xoffset - x_rescaled;
        CUFINUFFT_FLT y1 = (CUFINUFFT_FLT)ystart + yoffset - y_rescaled;
        eval_kernel_vec(ker1, x1, ns, es_c, es_beta);
        eval_kernel_vec(ker2, y1, ns, es_c, es_beta);

        for (int yy = ystart; yy <= yend; yy++) {
            iy = yy + ceil(ns / 2.0);
            if (iy >= (bin_size_y + (int)ceil(ns / 2.0) * 2) || iy < 0)
                break;
            for (int xx = xstart; xx <= xend; xx++) {
                ix = xx + ceil(ns / 2.0);
                if (ix >= (bin_size_x + (int)ceil(ns / 2.0) * 2) || ix < 0)
                    break;
                outidx = ix + iy * (bin_size_x + ceil(ns / 2.0) * 2);
                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
                atomicAdd(&fwshared[outidx].x, cnow.x * kervalue1 * kervalue2);
                atomicAdd(&fwshared[outidx].y, cnow.y * kervalue1 * kervalue2);
            }
        }
    }
    __syncthreads();
    /* write to global memory */
    for (int k = threadIdx.x; k < N; k += blockDim.x) {
        int i = k % (int)(bin_size_x + 2 * ceil(ns / 2.0));
        int j = k / (bin_size_x + 2 * ceil(ns / 2.0));
        ix = xoffset - ceil(ns / 2.0) + i;
        iy = yoffset - ceil(ns / 2.0) + j;
        if (ix < (nf1 + ceil(ns / 2.0)) && iy < (nf2 + ceil(ns / 2.0))) {
            ix = ix < 0 ? ix + nf1 : (ix > nf1 - 1 ? ix - nf1 : ix);
            iy = iy < 0 ? iy + nf2 : (iy > nf2 - 1 ? iy - nf2 : iy);
            outidx = ix + iy * nf1;
            int sharedidx = i + j * (bin_size_x + ceil(ns / 2.0) * 2);
            atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
            atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
        }
    }
}

__global__ void spread_2d_subprob_horner(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
                                         int nf1, int nf2, CUFINUFFT_FLT sigma, int *binstartpts, int *bin_size,
                                         int bin_size_x, int bin_size_y, int *subprob_to_bin, int *subprobstartpts,
                                         int *numsubprob, int maxsubprobsize, int nbinx, int nbiny, int *idxnupts,
                                         int pirange) {
    extern __shared__ CUCPX fwshared[];

    int xstart, ystart, xend, yend;
    int subpidx = blockIdx.x;
    int bidx = subprob_to_bin[subpidx];
    int binsubp_idx = subpidx - subprobstartpts[bidx];
    int ix, iy, outidx;
    int ptstart = binstartpts[bidx] + binsubp_idx * maxsubprobsize;
    int nupts = min(maxsubprobsize, bin_size[bidx] - binsubp_idx * maxsubprobsize);

    int xoffset = (bidx % nbinx) * bin_size_x;
    int yoffset = (bidx / nbinx) * bin_size_y;

    int N = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0));

    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];

    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        fwshared[i].x = 0.0;
        fwshared[i].y = 0.0;
    }
    __syncthreads();

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUCPX cnow;
    for (int i = threadIdx.x; i < nupts; i += blockDim.x) {
        int idx = ptstart + i;
        x_rescaled = RESCALE(x[idxnupts[idx]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[idx]], nf2, pirange);
        cnow = c[idxnupts[idx]];

        xstart = ceil(x_rescaled - ns / 2.0) - xoffset;
        ystart = ceil(y_rescaled - ns / 2.0) - yoffset;
        xend = floor(x_rescaled + ns / 2.0) - xoffset;
        yend = floor(y_rescaled + ns / 2.0) - yoffset;

        eval_kernel_vec_horner(ker1, xstart + xoffset - x_rescaled, ns, sigma);
        eval_kernel_vec_horner(ker2, ystart + yoffset - y_rescaled, ns, sigma);

        for (int yy = ystart; yy <= yend; yy++) {
            iy = yy + ceil(ns / 2.0);
            if (iy >= (bin_size_y + (int)ceil(ns / 2.0) * 2) || iy < 0)
                break;
            CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
            for (int xx = xstart; xx <= xend; xx++) {
                ix = xx + ceil(ns / 2.0);
                if (ix >= (bin_size_x + (int)ceil(ns / 2.0) * 2) || ix < 0)
                    break;
                outidx = ix + iy * (bin_size_x + (int)ceil(ns / 2.0) * 2);
                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                atomicAdd(&fwshared[outidx].x, cnow.x * kervalue1 * kervalue2);
                atomicAdd(&fwshared[outidx].y, cnow.y * kervalue1 * kervalue2);
            }
        }
    }
    __syncthreads();

    /* write to global memory */
    for (int k = threadIdx.x; k < N; k += blockDim.x) {
        int i = k % (int)(bin_size_x + 2 * ceil(ns / 2.0));
        int j = k / (bin_size_x + 2 * ceil(ns / 2.0));
        ix = xoffset - ceil(ns / 2.0) + i;
        iy = yoffset - ceil(ns / 2.0) + j;
        if (ix < (nf1 + ceil(ns / 2.0)) && iy < (nf2 + ceil(ns / 2.0))) {
            ix = ix < 0 ? ix + nf1 : (ix > nf1 - 1 ? ix - nf1 : ix);
            iy = iy < 0 ? iy + nf2 : (iy > nf2 - 1 ? iy - nf2 : iy);
            outidx = ix + iy * nf1;
            int sharedidx = i + j * (bin_size_x + ceil(ns / 2.0) * 2);
            atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
            atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
        }
    }
}

/* --------------------- 2d Interpolation Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */
__global__ void interp_2d_nupts_driven(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
                                       int nf1, int nf2, CUFINUFFT_FLT es_c, CUFINUFFT_FLT es_beta, int *idxnupts,
                                       int pirange) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
        CUFINUFFT_FLT x_rescaled = RESCALE(x[idxnupts[i]], nf1, pirange);
        CUFINUFFT_FLT y_rescaled = RESCALE(y[idxnupts[i]], nf2, pirange);

        int xstart = ceil(x_rescaled - ns / 2.0);
        int ystart = ceil(y_rescaled - ns / 2.0);
        int xend = floor(x_rescaled + ns / 2.0);
        int yend = floor(y_rescaled + ns / 2.0);
        CUCPX cnow;
        cnow.x = 0.0;
        cnow.y = 0.0;
        CUFINUFFT_FLT ker1[MAX_NSPREAD];
        CUFINUFFT_FLT ker2[MAX_NSPREAD];

        CUFINUFFT_FLT x1 = (CUFINUFFT_FLT)xstart - x_rescaled;
        CUFINUFFT_FLT y1 = (CUFINUFFT_FLT)ystart - y_rescaled;
        eval_kernel_vec(ker1, x1, ns, es_c, es_beta);
        eval_kernel_vec(ker2, y1, ns, es_c, es_beta);

        for (int yy = ystart; yy <= yend; yy++) {
            CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
            for (int xx = xstart; xx <= xend; xx++) {
                int ix = xx < 0 ? xx + nf1 : (xx > nf1 - 1 ? xx - nf1 : xx);
                int iy = yy < 0 ? yy + nf2 : (yy > nf2 - 1 ? yy - nf2 : yy);
                int inidx = ix + iy * nf1;
                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                cnow.x += fw[inidx].x * kervalue1 * kervalue2;
                cnow.y += fw[inidx].y * kervalue1 * kervalue2;
            }
        }
        c[idxnupts[i]].x = cnow.x;
        c[idxnupts[i]].y = cnow.y;
    }
}

__global__ void interp_2d_nupts_driven_horner(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M,
                                              const int ns, int nf1, int nf2, CUFINUFFT_FLT sigma, int *idxnupts,
                                              int pirange) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
        CUFINUFFT_FLT x_rescaled = RESCALE(x[idxnupts[i]], nf1, pirange);
        CUFINUFFT_FLT y_rescaled = RESCALE(y[idxnupts[i]], nf2, pirange);

        int xstart = ceil(x_rescaled - ns / 2.0);
        int ystart = ceil(y_rescaled - ns / 2.0);
        int xend = floor(x_rescaled + ns / 2.0);
        int yend = floor(y_rescaled + ns / 2.0);

        CUCPX cnow;
        cnow.x = 0.0;
        cnow.y = 0.0;
        CUFINUFFT_FLT ker1[MAX_NSPREAD];
        CUFINUFFT_FLT ker2[MAX_NSPREAD];

        eval_kernel_vec_horner(ker1, xstart - x_rescaled, ns, sigma);
        eval_kernel_vec_horner(ker2, ystart - y_rescaled, ns, sigma);

        for (int yy = ystart; yy <= yend; yy++) {
            CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
            for (int xx = xstart; xx <= xend; xx++) {
                int ix = xx < 0 ? xx + nf1 : (xx > nf1 - 1 ? xx - nf1 : xx);
                int iy = yy < 0 ? yy + nf2 : (yy > nf2 - 1 ? yy - nf2 : yy);
                int inidx = ix + iy * nf1;
                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                cnow.x += fw[inidx].x * kervalue1 * kervalue2;
                cnow.y += fw[inidx].y * kervalue1 * kervalue2;
            }
        }
        c[idxnupts[i]].x = cnow.x;
        c[idxnupts[i]].y = cnow.y;
    }
}

/* Kernels for Subprob Method */
__global__ void interp_2d_subprob(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns, int nf1,
                                  int nf2, CUFINUFFT_FLT es_c, CUFINUFFT_FLT es_beta, CUFINUFFT_FLT sigma,
                                  int *binstartpts, int *bin_size, int bin_size_x, int bin_size_y, int *subprob_to_bin,
                                  int *subprobstartpts, int *numsubprob, int maxsubprobsize, int nbinx, int nbiny,
                                  int *idxnupts, int pirange) {
    extern __shared__ CUCPX fwshared[];

    int xstart, ystart, xend, yend;
    int subpidx = blockIdx.x;
    int bidx = subprob_to_bin[subpidx];
    int binsubp_idx = subpidx - subprobstartpts[bidx];
    int ix, iy;
    int outidx;
    int ptstart = binstartpts[bidx] + binsubp_idx * maxsubprobsize;
    int nupts = min(maxsubprobsize, bin_size[bidx] - binsubp_idx * maxsubprobsize);

    int xoffset = (bidx % nbinx) * bin_size_x;
    int yoffset = (bidx / nbinx) * bin_size_y;
    int N = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0));

    for (int k = threadIdx.x; k < N; k += blockDim.x) {
        int i = k % (int)(bin_size_x + 2 * ceil(ns / 2.0));
        int j = k / (bin_size_x + 2 * ceil(ns / 2.0));
        ix = xoffset - ceil(ns / 2.0) + i;
        iy = yoffset - ceil(ns / 2.0) + j;
        if (ix < (nf1 + ceil(ns / 2.0)) && iy < (nf2 + ceil(ns / 2.0))) {
            ix = ix < 0 ? ix + nf1 : (ix > nf1 - 1 ? ix - nf1 : ix);
            iy = iy < 0 ? iy + nf2 : (iy > nf2 - 1 ? iy - nf2 : iy);
            outidx = ix + iy * nf1;
            int sharedidx = i + j * (bin_size_x + ceil(ns / 2.0) * 2);
            fwshared[sharedidx].x = fw[outidx].x;
            fwshared[sharedidx].y = fw[outidx].y;
        }
    }
    __syncthreads();

    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUCPX cnow;
    for (int i = threadIdx.x; i < nupts; i += blockDim.x) {
        int idx = ptstart + i;
        x_rescaled = RESCALE(x[idxnupts[idx]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[idx]], nf2, pirange);
        cnow.x = 0.0;
        cnow.y = 0.0;

        xstart = ceil(x_rescaled - ns / 2.0) - xoffset;
        ystart = ceil(y_rescaled - ns / 2.0) - yoffset;
        xend = floor(x_rescaled + ns / 2.0) - xoffset;
        yend = floor(y_rescaled + ns / 2.0) - yoffset;

        CUFINUFFT_FLT x1 = (CUFINUFFT_FLT)xstart + xoffset - x_rescaled;
        CUFINUFFT_FLT y1 = (CUFINUFFT_FLT)ystart + yoffset - y_rescaled;

        eval_kernel_vec(ker1, x1, ns, es_c, es_beta);
        eval_kernel_vec(ker2, y1, ns, es_c, es_beta);
        for (int yy = ystart; yy <= yend; yy++) {
            CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
            for (int xx = xstart; xx <= xend; xx++) {
                ix = xx + ceil(ns / 2.0);
                iy = yy + ceil(ns / 2.0);
                outidx = ix + iy * (bin_size_x + ceil(ns / 2.0) * 2);
                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                cnow.x += fwshared[outidx].x * kervalue1 * kervalue2;
                cnow.y += fwshared[outidx].y * kervalue1 * kervalue2;
            }
        }
        c[idxnupts[idx]] = cnow;
    }
}

__global__ void interp_2d_subprob_horner(CUFINUFFT_FLT *x, CUFINUFFT_FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
                                         int nf1, int nf2, CUFINUFFT_FLT sigma, int *binstartpts, int *bin_size,
                                         int bin_size_x, int bin_size_y, int *subprob_to_bin, int *subprobstartpts,
                                         int *numsubprob, int maxsubprobsize, int nbinx, int nbiny, int *idxnupts,
                                         int pirange) {
    extern __shared__ CUCPX fwshared[];

    int xstart, ystart, xend, yend;
    int subpidx = blockIdx.x;
    int bidx = subprob_to_bin[subpidx];
    int binsubp_idx = subpidx - subprobstartpts[bidx];
    int ix, iy;
    int outidx;
    int ptstart = binstartpts[bidx] + binsubp_idx * maxsubprobsize;
    int nupts = min(maxsubprobsize, bin_size[bidx] - binsubp_idx * maxsubprobsize);

    int xoffset = (bidx % nbinx) * bin_size_x;
    int yoffset = (bidx / nbinx) * bin_size_y;

    int N = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0));

    for (int k = threadIdx.x; k < N; k += blockDim.x) {
        int i = k % (int)(bin_size_x + 2 * ceil(ns / 2.0));
        int j = k / (bin_size_x + 2 * ceil(ns / 2.0));
        ix = xoffset - ceil(ns / 2.0) + i;
        iy = yoffset - ceil(ns / 2.0) + j;
        if (ix < (nf1 + ceil(ns / 2.0)) && iy < (nf2 + ceil(ns / 2.0))) {
            ix = ix < 0 ? ix + nf1 : (ix > nf1 - 1 ? ix - nf1 : ix);
            iy = iy < 0 ? iy + nf2 : (iy > nf2 - 1 ? iy - nf2 : iy);
            outidx = ix + iy * nf1;
            int sharedidx = i + j * (bin_size_x + ceil(ns / 2.0) * 2);
            fwshared[sharedidx].x = fw[outidx].x;
            fwshared[sharedidx].y = fw[outidx].y;
        }
    }
    __syncthreads();

    CUFINUFFT_FLT ker1[MAX_NSPREAD];
    CUFINUFFT_FLT ker2[MAX_NSPREAD];

    CUFINUFFT_FLT x_rescaled, y_rescaled;
    CUCPX cnow;
    for (int i = threadIdx.x; i < nupts; i += blockDim.x) {
        int idx = ptstart + i;
        x_rescaled = RESCALE(x[idxnupts[idx]], nf1, pirange);
        y_rescaled = RESCALE(y[idxnupts[idx]], nf2, pirange);
        cnow.x = 0.0;
        cnow.y = 0.0;

        xstart = ceil(x_rescaled - ns / 2.0) - xoffset;
        ystart = ceil(y_rescaled - ns / 2.0) - yoffset;
        xend = floor(x_rescaled + ns / 2.0) - xoffset;
        yend = floor(y_rescaled + ns / 2.0) - yoffset;

        eval_kernel_vec_horner(ker1, xstart + xoffset - x_rescaled, ns, sigma);
        eval_kernel_vec_horner(ker2, ystart + yoffset - y_rescaled, ns, sigma);

        for (int yy = ystart; yy <= yend; yy++) {
            CUFINUFFT_FLT kervalue2 = ker2[yy - ystart];
            for (int xx = xstart; xx <= xend; xx++) {
                ix = xx + ceil(ns / 2.0);
                iy = yy + ceil(ns / 2.0);
                outidx = ix + iy * (bin_size_x + ceil(ns / 2.0) * 2);

                CUFINUFFT_FLT kervalue1 = ker1[xx - xstart];
                cnow.x += fwshared[outidx].x * kervalue1 * kervalue2;
                cnow.y += fwshared[outidx].y * kervalue1 * kervalue2;
            }
        }
        c[idxnupts[idx]] = cnow;
    }
}

} // namespace spreadinterp
} // namespace cufinufft
