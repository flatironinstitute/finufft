#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::common;

namespace cufinufft {
namespace spreadinterp {
// only relates to the locations of the nodes, which only needs to be done once
template <typename T>
int cuspread2d_paul_prop(int nf1, int nf2, int M, cufinufft_plan_template<T> *d_plan) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
#ifdef DEBUG
    std::cout << "[debug ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[debug ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
#ifdef DEBUG
    T *h_kx;
    T *h_ky;
    h_kx = (T *)malloc(M * sizeof(T));
    h_ky = (T *)malloc(M * sizeof(T));

    checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_ky, d_ky, M * sizeof(T), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ]";
        std::cout << " (" << setw(3) << h_kx[i] << "," << setw(3) << h_ky[i] << ")" << std::endl;
    }
#endif
    int *d_binsize = d_plan->binsize;
    int *d_finegridsize = d_plan->finegridsize;
    int *d_sortidx = d_plan->sortidx;
    int *d_fgstartpts = d_plan->fgstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_numsubprob = d_plan->numsubprob;

    int pirange = d_plan->spopts.pirange;

    void *d_temp_storage = NULL;

    hipEventRecord(start);
    checkCudaErrors(hipMemset(d_finegridsize, 0, nf1 * nf2 * sizeof(int)));
    LocateFineGridPos_Paul<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                            d_binsize, ns, d_kx, d_ky, d_sortidx, d_finegridsize,
                                                            pirange);
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel LocateFineGridPos \t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    printf("[debug ] ns = %d\n", ns);
    int binx, biny, binidx;
    int *h_finegridsize;
    h_finegridsize = (int *)malloc(nf1 * nf2 * sizeof(int));

    checkCudaErrors(hipMemcpy(h_finegridsize, d_finegridsize, nf1 * nf2 * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < nf2; j++) {
        if (j % d_plan->opts.gpu_binsizey == 0)
            printf("\n");
        biny = floor(j / bin_size_y);
        std::cout << "[debug ] ";
        for (int i = 0; i < nf1; i++) {
            if (i % d_plan->opts.gpu_binsizex == 0 && i != 0)
                printf(" |");
            binx = floor(i / bin_size_x);
            binidx = binx + biny * numbins[0];
            if (i != 0)
                std::cout << " ";
            std::cout << setw(2)
                      << h_finegridsize[binidx * bin_size_x * bin_size_y + (i - binx * bin_size_x) +
                                        (j - bin_size_y * biny) * bin_size_x];
        }
        std::cout << std::endl;
    }
    std::cout << "[debug ] ------------------------------------------------" << std::endl;

    free(h_finegridsize);
#endif
#ifdef DEBUG
    int *h_binsize; // For debug
    h_binsize = (int *)malloc(numbins[0] * numbins[1] * sizeof(int));
    checkCudaErrors(hipMemcpy(h_binsize, d_binsize, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] bin size:" << std::endl;
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << " bin[" << setw(3) << i << "," << setw(3) << j << "]=" << h_binsize[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_binsize);
#endif
#ifdef DEBUG
    std::cout << "[debug ] ------------------------------------------------" << std::endl;
    int *h_sortidx;
    h_sortidx = (int *)malloc(M * sizeof(int));

    checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ]";
    for (int i = 0; i < M; i++) {
        std::cout << "point[" << setw(3) << i << "]=" << setw(3) << h_sortidx[i] << std::endl;
    }
#endif
    int n = nf1 * nf2;
    hipEventRecord(start);
    thrust::device_ptr<int> d_ptr(d_finegridsize);
    thrust::device_ptr<int> d_result(d_fgstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Scan fingridsize array\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_fgstartpts;
    h_fgstartpts = (int *)malloc((nf1 * nf2) * sizeof(int));
    checkCudaErrors(hipMemcpy(h_fgstartpts, d_fgstartpts, (nf1 * nf2) * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] Result of scan finegridsize array:" << std::endl;
    for (int j = 0; j < nf2; j++) {
        if (j % d_plan->opts.gpu_binsizey == 0)
            printf("\n");
        biny = floor(j / bin_size_y);
        std::cout << "[debug ] ";
        for (int i = 0; i < nf1; i++) {
            if (i % d_plan->opts.gpu_binsizex == 0 && i != 0)
                printf(" |");
            binx = floor(i / bin_size_x);
            binidx = binx + biny * numbins[0];
            if (i != 0)
                std::cout << " ";
            std::cout << setw(2)
                      << h_fgstartpts[binidx * bin_size_x * bin_size_y + (i - binx * bin_size_x) +
                                      (j - bin_size_y * biny) * bin_size_x];
        }
        std::cout << std::endl;
    }
    free(h_fgstartpts);
    std::cout << "[debug ] -----------------------------------------------" << std::endl;
#endif
    hipEventRecord(start);
    CalcInvertofGlobalSortIdx_Paul<<<(M + 1024 - 1) / 1024, 1024>>>(nf1, nf2, M, bin_size_x, bin_size_y, numbins[0],
                                                                    numbins[1], ns, d_kx, d_ky, d_fgstartpts, d_sortidx,
                                                                    d_idxnupts, pirange);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tCalcInvertofGlobalSortIdx_Paul\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_idxnupts;
    h_idxnupts = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "idx=" << h_idxnupts[i] << " ";
    }
    std::cout << std::endl;
    free(h_idxnupts);
#endif
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    hipEventRecord(start);
    int blocksize = bin_size_x * bin_size_y;
    hipEventRecord(start);
    CalcSubProb_2d_Paul<<<numbins[0] * numbins[1], blocksize>>>(d_finegridsize, d_numsubprob, maxsubprobsize,
                                                                bin_size_x, bin_size_y);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tCalcSubProb_2d_Paul\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_numsubprob;
    h_numsubprob = (int *)malloc(n * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_numsubprob, d_numsubprob, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << setw(3) << i << "," << setw(3) << j << "] = " << setw(2)
                      << h_numsubprob[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_numsubprob);
#endif
    int *d_subprobstartpts = d_plan->subprobstartpts;
    n = numbins[0] * numbins[1];
    hipEventRecord(start);
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tScan subproblem size array\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
    printf("[debug ] Subproblem start points\n");
    int *h_subprobstartpts;
    h_subprobstartpts = (int *)malloc((n + 1) * sizeof(int));
    checkCudaErrors(hipMemcpy(h_subprobstartpts, d_subprobstartpts, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << setw(3) << i << "," << setw(3) << j << "] = " << setw(2)
                      << h_subprobstartpts[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
    free(h_subprobstartpts);
#endif
    int *d_subprob_to_bin;
    int totalnumsubprob;
    hipEventRecord(start);
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    // TODO: Warning! This gets malloc'ed but not freed
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    MapBintoSubProb_2d<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024>>>(d_subprob_to_bin, d_subprobstartpts,
                                                                              d_numsubprob, numbins[0] * numbins[1]);
    assert(d_subprob_to_bin != NULL);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != NULL);
    d_plan->totalnumsubprob = totalnumsubprob;
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tMap Subproblem to Bins\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    printf("[debug ] Map Subproblem to Bins\n");
    int *h_subprob_to_bin;
    h_subprob_to_bin = (int *)malloc((totalnumsubprob) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_subprob_to_bin, d_subprob_to_bin, (totalnumsubprob) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < totalnumsubprob; j++) {
        std::cout << "[debug ] ";
        std::cout << "nsub[" << j << "] = " << setw(2) << h_subprob_to_bin[j];
        std::cout << std::endl;
    }
#endif
    hipFree(d_temp_storage);
    return 0;
}

template <typename T>
int cuspread2d_paul(int nf1, int nf2, int M, cufinufft_plan_template<T> *d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
#ifdef INFO
    std::cout << "[info  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[info  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_fgstartpts = d_plan->fgstartpts;
    int *d_finegridsize = d_plan->finegridsize;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    int pirange = d_plan->spopts.pirange;
    T sigma = d_plan->opts.upsampfac;
    hipEventRecord(start);
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }
    for (int t = 0; t < blksize; t++) {
        Spread_2d_Subprob_Paul<<<totalnumsubprob, 1024, sharedplanorysize>>>(
            d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
            d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
            numbins[0], numbins[1], d_idxnupts, d_fgstartpts, d_finegridsize, pirange);
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_2d_Subprob_Paul \t%.3g ms\n", milliseconds);
#endif
    return 0;
}

} // namespace spreadinterp
} // namespace cufinufft
