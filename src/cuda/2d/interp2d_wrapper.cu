#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/profile.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

int CUFINUFFT_INTERP2D(int nf1, int nf2, CUCPX *d_fw, int M, CUFINUFFT_FLT *d_kx, CUFINUFFT_FLT *d_ky, CUCPX *d_c,
                       CUFINUFFT_PLAN d_plan)
/*
    This c function is written for only doing 2D interpolation. See
    test/interp2d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    ier = ALLOCGPUMEM2D_PLAN(d_plan);
    ier = ALLOCGPUMEM2D_NUPTS(d_plan);
    if (d_plan->opts.gpu_method == 1) {
        ier = CUSPREAD2D_NUPTSDRIVEN_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 2) {
        ier = CUSPREAD2D_SUBPROB_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    ier = CUINTERP2D(d_plan, 1);

    FREEGPUMEMORY2D(d_plan);

    return ier;
}

int CUINTERP2D(CUFINUFFT_PLAN d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int M = d_plan->M;

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = CUINTERP2D_NUPTSDRIVEN(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread2d_gpu_nuptsdriven" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        ier = CUINTERP2D_SUBPROB(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cuinterp2d_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1 or 2" << std::endl;
        return 2;
    }

    return ier;
}

int CUINTERP2D_NUPTSDRIVEN(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan, int blksize) {
    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    CUFINUFFT_FLT es_c = d_plan->spopts.ES_c;
    CUFINUFFT_FLT es_beta = d_plan->spopts.ES_beta;
    CUFINUFFT_FLT sigma = d_plan->opts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;

    CUFINUFFT_FLT *d_kx = d_plan->kx;
    CUFINUFFT_FLT *d_ky = d_plan->ky;
    CUCPX *d_c = d_plan->c;
    CUCPX *d_fw = d_plan->fw;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            interp_2d_nupts_driven_horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M,
                                                                       ns, nf1, nf2, sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            interp_2d_nupts_driven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns,
                                                                nf1, nf2, es_c, es_beta, d_idxnupts, pirange);
        }
    }

    return 0;
}

int CUINTERP2D_SUBPROB(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan, int blksize) {
    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    CUFINUFFT_FLT es_c = d_plan->spopts.ES_c;
    CUFINUFFT_FLT es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((CUFINUFFT_FLT)nf1 / bin_size_x);
    numbins[1] = ceil((CUFINUFFT_FLT)nf2 / bin_size_y);

    CUFINUFFT_FLT *d_kx = d_plan->kx;
    CUFINUFFT_FLT *d_ky = d_plan->ky;
    CUCPX *d_c = d_plan->c;
    CUCPX *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;
    int totalnumsubprob = d_plan->totalnumsubprob;
    int pirange = d_plan->spopts.pirange;

    CUFINUFFT_FLT sigma = d_plan->opts.upsampfac;
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) * sizeof(CUCPX);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            interp_2d_subprob_horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, sigma, d_binstartpts, d_binsize,
                bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins[0],
                numbins[1], d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            interp_2d_subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
        }
    }

    return 0;
}

} // namespace spreadinterp
} // namespace cufinufft
