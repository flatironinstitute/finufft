#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/profile.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::memtransfer;

#include "spreadinterp2d.cuh"

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cufinufft_interp2d(int nf1, int nf2, cuda_complex<T> *d_fw, int M, T *d_kx, T *d_ky, cuda_complex<T> *d_c,
                       cufinufft_plan_template<T> d_plan)
/*
    This c function is written for only doing 2D interpolation. See
    test/interp2d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    hipEventRecord(start);
    ier = allocgpumem2d_plan<T>(d_plan);
    ier = allocgpumem2d_nupts<T>(d_plan);
    if (d_plan->opts.gpu_method == 1) {
        ier = cuspread2d_nuptsdriven_prop<T>(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 2) {
        ier = cuspread2d_subprob_prop<T>(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
#ifdef TIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Obtain Interp Prop\t %.3g ms\n", milliseconds);
#endif
    hipEventRecord(start);
    ier = cuinterp2d<T>(d_plan, 1);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Interp (%d)\t\t %.3g ms\n", d_plan->opts.gpu_method, milliseconds);
#endif
    hipEventRecord(start);
    freegpumemory2d<T>(d_plan);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
    return ier;
}

template <typename T>
int cuinterp2d(cufinufft_plan_template<T> d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int M = d_plan->M;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        hipEventRecord(start);
        {
            PROFILE_CUDA_GROUP("Spreading", 6);
            ier = cuinterp2d_nuptsdriven<T>(nf1, nf2, M, d_plan, blksize);
            if (ier != 0) {
                std::cout << "error: cnufftspread2d_gpu_nuptsdriven" << std::endl;
                return 1;
            }
        }
    } break;
    case 2: {
        hipEventRecord(start);
        ier = cuinterp2d_subprob<T>(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cuinterp2d_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1 or 2" << std::endl;
        return 2;
    }
#ifdef SPREADTIME
    float milliseconds;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "[time  ]"
              << " Interp " << milliseconds << " ms" << std::endl;
#endif
    return ier;
}

template <typename T>
int cuinterp2d_nuptsdriven(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->opts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    hipEventRecord(start);
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Interp_2d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M,
                                                                      ns, nf1, nf2, sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Interp_2d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns,
                                                               nf1, nf2, es_c, es_beta, d_idxnupts, pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Interp_2d_NUptsdriven (%d)\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

template <typename T>
int cuinterp2d_subprob(int nf1, int nf2, int M, cufinufft_plan_template<T> d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
#ifdef INFO
    std::cout << "[info  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[info  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;
    int totalnumsubprob = d_plan->totalnumsubprob;
    int pirange = d_plan->spopts.pirange;

    T sigma = d_plan->opts.upsampfac;
    hipEventRecord(start);
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Interp_2d_Subprob_Horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, sigma, d_binstartpts, d_binsize,
                bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins[0],
                numbins[1], d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Interp_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Interp_2d_Subprob (%d)\t\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

template int cufinufft_interp2d(int nf1, int nf2, cuda_complex<float> *d_fw, int M, float *d_kx, float *d_ky,
                                cuda_complex<float> *d_c, cufinufft_plan_template<float> d_plan);
template int cufinufft_interp2d(int nf1, int nf2, cuda_complex<double> *d_fw, int M, double *d_kx, double *d_ky,
                                cuda_complex<double> *d_c, cufinufft_plan_template<double> d_plan);

template int cuinterp2d<float>(cufinufft_plan_template<float> d_plan, int blksize);
template int cuinterp2d<double>(cufinufft_plan_template<double> d_plan, int blksize);

} // namespace spreadinterp
} // namespace cufinufft
