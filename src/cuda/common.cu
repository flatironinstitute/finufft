#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits>
#include <vector>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include <cufinufft/common.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <cufinufft/defs.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/utils.h>

namespace cufinufft {
namespace common {
using namespace cufinufft::spreadinterp;
using std::max;

/** Kernel for computing approximations of exact Fourier series coeffs of
 *  cnufftspread's real symmetric kernel.
 * phase, f are intermediate results from function onedim_fseries_kernel_precomp().
 * this is the equispaced frequency case, used by type 1 & 2, matching
 * onedim_fseries_kernel in CPU code. Used by functions below in this file.
 */
template<typename T>
__global__ void cu_fseries_kernel_compute(int nf1, int nf2, int nf3, T *f, T *phase,
                                          T *fwkerhalf1, T *fwkerhalf2, T *fwkerhalf3,
                                          int ns) {
  T J2  = ns / 2.0;
  int q = (int)(2 + 3.0 * J2);
  int nf;
  T *phaset = phase + threadIdx.y * MAX_NQUAD;
  T *ft     = f + threadIdx.y * MAX_NQUAD;
  T *oarr;
  // standard parallelism pattern in cuda. using a 2D grid, this allows to leverage more
  // threads as the parallelism is x*y*z
  // each thread check the y index to determine which array to use
  if (threadIdx.y == 0) {
    oarr = fwkerhalf1;
    nf   = nf1;
  } else if (threadIdx.y == 1) {
    oarr = fwkerhalf2;
    nf   = nf2;
  } else {
    oarr = fwkerhalf3;
    nf   = nf3;
  }

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf / 2 + 1;
       i += blockDim.x * gridDim.x) {
    T x = 0.0;
    for (int n = 0; n < q; n++) {
      // in type 1/2 2*PI/nf -> k[i]
      x += ft[n] * T(2) * std::cos(T(i) * phaset[n]);
    }
    oarr[i] = x * T(i % 2 ? -1 : 1); // signflip for the kernel origin being at PI
  }
}

/** Kernel for computing approximations of exact Fourier series coeffs of
 *  cnufftspread's real symmetric kernel.
 * a , f are intermediate results from function onedim_fseries_kernel_precomp().
 * this is the arbitrary frequency case (hence the extra kx, ky, kx arguments), used by
 * type 3, matching onedim_nuft_kernel in CPU code. Used by functions below in this file.
 */
template<typename T>
__global__ void cu_nuft_kernel_compute(int nf1, int nf2, int nf3, T *f, T *z, T *kx,
                                       T *ky, T *kz, T *fwkerhalf1, T *fwkerhalf2,
                                       T *fwkerhalf3, int ns) {
  T J2  = ns / 2.0;
  int q = (int)(2 + 2.0 * J2);
  int nf;
  T *at = z + threadIdx.y * MAX_NQUAD;
  T *ft = f + threadIdx.y * MAX_NQUAD;
  T *oarr, *k;
  // standard parallelism pattern in cuda. using a 2D grid, this allows to leverage more
  // threads as the parallelism is x*y*z
  // each thread check the y index to determine which array to use
  if (threadIdx.y == 0) {
    k    = kx;
    oarr = fwkerhalf1;
    nf   = nf1;
  } else if (threadIdx.y == 1) {
    k    = ky;
    oarr = fwkerhalf2;
    nf   = nf2;
  } else {
    k    = kz;
    oarr = fwkerhalf3;
    nf   = nf3;
  }
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf;
       i += blockDim.x * gridDim.x) {
    T x = 0.0;
    for (int n = 0; n < q; n++) {
      x += ft[n] * T(2) * std::cos(k[i] * at[n]);
    }
    oarr[i] = x;
  }
}

template<typename T>
int fseries_kernel_compute(int dim, int nf1, int nf2, int nf3, T *d_f, T *d_phase,
                           T *d_fwkerhalf1, T *d_fwkerhalf2, T *d_fwkerhalf3, int ns,
                           hipStream_t stream)
/*
    wrapper for approximation of Fourier series of real symmetric spreading
    kernel.

    Melody Shih 2/20/22
*/
{
  int nout = max(max(nf1 / 2 + 1, nf2 / 2 + 1), nf3 / 2 + 1);

  dim3 threadsPerBlock(16, dim);
  dim3 numBlocks((nout + 16 - 1) / 16, 1);

  cu_fseries_kernel_compute<<<numBlocks, threadsPerBlock, 0, stream>>>(
      nf1, nf2, nf3, d_f, d_phase, d_fwkerhalf1, d_fwkerhalf2, d_fwkerhalf3, ns);
  RETURN_IF_CUDA_ERROR

  return 0;
}

template<typename T>
int nuft_kernel_compute(int dim, int nf1, int nf2, int nf3, T *d_f, T *d_z, T *d_kx,
                        T *d_ky, T *d_kz, T *d_fwkerhalf1, T *d_fwkerhalf2,
                        T *d_fwkerhalf3, int ns, hipStream_t stream)
/*
    Approximates exact Fourier transform of cnufftspread's real symmetric
    kernel, directly via q-node quadrature on Euler-Fourier formula, exploiting
    narrowness of kernel. Evaluates at set of arbitrary freqs k in [-pi, pi),
    for a kernel with x measured in grid-spacings. (See previous routine for
    FT definition).
    It implements onedim_nuft_kernel in CPU code. Except it combines up to three
    onedimensional kernel evaluations at once (for efficiency).

    Marco Barbone 08/28/2024
*/
{
  int nout = max(max(nf1, nf2), nf3);

  dim3 threadsPerBlock(16, dim);
  dim3 numBlocks((nout + 16 - 1) / 16, 1);

  cu_nuft_kernel_compute<<<numBlocks, threadsPerBlock, 0, stream>>>(
      nf1, nf2, nf3, d_f, d_z, d_kx, d_ky, d_kz, d_fwkerhalf1, d_fwkerhalf2, d_fwkerhalf3,
      ns);
  RETURN_IF_CUDA_ERROR

  return 0;
}

template<typename T>
int setup_spreader_for_nufft(finufft_spread_opts &spopts, T eps, cufinufft_opts opts)
// Set up the spreader parameters given eps, and pass across various nufft
// options. Report status of setup_spreader. Just a wrapper following the CPU code.
{
  int ier = setup_spreader(spopts, eps, (T)opts.upsampfac, opts.gpu_kerevalmeth,
                           opts.debug, opts.gpu_spreadinterponly);
  return ier;
}

void set_nf_type12(CUFINUFFT_BIGINT ms, cufinufft_opts opts, finufft_spread_opts spopts,
                   CUFINUFFT_BIGINT *nf, CUFINUFFT_BIGINT bs)
// type 1 & 2 recipe for how to set 1d size of upsampled array, nf, given opts
// and requested number of Fourier modes ms.
{
  // round up to handle small cases
  *nf = static_cast<CUFINUFFT_BIGINT>(std::ceil(opts.upsampfac * ms));
  if (*nf < 2 * spopts.nspread) *nf = 2 * spopts.nspread; // otherwise spread fails
  if (*nf < MAX_NF) {                                     // otherwise will fail anyway
    *nf = utils::next235beven(*nf, opts.gpu_method == 4 ? bs : 1); // expensive at huge nf
  }
}

/*
  Precomputation of approximations of exact Fourier series coeffs of cnufftspread's
  real symmetric kernel.

  Inputs:
  nf - size of 1d uniform spread grid, must be even.
  opts - spreading opts object, needed to eval kernel (must be already set up)
  phase_winding - if true (type 1-2), scaling for the equispaced case else (type 3)
                  scaling for the general kx,ky,kz case

  Outputs:
  a - vector of phases to be used for cosines on the GPU;
  f - function values at quadrature nodes multiplied with quadrature weights (a, f are
      provided as the inputs of onedim_fseries_kernel_compute() defined below)
*/

template<typename T>
void onedim_fseries_kernel_precomp(CUFINUFFT_BIGINT nf, T *f, T *phase,
                                   finufft_spread_opts opts) {
  T J2 = opts.nspread / 2.0; // J/2, half-width of ker z-support
  // # quadr nodes in z (from 0 to J/2; reflections will be added)...
  const auto q = (int)(2 + 3.0 * J2); // matches CPU code
  double z[2 * MAX_NQUAD];
  double w[2 * MAX_NQUAD];
  cufinufft::utils::gaussquad(2 * q, z, w); // only half the nodes used, for (0,1)
  for (int n = 0; n < q; ++n) {             // set up nodes z_n and vals f_n
    z[n] *= J2;                             // rescale nodes
    f[n]     = J2 * w[n] * evaluate_kernel((T)z[n], opts); // vals & quadr wei
    phase[n] = T(2.0 * M_PI * z[n] / T(nf));               // phase winding rates
  }
}

template<typename T>
void onedim_nuft_kernel_precomp(T *f, T *z, finufft_spread_opts opts) {
  // it implements the first half of onedim_nuft_kernel in CPU code
  T J2 = opts.nspread / 2.0; // J/2, half-width of ker z-support
  // # quadr nodes in z (from 0 to J/2; reflections will be added)...
  int q = (int)(2 + 2.0 * J2); // matches CPU code
  double z_local[2 * MAX_NQUAD];
  double w_local[2 * MAX_NQUAD];
  cufinufft::utils::gaussquad(2 * q, z_local, w_local);   // half the nodes, (0,1)
  for (int n = 0; n < q; ++n) {                           // set up nodes z_n and vals f_n
    z[n] = J2 * T(z_local[n]);                            // rescale nodes
    f[n] = J2 * w_local[n] * evaluate_kernel(z[n], opts); // vals & quadr wei
  }
}

template<typename T>
std::size_t shared_memory_required(int dim, int ns, int bin_size_x, int bin_size_y,
                                   int bin_size_z) {
  // Helper to compute the shared memory required for the spreader when using SM
  int adjusted_ns = bin_size_x + ((ns + 1) / 2) * 2;

  if (dim == 1) {
    return adjusted_ns * sizeof(cuda_complex<T>);
  }

  adjusted_ns *= (bin_size_y + ((ns + 1) / 2) * 2);

  if (dim == 2) {
    return adjusted_ns * sizeof(cuda_complex<T>);
  }

  adjusted_ns *= (bin_size_z + ((ns + 1) / 2) * 2);

  return adjusted_ns * sizeof(cuda_complex<T>);
}

// Function to find bin_size_x == bin_size_y
// where bin_size_x * bin_size_y * bin_size_z < mem_size
// TODO: this can be done without a loop by using a direct formula
template<typename T> int find_bin_size(std::size_t mem_size, int dim, int ns) {
  int binsize = 1; // Start with the smallest possible bin size
  while (true) {
    // Calculate the shared memory required for the current bin_size_x and bin_size_y
    std::size_t required_memory =
        shared_memory_required<T>(dim, ns, binsize, binsize, binsize);

    // Check if the required memory is less than the available memory
    if (required_memory > mem_size) {
      // If the condition is met, return the current bin_size_x
      return binsize - 1;
    }

    // Increment bin_size_x for the next iteration
    binsize++;
  }
}

template<typename T>
void cufinufft_setup_binsize(int type, int ns, int dim, cufinufft_opts *opts) {
  // Marco Barbone 07/26/24. Using the shared memory available on the device, to
  // determine the optimal binsize for the spreader.
  // WARNING: This function does not check for CUDA errors, the caller should check and
  // handle them.
  // TODO: This can still be improved some sizes are hardcoded still
  int shared_mem_per_block{}, device_id{};
  switch (dim) {
  case 1: {
    if (opts->gpu_binsizex == 0) {
      hipGetDevice(&device_id);
      hipDeviceGetAttribute(&shared_mem_per_block,
                             hipDeviceAttributeSharedMemPerBlockOptin, device_id);
      // CUDA error handled by the caller not checking them here.
      // use 1/6 of the shared memory for the binsize
      // From experiments on multiple GPUs this gives the best tradeoff.
      // It is within 90% of the maximum performance for all GPUs tested.
      shared_mem_per_block /= 6;
      const int bin_size =
          shared_mem_per_block / sizeof(cuda_complex<T>) - ((ns + 1) / 2) * 2;
      opts->gpu_binsizex = bin_size;
    }
    opts->gpu_binsizey = 1;
    opts->gpu_binsizez = 1;
  } break;
  case 2: {
    if (opts->gpu_binsizex == 0 || opts->gpu_binsizey == 0) {
      switch (opts->gpu_method) {
      case 0:
      case 2: {
        opts->gpu_binsizex = 32;
        opts->gpu_binsizey = 32;
      } break;
      case 1: {
        hipGetDevice(&device_id);
        hipDeviceGetAttribute(&shared_mem_per_block,
                               hipDeviceAttributeSharedMemPerBlockOptin, device_id);
        const auto binsize = find_bin_size<T>(shared_mem_per_block, dim, ns);
        // in 2D 1/6 is too small, it gets slower because of the excessive padding
        opts->gpu_binsizex = binsize;
        opts->gpu_binsizey = binsize;
      } break;
      }
    }
    opts->gpu_binsizez = 1;
  } break;
  case 3: {
    switch (opts->gpu_method) {
    case 0:
    case 1:
    case 2: {
      if (opts->gpu_binsizex == 0 || opts->gpu_binsizey == 0 || opts->gpu_binsizez == 0) {
        opts->gpu_binsizex = 16;
        opts->gpu_binsizey = 16;
        opts->gpu_binsizez = 2;
      }
    } break;
    case 4: {
      opts->gpu_obinsizex = (opts->gpu_obinsizex == 0) ? 8 : opts->gpu_obinsizex;
      opts->gpu_obinsizey = (opts->gpu_obinsizey == 0) ? 8 : opts->gpu_obinsizey;
      opts->gpu_obinsizez = (opts->gpu_obinsizez == 0) ? 8 : opts->gpu_obinsizez;
      opts->gpu_binsizex  = (opts->gpu_binsizex == 0) ? 4 : opts->gpu_binsizex;
      opts->gpu_binsizey  = (opts->gpu_binsizey == 0) ? 4 : opts->gpu_binsizey;
      opts->gpu_binsizez  = (opts->gpu_binsizez == 0) ? 4 : opts->gpu_binsizez;
    } break;
    }
  } break;
  }
}

template int setup_spreader_for_nufft(finufft_spread_opts &spopts, float eps,
                                      cufinufft_opts opts);
template int setup_spreader_for_nufft(finufft_spread_opts &spopts, double eps,
                                      cufinufft_opts opts);
template void onedim_fseries_kernel_precomp<float>(CUFINUFFT_BIGINT nf, float *f,
                                                   float *a, finufft_spread_opts opts);
template void onedim_fseries_kernel_precomp<double>(CUFINUFFT_BIGINT nf, double *f,
                                                    double *a, finufft_spread_opts opts);
template void onedim_nuft_kernel_precomp<float>(float *f, float *a,
                                                finufft_spread_opts opts);
template void onedim_nuft_kernel_precomp<double>(double *f, double *a,
                                                 finufft_spread_opts opts);
template int fseries_kernel_compute(int dim, int nf1, int nf2, int nf3, float *d_f,
                                    float *d_a, float *d_fwkerhalf1, float *d_fwkerhalf2,
                                    float *d_fwkerhalf3, int ns, hipStream_t stream);
template int fseries_kernel_compute(
    int dim, int nf1, int nf2, int nf3, double *d_f, double *d_a, double *d_fwkerhalf1,
    double *d_fwkerhalf2, double *d_fwkerhalf3, int ns, hipStream_t stream);
template int nuft_kernel_compute<float>(int dim, int nf1, int nf2, int nf3, float *d_f,
                                        float *d_a, float *d_kx, float *d_ky, float *d_kz,
                                        float *d_fwkerhalf1, float *d_fwkerhalf2,
                                        float *d_fwkerhalf3, int ns, hipStream_t stream);
template int nuft_kernel_compute<double>(
    int dim, int nf1, int nf2, int nf3, double *d_f, double *d_a, double *d_kx,
    double *d_ky, double *d_kz, double *d_fwkerhalf1, double *d_fwkerhalf2,
    double *d_fwkerhalf3, int ns, hipStream_t stream);

template std::size_t shared_memory_required<float>(int dim, int ns, int bin_size_x,
                                                   int bin_size_y, int bin_size_z);
template std::size_t shared_memory_required<double>(int dim, int ns, int bin_size_x,
                                                    int bin_size_y, int bin_size_z);

template void cufinufft_setup_binsize<float>(int type, int ns, int dim,
                                             cufinufft_opts *opts);
template void cufinufft_setup_binsize<double>(int type, int ns, int dim,
                                              cufinufft_opts *opts);
} // namespace common
} // namespace cufinufft
