#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include <cufinufft/common.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <cufinufft/defs.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/utils.h>

namespace cufinufft {
namespace common {
using namespace cufinufft::spreadinterp;
using namespace finufft::common;
using std::max;

/** Kernel for computing approximations of exact Fourier series coeffs of
 *  cnufftspread's real symmetric kernel.
 * phase, f are intermediate results from function onedim_fseries_kernel_precomp().
 * this is the equispaced frequency case, used by type 1 & 2, matching
 * onedim_fseries_kernel in CPU code. Used by functions below in this file.
 */
template<typename T>
__global__ void cu_fseries_kernel_compute(int nf1, int nf2, int nf3, T *f, T *phase,
                                          T *fwkerhalf1, T *fwkerhalf2, T *fwkerhalf3,
                                          int ns) {
  T J2  = ns / 2.0;
  int q = (int)(2 + 3.0 * J2);
  int nf;
  T *phaset = phase + threadIdx.y * MAX_NQUAD;
  T *ft     = f + threadIdx.y * MAX_NQUAD;
  T *oarr;
  // standard parallelism pattern in cuda. using a 2D grid, this allows to leverage more
  // threads as the parallelism is x*y*z
  // each thread check the y index to determine which array to use
  if (threadIdx.y == 0) {
    oarr = fwkerhalf1;
    nf   = nf1;
  } else if (threadIdx.y == 1) {
    oarr = fwkerhalf2;
    nf   = nf2;
  } else {
    oarr = fwkerhalf3;
    nf   = nf3;
  }

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf / 2 + 1;
       i += blockDim.x * gridDim.x) {
    T x = 0.0;
    for (int n = 0; n < q; n++) {
      // in type 1/2 2*PI/nf -> k[i]
      x += ft[n] * T(2) * std::cos(T(i) * phaset[n]);
    }
    oarr[i] = x * T(i % 2 ? -1 : 1); // signflip for the kernel origin being at PI
  }
}

/** Kernel for computing approximations of exact Fourier series coeffs of
 *  cnufftspread's real symmetric kernel.
 * a , f are intermediate results from function onedim_fseries_kernel_precomp().
 * this is the arbitrary frequency case (hence the extra kx, ky, kx arguments), used by
 * type 3, matching onedim_nuft_kernel in CPU code. Used by functions below in this file.
 */
template<typename T>
__global__ void cu_nuft_kernel_compute(int nf1, int nf2, int nf3, T *f, T *z, T *kx,
                                       T *ky, T *kz, T *fwkerhalf1, T *fwkerhalf2,
                                       T *fwkerhalf3, int ns) {
  T J2  = ns / 2.0;
  int q = (int)(2 + 2.0 * J2);
  int nf;
  T *at = z + threadIdx.y * MAX_NQUAD;
  T *ft = f + threadIdx.y * MAX_NQUAD;
  T *oarr, *k;
  // standard parallelism pattern in cuda. using a 2D grid, this allows to leverage more
  // threads as the parallelism is x*y*z
  // each thread check the y index to determine which array to use
  if (threadIdx.y == 0) {
    k    = kx;
    oarr = fwkerhalf1;
    nf   = nf1;
  } else if (threadIdx.y == 1) {
    k    = ky;
    oarr = fwkerhalf2;
    nf   = nf2;
  } else {
    k    = kz;
    oarr = fwkerhalf3;
    nf   = nf3;
  }
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf;
       i += blockDim.x * gridDim.x) {
    T x = 0.0;
    for (int n = 0; n < q; n++) {
      x += ft[n] * T(2) * std::cos(k[i] * at[n]);
    }
    oarr[i] = x;
  }
}

template<typename T>
int fseries_kernel_compute(int dim, int nf1, int nf2, int nf3, T *d_f, T *d_phase,
                           T *d_fwkerhalf1, T *d_fwkerhalf2, T *d_fwkerhalf3, int ns,
                           hipStream_t stream)
/*
    wrapper for approximation of Fourier series of real symmetric spreading
    kernel.

    Melody Shih 2/20/22
*/
{
  int nout = max(max(nf1 / 2 + 1, nf2 / 2 + 1), nf3 / 2 + 1);

  dim3 threadsPerBlock(16, dim);
  dim3 numBlocks((nout + 16 - 1) / 16, 1);

  cu_fseries_kernel_compute<<<numBlocks, threadsPerBlock, 0, stream>>>(
      nf1, nf2, nf3, d_f, d_phase, d_fwkerhalf1, d_fwkerhalf2, d_fwkerhalf3, ns);
  RETURN_IF_CUDA_ERROR

  return 0;
}

template<typename T>
int nuft_kernel_compute(int dim, int nf1, int nf2, int nf3, T *d_f, T *d_z, T *d_kx,
                        T *d_ky, T *d_kz, T *d_fwkerhalf1, T *d_fwkerhalf2,
                        T *d_fwkerhalf3, int ns, hipStream_t stream)
/*
    Approximates exact Fourier transform of cnufftspread's real symmetric
    kernel, directly via q-node quadrature on Euler-Fourier formula, exploiting
    narrowness of kernel. Evaluates at set of arbitrary freqs k in [-pi, pi),
    for a kernel with x measured in grid-spacings. (See previous routine for
    FT definition).
    It implements onedim_nuft_kernel in CPU code. Except it combines up to three
    onedimensional kernel evaluations at once (for efficiency).

    Marco Barbone 08/28/2024
*/
{
  int nout = max(max(nf1, nf2), nf3);

  dim3 threadsPerBlock(16, dim);
  dim3 numBlocks((nout + 16 - 1) / 16, 1);

  cu_nuft_kernel_compute<<<numBlocks, threadsPerBlock, 0, stream>>>(
      nf1, nf2, nf3, d_f, d_z, d_kx, d_ky, d_kz, d_fwkerhalf1, d_fwkerhalf2, d_fwkerhalf3,
      ns);
  RETURN_IF_CUDA_ERROR

  return 0;
}

template<typename T>
int setup_spreader_for_nufft(finufft_spread_opts &spopts, T eps, cufinufft_opts opts)
// Set up the spreader parameters given eps, and pass across various nufft
// options. Report status of setup_spreader. Just a wrapper following the CPU code.
{
  int ier = setup_spreader(spopts, eps, (T)opts.upsampfac, opts.gpu_kerevalmeth,
                           opts.debug, opts.gpu_spreadinterponly);
  return ier;
}

void set_nf_type12(CUFINUFFT_BIGINT ms, cufinufft_opts opts, finufft_spread_opts spopts,
                   CUFINUFFT_BIGINT *nf, CUFINUFFT_BIGINT bs)
// type 1 & 2 recipe for how to set 1d size of upsampled array, nf, given opts
// and requested number of Fourier modes ms.
{
  // round up to handle small cases
  *nf = static_cast<CUFINUFFT_BIGINT>(std::ceil(opts.upsampfac * ms));
  if (*nf < 2 * spopts.nspread) *nf = 2 * spopts.nspread; // otherwise spread fails
  if (*nf < MAX_NF) {                                     // otherwise will fail anyway
    *nf = utils::next235beven(*nf, opts.gpu_method == 4 ? bs : 1); // expensive at huge nf
  }
}

/*
  Precomputation of approximations of exact Fourier series coeffs of cnufftspread's
  real symmetric kernel.

  Inputs:
  nf - size of 1d uniform spread grid, must be even.
  opts - spreading opts object, needed to eval kernel (must be already set up)
  phase_winding - if true (type 1-2), scaling for the equispaced case else (type 3)
                  scaling for the general kx,ky,kz case

  Outputs:
  a - vector of phases to be used for cosines on the GPU;
  f - function values at quadrature nodes multiplied with quadrature weights (a, f are
      provided as the inputs of onedim_fseries_kernel_compute() defined below)
*/

template<typename T>
void onedim_fseries_kernel_precomp(CUFINUFFT_BIGINT nf, T *f, T *phase,
                                   finufft_spread_opts opts) {
  T J2 = opts.nspread / 2.0; // J/2, half-width of ker z-support
  // # quadr nodes in z (from 0 to J/2; reflections will be added)...
  const auto q = (int)(2 + 3.0 * J2); // matches CPU code
  double z[2 * MAX_NQUAD];
  double w[2 * MAX_NQUAD];
  gaussquad(2 * q, z, w);       // only half the nodes used, for (0,1)
  for (int n = 0; n < q; ++n) { // set up nodes z_n and vals f_n
    z[n] *= J2;                 // rescale nodes
    f[n]     = J2 * w[n] * evaluate_kernel((T)z[n], opts); // vals & quadr wei
    phase[n] = T(2.0 * PI * z[n] / T(nf));                 // phase winding rates
  }
}

template<typename T>
void onedim_nuft_kernel_precomp(T *f, T *z, finufft_spread_opts opts) {
  // it implements the first half of onedim_nuft_kernel in CPU code
  T J2 = opts.nspread / 2.0; // J/2, half-width of ker z-support
  // # quadr nodes in z (from 0 to J/2; reflections will be added)...
  int q = (int)(2 + 2.0 * J2); // matches CPU code
  double z_local[2 * MAX_NQUAD];
  double w_local[2 * MAX_NQUAD];
  gaussquad(2 * q, z_local, w_local);                     // half the nodes, (0,1)
  for (int n = 0; n < q; ++n) {                           // set up nodes z_n and vals f_n
    z[n] = J2 * T(z_local[n]);                            // rescale nodes
    f[n] = J2 * w_local[n] * evaluate_kernel(z[n], opts); // vals & quadr wei
  }
}

template<typename T> std::size_t shared_memory_per_point(int dim, int ns) {
  return ns * sizeof(T) * dim       // kernel evaluations
         + sizeof(int) * dim        // indexes
         + sizeof(cuda_complex<T>); // strength
}

// Marco: 4/18/25 not 100% happy of having np here, but the alternatives seem worse to me
template<typename T>
std::size_t shared_memory_required(int dim, int ns, int bin_size_x, int bin_size_y,
                                   int bin_size_z, int np) {
  const auto shmem_per_point = shared_memory_per_point<T>(dim, ns);
  const int ns_2             = (ns + 1) / 2;
  std::size_t grid_size      = bin_size_x + 2 * ns_2;
  if (dim > 1) grid_size *= bin_size_y + 2 * ns_2;
  if (dim > 2) grid_size *= bin_size_z + 2 * ns_2;
  return grid_size * sizeof(cuda_complex<T>) + shmem_per_point * np;
}

// Function to find bin_size_x == bin_size_y
// where bin_size_x * bin_size_y * bin_size_z < mem_size
template<typename T> int find_bin_size(std::size_t mem_size, int dim, int ns) {
  const auto elements        = mem_size / sizeof(cuda_complex<T>);
  const auto padded_bin_size = int(std::floor(std::pow(elements, 1.0 / dim)));
  const auto bin_size        = padded_bin_size - (2 * (ns + 1) / 2);
  // TODO: over one dimension we could increase this a bit
  //       maybe the shape should not be uniform
  return bin_size;
}
template<typename T>
void cufinufft_setup_binsize(int type, int ns, int dim, cufinufft_opts *opts) {
  int shared_mem_per_block{}, device_id{opts->gpu_device_id};
  hipDeviceGetAttribute(&shared_mem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock,
                         device_id);

  auto try_find_binsize = [&](int shmem) -> int {
    return find_bin_size<T>(shmem, dim, ns);
  };

  auto set_binsizes_if_unset = [&](int binsize) {
    if (binsize <= 1) return;

    opts->gpu_binsizex = opts->gpu_binsizex == 0 ? binsize : opts->gpu_binsizex;
    opts->gpu_binsizey =
        (dim < 2) ? 1 : (opts->gpu_binsizey == 0 ? binsize : opts->gpu_binsizey);
    opts->gpu_binsizez =
        (dim < 3) ? 1 : (opts->gpu_binsizez == 0 ? binsize : opts->gpu_binsizez);
  };

  auto ensure_optin_shmem = [&]() {
    hipDeviceGetAttribute(&shared_mem_per_block, hipDeviceAttributeSharedMemPerBlockOptin,
                           device_id);
  };

  switch (opts->gpu_method) {
  case 1: {
    ensure_optin_shmem();
    int binsize = try_find_binsize(shared_mem_per_block);
    set_binsizes_if_unset(binsize);
    break;
  }
  case 2: {
    int binsize = try_find_binsize(shared_mem_per_block);
    if (binsize < 1) {
      ensure_optin_shmem();
      binsize = try_find_binsize(shared_mem_per_block);
    }
    set_binsizes_if_unset(binsize);
    break;
  }
  case 3: {
    const int shmem_per_point = shared_memory_per_point<T>(dim, ns);
    const int min_np_shmem    = shmem_per_point * opts->gpu_np;
    int binsize               = try_find_binsize(shared_mem_per_block - min_np_shmem);

    if (binsize < 1) {
      ensure_optin_shmem();
      binsize = try_find_binsize(shared_mem_per_block - min_np_shmem);
      if (binsize < 1) {
        throw std::runtime_error(
            "[cufinufft] ERROR: Not enough shared memory for the number of points.");
      }
    }

    set_binsizes_if_unset(binsize);

    const int shmem_required = shared_memory_required<T>(
        dim, ns, opts->gpu_binsizex, opts->gpu_binsizey, opts->gpu_binsizez, 0);
    const int shmem_left = shared_mem_per_block - shmem_required;
    const int max_np     = (shmem_left / shmem_per_point) & static_cast<unsigned>(-16);

    if (opts->debug) {
      const int required_shmem = shared_memory_required<T>(
          dim, ns, opts->gpu_binsizex, opts->gpu_binsizey, opts->gpu_binsizez, max_np);
      printf("[cufinufft] Shared memory required: %d bytes (limit: %d bytes)\n",
             required_shmem, shared_mem_per_block);
      printf("[cufinufft]   min_np_shmem     = %d\n", min_np_shmem);
      printf("[cufinufft]   shmem_per_point  = %d\n", shmem_per_point);
      printf("[cufinufft]   shmem_required   = %d\n", shmem_required);
      printf("[cufinufft]   shmem_left       = %d\n", shmem_left);
      printf("[cufinufft]   min_np           = %d\n", opts->gpu_np);
      printf("[cufinufft]   max_np           = %d\n", max_np);
      printf("[cufinufft]   found bin size   = %d\n", binsize);
      printf("[cufinufft]   binsize         = %d\n", opts->gpu_binsizex);
      assert(required_shmem < shared_mem_per_block);
    }

    opts->gpu_np = max_np;
    break;
  }
  case 4: {
    opts->gpu_obinsizex = opts->gpu_obinsizex == 0 ? 8 : opts->gpu_obinsizex;
    opts->gpu_obinsizey = opts->gpu_obinsizey == 0 ? 8 : opts->gpu_obinsizey;
    opts->gpu_obinsizez = opts->gpu_obinsizez == 0 ? 8 : opts->gpu_obinsizez;
    opts->gpu_binsizex  = opts->gpu_binsizex == 0 ? 4 : opts->gpu_binsizex;
    opts->gpu_binsizey  = opts->gpu_binsizey == 0 ? 4 : opts->gpu_binsizey;
    opts->gpu_binsizez  = opts->gpu_binsizez == 0 ? 4 : opts->gpu_binsizez;
    break;
  }
  }
}

template int setup_spreader_for_nufft(finufft_spread_opts &spopts, float eps,
                                      cufinufft_opts opts);
template int setup_spreader_for_nufft(finufft_spread_opts &spopts, double eps,
                                      cufinufft_opts opts);
template void onedim_fseries_kernel_precomp<float>(CUFINUFFT_BIGINT nf, float *f,
                                                   float *a, finufft_spread_opts opts);
template void onedim_fseries_kernel_precomp<double>(CUFINUFFT_BIGINT nf, double *f,
                                                    double *a, finufft_spread_opts opts);
template void onedim_nuft_kernel_precomp<float>(float *f, float *a,
                                                finufft_spread_opts opts);
template void onedim_nuft_kernel_precomp<double>(double *f, double *a,
                                                 finufft_spread_opts opts);
template int fseries_kernel_compute(int dim, int nf1, int nf2, int nf3, float *d_f,
                                    float *d_a, float *d_fwkerhalf1, float *d_fwkerhalf2,
                                    float *d_fwkerhalf3, int ns, hipStream_t stream);
template int fseries_kernel_compute(
    int dim, int nf1, int nf2, int nf3, double *d_f, double *d_a, double *d_fwkerhalf1,
    double *d_fwkerhalf2, double *d_fwkerhalf3, int ns, hipStream_t stream);
template int nuft_kernel_compute<float>(int dim, int nf1, int nf2, int nf3, float *d_f,
                                        float *d_a, float *d_kx, float *d_ky, float *d_kz,
                                        float *d_fwkerhalf1, float *d_fwkerhalf2,
                                        float *d_fwkerhalf3, int ns, hipStream_t stream);
template int nuft_kernel_compute<double>(
    int dim, int nf1, int nf2, int nf3, double *d_f, double *d_a, double *d_kx,
    double *d_ky, double *d_kz, double *d_fwkerhalf1, double *d_fwkerhalf2,
    double *d_fwkerhalf3, int ns, hipStream_t stream);

template std::size_t shared_memory_required<float>(
    int dim, int ns, int bin_size_x, int bin_size_y, int bin_size_z, int np);
template std::size_t shared_memory_required<double>(
    int dim, int ns, int bin_size_x, int bin_size_y, int bin_size_z, int np);
template std::size_t shared_memory_per_point<float>(int dim, int ns);
template std::size_t shared_memory_per_point<double>(int dim, int ns);
template void cufinufft_setup_binsize<float>(int type, int ns, int dim,
                                             cufinufft_opts *opts);
template void cufinufft_setup_binsize<double>(int type, int ns, int dim,
                                              cufinufft_opts *opts);
} // namespace common
} // namespace cufinufft
