#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <vector>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include <cufinufft/common.h>
#include <cufinufft/defs.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/utils.h>

#include <legendre_rule_fast.h>

namespace cufinufft {
namespace common {
using namespace cufinufft::spreadinterp;
using std::max;

/* Kernel for computing approximations of exact Fourier series coeffs of
   cnufftspread's real symmetric kernel. */
// a , f are intermediate results from function onedim_fseries_kernel_precomp()
// (see cufinufft/contrib/common.cpp for description)
__global__ void fseries_kernel_compute(int nf1, int nf2, int nf3, CUFINUFFT_FLT *f, hipDoubleComplex *a,
                                       CUFINUFFT_FLT *fwkerhalf1, CUFINUFFT_FLT *fwkerhalf2, CUFINUFFT_FLT *fwkerhalf3,
                                       int ns) {
    CUFINUFFT_FLT J2 = ns / 2.0;
    int q = (int)(2 + 3.0 * J2);
    int nf;
    hipDoubleComplex *at = a + threadIdx.y * MAX_NQUAD;
    CUFINUFFT_FLT *ft = f + threadIdx.y * MAX_NQUAD;
    CUFINUFFT_FLT *oarr;
    if (threadIdx.y == 0) {
        oarr = fwkerhalf1;
        nf = nf1;
    } else if (threadIdx.y == 1) {
        oarr = fwkerhalf2;
        nf = nf2;
    } else {
        oarr = fwkerhalf3;
        nf = nf3;
    }

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf / 2 + 1; i += blockDim.x * gridDim.x) {
        int brk = 0.5 + i;
        CUFINUFFT_FLT x = 0.0;
        for (int n = 0; n < q; n++) {
            x += ft[n] * 2 * (pow(cabs(at[n]), brk) * cos(brk * carg(at[n])));
        }
        oarr[i] = x;
    }
}

int CUFSERIESKERNELCOMPUTE(int dim, int nf1, int nf2, int nf3, CUFINUFFT_FLT *d_f, hipDoubleComplex *d_a,
                           CUFINUFFT_FLT *d_fwkerhalf1, CUFINUFFT_FLT *d_fwkerhalf2, CUFINUFFT_FLT *d_fwkerhalf3,
                           int ns)
/*
    wrapper for approximation of Fourier series of real symmetric spreading
    kernel.

    Melody Shih 2/20/22
*/
{
    int nout = max(max(nf1 / 2 + 1, nf2 / 2 + 1), nf3 / 2 + 1);

    dim3 threadsPerBlock(16, dim);
    dim3 numBlocks((nout + 16 - 1) / 16, 1);

    fseries_kernel_compute<<<numBlocks, threadsPerBlock>>>(nf1, nf2, nf3, d_f, d_a, d_fwkerhalf1, d_fwkerhalf2,
                                                           d_fwkerhalf3, ns);
    return 0;
}

int setup_spreader_for_nufft(finufft_spread_opts &spopts, CUFINUFFT_FLT eps, cufinufft_opts opts)
// Set up the spreader parameters given eps, and pass across various nufft
// options. Report status of setup_spreader.  Barnett 10/30/17
{
    int ier = setup_spreader(spopts, eps, opts.upsampfac, opts.gpu_kerevalmeth);
    spopts.pirange = 1; // could allow user control?
    return ier;
}

void SET_NF_TYPE12(CUFINUFFT_BIGINT ms, cufinufft_opts opts, finufft_spread_opts spopts, CUFINUFFT_BIGINT *nf,
                   CUFINUFFT_BIGINT bs)
// type 1 & 2 recipe for how to set 1d size of upsampled array, nf, given opts
// and requested number of Fourier modes ms.
{
    *nf = (CUFINUFFT_BIGINT)(opts.upsampfac * ms);
    if (*nf < 2 * spopts.nspread)
        *nf = 2 * spopts.nspread; // otherwise spread fails
    if (*nf < MAX_NF) {           // otherwise will fail anyway
        if (opts.gpu_method == 4) // expensive at huge nf
            *nf = utils::next235beven(*nf, bs);
        else
            *nf = utils::next235beven(*nf, 1);
    }
}

void onedim_fseries_kernel(CUFINUFFT_BIGINT nf, CUFINUFFT_FLT *fwkerhalf, finufft_spread_opts opts)
/*
  Approximates exact Fourier series coeffs of cnufftspread's real symmetric
  kernel, directly via q-node quadrature on Euler-Fourier formula, exploiting
  narrowness of kernel. Uses phase winding for cheap eval on the regular freq
  grid. Note that this is also the Fourier transform of the non-periodized
  kernel. The FT definition is f(k) = int e^{-ikx} f(x) dx. The output has an
  overall prefactor of 1/h, which is needed anyway for the correction, and
  arises because the quadrature weights are scaled for grid units not x units.

  Inputs:
  nf - size of 1d uniform spread grid, must be even.
  opts - spreading opts object, needed to eval kernel (must be already set up)

  Outputs:
  fwkerhalf - real Fourier series coeffs from indices 0 to nf/2 inclusive,
              divided by h = 2pi/n.
              (should be allocated for at least nf/2+1 CUFINUFFT_FLTs)

  Compare onedim_dct_kernel which has same interface, but computes DFT of
  sampled kernel, not quite the same object.

  Barnett 2/7/17. openmp (since slow vs fftw in 1D large-N case) 3/3/18
  Melody 2/20/22 separate into precomp & comp functions defined below.
 */
{
    CUFINUFFT_FLT f[MAX_NQUAD];
    dcomplex a[MAX_NQUAD];
    onedim_fseries_kernel_precomp(nf, f, a, opts);
    onedim_fseries_kernel_compute(nf, f, a, fwkerhalf, opts);
}

/*
  Precomputation of approximations of exact Fourier series coeffs of cnufftspread's
  real symmetric kernel.

  Inputs:
  nf - size of 1d uniform spread grid, must be even.
  opts - spreading opts object, needed to eval kernel (must be already set up)

  Outputs:
  a - phase winding rates
  f - funciton values at quadrature nodes multiplied with quadrature weights
  (a, f are provided as the inputs of onedim_fseries_kernel_compute() defined below)
*/
void onedim_fseries_kernel_precomp(CUFINUFFT_BIGINT nf, CUFINUFFT_FLT *f, dcomplex *a, finufft_spread_opts opts) {
    CUFINUFFT_FLT J2 = opts.nspread / 2.0; // J/2, half-width of ker z-support
    // # quadr nodes in z (from 0 to J/2; reflections will be added)...
    int q = (int)(2 + 3.0 * J2); // not sure why so large? cannot exceed MAX_NQUAD
    double z[2 * MAX_NQUAD], w[2 * MAX_NQUAD];

    finufft::quadrature::legendre_compute_glr(2 * q, z, w); // only half the nodes used, eg on (0,1)
    for (int n = 0; n < q; ++n) {                           // set up nodes z_n and vals f_n
        z[n] *= J2;                                         // rescale nodes
        f[n] = J2 * (CUFINUFFT_FLT)w[n] * evaluate_kernel((CUFINUFFT_FLT)z[n], opts);  // vals & quadr wei
        a[n] = exp(2 * PI * IMA * (CUFINUFFT_FLT)(nf / 2 - z[n]) / (CUFINUFFT_FLT)nf); // phase winding rates
    }
}

void onedim_fseries_kernel_compute(CUFINUFFT_BIGINT nf, CUFINUFFT_FLT *f, dcomplex *a, CUFINUFFT_FLT *fwkerhalf,
                                   finufft_spread_opts opts) {
    CUFINUFFT_FLT J2 = opts.nspread / 2.0;        // J/2, half-width of ker z-support
    int q = (int)(2 + 3.0 * J2);                  // not sure why so large? cannot exceed MAX_NQUAD
    CUFINUFFT_BIGINT nout = nf / 2 + 1;           // how many values we're writing to
    int nt = MIN(nout, MY_OMP_GET_MAX_THREADS()); // how many chunks
    std::vector<CUFINUFFT_BIGINT> brk(nt + 1);    // start indices for each thread
    for (int t = 0; t <= nt; ++t)                 // split nout mode indices btw threads
        brk[t] = (CUFINUFFT_BIGINT)(0.5 + nout * t / (double)nt);
#pragma omp parallel
    {
        int t = MY_OMP_GET_THREAD_NUM();
        if (t < nt) {               // could be nt < actual # threads
            dcomplex aj[MAX_NQUAD]; // phase rotator for this thread
            for (int n = 0; n < q; ++n)
                aj[n] = pow(a[n], (CUFINUFFT_FLT)brk[t]);            // init phase factors for chunk
            for (CUFINUFFT_BIGINT j = brk[t]; j < brk[t + 1]; ++j) { // loop along output array
                CUFINUFFT_FLT x = 0.0;                               // accumulator for answer at this j
                for (int n = 0; n < q; ++n) {
                    x += f[n] * 2 * real(aj[n]); // include the negative freq
                    aj[n] *= a[n];               // wind the phases
                }
                fwkerhalf[j] = x;
            }
        }
    }
}

} // namespace common
} // namespace cufinufft
