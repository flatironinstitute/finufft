#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

#include "spreadinterp3d.cuh"

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cuspread3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem
    (4) Block gather

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int M = d_plan->M;

    int ier = 0;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuspread3d_nuptsdriven<T>(nf1, nf2, nf3, M, d_plan, blksize);
    } break;
    case 2: {
        ier = cuspread3d_subprob<T>(nf1, nf2, nf3, M, d_plan, blksize);
    } break;
    case 4: {
        ier = cuspread3d_blockgather<T>(nf1, nf2, nf3, M, d_plan, blksize);
    } break;
    default:
        std::cerr << "[cuspread3d] error: incorrect method, should be 1,2,4" << std::endl;
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

    return ier;
}

template <typename T>
int cuspread3d_nuptsdriven_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    auto &stream = d_plan->stream;

    if (d_plan->opts.gpu_sort) {
        int bin_size_x = d_plan->opts.gpu_binsizex;
        int bin_size_y = d_plan->opts.gpu_binsizey;
        int bin_size_z = d_plan->opts.gpu_binsizez;
        if (bin_size_x < 0 || bin_size_y < 0 || bin_size_z < 0) {
            std::cerr << "[cuspread3d_nuptsdriven_prop] error: invalid binsize (binsizex, binsizey, binsizez) = (";
            std::cerr << bin_size_x << "," << bin_size_y << "," << bin_size_z << ")" << std::endl;
            return FINUFFT_ERR_BINSIZE_NOTVALID;
        }

        int numbins[3];
        numbins[0] = ceil((T)nf1 / bin_size_x);
        numbins[1] = ceil((T)nf2 / bin_size_y);
        numbins[2] = ceil((T)nf3 / bin_size_z);

        T *d_kx = d_plan->kx;
        T *d_ky = d_plan->ky;
        T *d_kz = d_plan->kz;

        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;

        int ier;
        if ((ier = checkCudaErrors(
                 hipMemsetAsync(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int), stream))))
            return ier;
        calc_bin_size_noghost_3d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
            M, nf1, nf2, nf3, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binsize, d_kx,
            d_ky, d_kz, d_sortidx, pirange);
        RETURN_IF_CUDA_ERROR

        int n = numbins[0] * numbins[1] * numbins[2];
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

        calc_inverse_of_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
            M, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binstartpts, d_sortidx, d_kx,
            d_ky, d_kz, d_idxnupts, pirange, nf1, nf2, nf3);
        RETURN_IF_CUDA_ERROR
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        trivial_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(M, d_idxnupts);
        RETURN_IF_CUDA_ERROR
    }

    return 0;
}

template <typename T>
int cuspread3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;

    int *d_idxnupts = d_plan->idxnupts;
    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth == 1) {
        for (int t = 0; t < blksize; t++) {
            spread_3d_nupts_driven<T, 1>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M,
                                                         ns, nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            spread_3d_nupts_driven<T, 0>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M,
                                                         ns, nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template <typename T>
int cuspread3d_blockgather_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    auto &stream = d_plan->stream;

    dim3 threadsPerBlock;
    dim3 blocks;

    int pirange = d_plan->spopts.pirange;

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int o_bin_size_x = d_plan->opts.gpu_obinsizex;
    int o_bin_size_y = d_plan->opts.gpu_obinsizey;
    int o_bin_size_z = d_plan->opts.gpu_obinsizez;

    int numobins[3];
    if (nf1 % o_bin_size_x != 0 || nf2 % o_bin_size_y != 0 || nf3 % o_bin_size_z != 0) {
        std::cerr << "[cuspread3d_blockgather_prop] error:\n";
        std::cerr << "       mod(nf(1|2|3), opts.gpu_obinsize(x|y|z)) != 0" << std::endl;
        std::cerr << "       (nf1, nf2, nf3) = (" << nf1 << ", " << nf2 << ", " << nf3 << ")" << std::endl;
        std::cerr << "       (obinsizex, obinsizey, obinsizez) = (" << o_bin_size_x << ", " << o_bin_size_y << ", "
                  << o_bin_size_z << ")" << std::endl;
        return FINUFFT_ERR_BINSIZE_NOTVALID;
    }

    numobins[0] = ceil((T)nf1 / o_bin_size_x);
    numobins[1] = ceil((T)nf2 / o_bin_size_y);
    numobins[2] = ceil((T)nf3 / o_bin_size_z);

    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    if (o_bin_size_x % bin_size_x != 0 || o_bin_size_y % bin_size_y != 0 || o_bin_size_z % bin_size_z != 0) {
        std::cerr << "[cuspread3d_blockgather_prop] error:\n";
        std::cerr << "      mod(ops.gpu_obinsize(x|y|z), opts.gpu_binsize(x|y|z)) != 0" << std::endl;
        std::cerr << "      (binsizex, binsizey, binsizez) = (" << bin_size_x << ", " << bin_size_y << ", "
                  << bin_size_z << ")" << std::endl;
        std::cerr << "      (obinsizex, obinsizey, obinsizez) = (" << o_bin_size_x << ", " << o_bin_size_y << ", "
                  << o_bin_size_z << ")" << std::endl;
        return FINUFFT_ERR_BINSIZE_NOTVALID;
    }

    int binsperobinx, binsperobiny, binsperobinz;
    int numbins[3];
    binsperobinx = o_bin_size_x / bin_size_x + 2;
    binsperobiny = o_bin_size_y / bin_size_y + 2;
    binsperobinz = o_bin_size_z / bin_size_z + 2;
    numbins[0] = numobins[0] * (binsperobinx);
    numbins[1] = numobins[1] * (binsperobiny);
    numbins[2] = numobins[2] * (binsperobinz);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;

    int *d_binsize = d_plan->binsize;
    int *d_sortidx = d_plan->sortidx;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_idxnupts = NULL;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_subprob_to_bin = NULL;

    int ier;
    if ((ier = checkCudaErrors(
             hipMemsetAsync(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int), stream))))
        return ier;

    locate_nupts_to_bins_ghost<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numobins[0], numobins[1], numobins[2], binsperobinx, binsperobiny,
        binsperobinz, d_binsize, d_kx, d_ky, d_kz, d_sortidx, pirange, nf1, nf2, nf3);
    RETURN_IF_CUDA_ERROR

    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    threadsPerBlock.z = 8;

    blocks.x = (threadsPerBlock.x + numbins[0] - 1) / threadsPerBlock.x;
    blocks.y = (threadsPerBlock.y + numbins[1] - 1) / threadsPerBlock.y;
    blocks.z = (threadsPerBlock.z + numbins[2] - 1) / threadsPerBlock.z;

    fill_ghost_bins<<<blocks, threadsPerBlock, 0, stream>>>(binsperobinx, binsperobiny, binsperobinz, numobins[0],
                                                            numobins[1], numobins[2], d_binsize);
    RETURN_IF_CUDA_ERROR

    int n = numbins[0] * numbins[1] * numbins[2];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts + 1);
    thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

    if ((ier = checkCudaErrors(hipMemsetAsync(d_binstartpts, 0, sizeof(int), stream))))
        return ier;

    int totalNUpts;
    if ((ier = checkCudaErrors(
             hipMemcpyAsync(&totalNUpts, &d_binstartpts[n], sizeof(int), hipMemcpyDeviceToHost, stream))))
        return ier;
    hipStreamSynchronize(stream);
    if ((ier = checkCudaErrors(hipMallocAsync(&d_idxnupts, totalNUpts * sizeof(int), stream))))
        return ier;

    calc_inverse_of_global_sort_index_ghost<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numobins[0], numobins[1], numobins[2], binsperobinx, binsperobiny,
        binsperobinz, d_binstartpts, d_sortidx, d_kx, d_ky, d_kz, d_idxnupts, pirange, nf1, nf2, nf3);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[%s] Error: %s\n", __func__, hipGetErrorString(err));
        hipFree(d_idxnupts);
        return FINUFFT_ERR_CUDA_FAILURE;
    }

    threadsPerBlock.x = 2;
    threadsPerBlock.y = 2;
    threadsPerBlock.z = 2;

    blocks.x = (threadsPerBlock.x + numbins[0] - 1) / threadsPerBlock.x;
    blocks.y = (threadsPerBlock.y + numbins[1] - 1) / threadsPerBlock.y;
    blocks.z = (threadsPerBlock.z + numbins[2] - 1) / threadsPerBlock.z;

    ghost_bin_pts_index<<<blocks, threadsPerBlock, 0, stream>>>(binsperobinx, binsperobiny, binsperobinz, numobins[0],
                                                                numobins[1], numobins[2], d_binsize, d_idxnupts,
                                                                d_binstartpts, M);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[%s] Error: %s\n", __func__, hipGetErrorString(err));
        hipFree(d_idxnupts);
        return FINUFFT_ERR_CUDA_FAILURE;
    }

    hipFree(d_plan->idxnupts);
    d_plan->idxnupts = d_idxnupts;

    /* --------------------------------------------- */
    //        Determining Subproblem properties      //
    /* --------------------------------------------- */
    n = numobins[0] * numobins[1] * numobins[2];
    calc_subprob_3d_v1<<<(n + 1024 - 1) / 1024, 1024, 0, stream>>>(binsperobinx, binsperobiny, binsperobinz, d_binsize,
                                                                   d_numsubprob, maxsubprobsize,
                                                                   numobins[0] * numobins[1] * numobins[2]);
    RETURN_IF_CUDA_ERROR

    n = numobins[0] * numobins[1] * numobins[2];
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

    if ((ier = checkCudaErrors(hipMemsetAsync(d_subprobstartpts, 0, sizeof(int), stream))))
        return ier;

    int totalnumsubprob;
    if ((ier = checkCudaErrors(
             hipMemcpyAsync(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost, stream))))
        return ier;
    hipStreamSynchronize(stream);
    if ((ier = checkCudaErrors(hipMallocAsync(&d_subprob_to_bin, totalnumsubprob * sizeof(int), stream))))
        return ier;
    map_b_into_subprob_3d_v1<<<(n + 1024 - 1) / 1024, 1024, 0, stream>>>(d_subprob_to_bin, d_subprobstartpts,
                                                                         d_numsubprob, n);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[%s] Error: %s\n", __func__, hipGetErrorString(err));
        hipFree(d_subprob_to_bin);
        return FINUFFT_ERR_CUDA_FAILURE;
    }

    assert(d_subprob_to_bin != NULL);
    hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    d_plan->totalnumsubprob = totalnumsubprob;

    return 0;
}

template <typename T>
int cuspread3d_blockgather(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    int ns = d_plan->spopts.nspread;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    int obin_size_x = d_plan->opts.gpu_obinsizex;
    int obin_size_y = d_plan->opts.gpu_obinsizey;
    int obin_size_z = d_plan->opts.gpu_obinsizez;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numobins[3];
    numobins[0] = ceil((T)nf1 / obin_size_x);
    numobins[1] = ceil((T)nf2 / obin_size_y);
    numobins[2] = ceil((T)nf3 / obin_size_z);

    int binsperobinx, binsperobiny, binsperobinz;
    binsperobinx = obin_size_x / bin_size_x + 2;
    binsperobiny = obin_size_y / bin_size_y + 2;
    binsperobinz = obin_size_z / bin_size_z + 2;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binstartpts = d_plan->binstartpts;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    size_t sharedplanorysize = obin_size_x * obin_size_y * obin_size_z * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cerr << "[cuspread3d_blockgather] error: not enough shared memory" << std::endl;
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth == 1) {
            spread_3d_block_gather<T, 1><<<totalnumsubprob, 64, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, obin_size_x, obin_size_y, obin_size_z, binsperobinx * binsperobiny * binsperobinz,
                d_subprob_to_bin, d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1], numobins[2], d_idxnupts,
                pirange);
            RETURN_IF_CUDA_ERROR
        } else {
            spread_3d_block_gather<T, 0><<<totalnumsubprob, 64, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, obin_size_x, obin_size_y, obin_size_z, binsperobinx * binsperobiny * binsperobinz,
                d_subprob_to_bin, d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1], numobins[2], d_idxnupts,
                pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template <typename T>
int cuspread3d_subprob_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    auto &stream = d_plan->stream;

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    if (bin_size_x < 0 || bin_size_y < 0 || bin_size_z < 0) {
        std::cerr << "error: invalid binsize (binsizex, binsizey, binsizez) = (";
        std::cerr << bin_size_x << "," << bin_size_y << "," << bin_size_z << ")" << std::endl;
        return FINUFFT_ERR_BINSIZE_NOTVALID;
    }

    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;
    int pirange = d_plan->spopts.pirange;

    int ier;
    if ((ier = checkCudaErrors(
             hipMemsetAsync(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int), stream))))
        return ier;
    calc_bin_size_noghost_3d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, nf1, nf2, nf3, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binsize, d_kx, d_ky,
        d_kz, d_sortidx, pirange);
    RETURN_IF_CUDA_ERROR

    int n = numbins[0] * numbins[1] * numbins[2];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

    calc_inverse_of_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binstartpts, d_sortidx, d_kx, d_ky,
        d_kz, d_idxnupts, pirange, nf1, nf2, nf3);
    RETURN_IF_CUDA_ERROR
    /* --------------------------------------------- */
    //        Determining Subproblem properties      //
    /* --------------------------------------------- */
    calc_subprob_3d_v2<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(d_binsize, d_numsubprob, maxsubprobsize,
                                                                   numbins[0] * numbins[1] * numbins[2]);
    RETURN_IF_CUDA_ERROR

    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);
    int totalnumsubprob;
    if (checkCudaErrors(hipMemsetAsync(d_subprobstartpts, 0, sizeof(int), stream)) ||
        checkCudaErrors(
            hipMemcpyAsync(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost, stream)
            )
        )
        return FINUFFT_ERR_CUDA_FAILURE;
    hipStreamSynchronize(stream);
    if(checkCudaErrors(hipMallocAsync(&d_subprob_to_bin, totalnumsubprob * sizeof(int), stream)))
        return FINUFFT_ERR_CUDA_FAILURE;

    map_b_into_subprob_3d_v2<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024, 0, stream>>>(
        d_subprob_to_bin, d_subprobstartpts, d_numsubprob, numbins[0] * numbins[1] * numbins[2]);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[%s] Error: %s\n", __func__, hipGetErrorString(err));
        hipFree(d_subprob_to_bin);
        return FINUFFT_ERR_CUDA_FAILURE;
    }

    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != nullptr);
    d_plan->totalnumsubprob = totalnumsubprob;

    return 0;
}

template <typename T>
int cuspread3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) *
                               (bin_size_z + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cerr << "[cuspread3d_subprob] error: not enough shared memory (" << sharedplanorysize << ")" << std::endl;
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth) {
            spread_3d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, sigma, es_c, es_beta,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        } else {
            spread_3d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, sigma, es_c, es_beta,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template int cuspread3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuspread3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cuspread3d_nuptsdriven_prop<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan);
template int cuspread3d_nuptsdriven_prop<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan);
template int cuspread3d_subprob_prop<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan);
template int cuspread3d_subprob_prop<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan);
template int cuspread3d_blockgather_prop<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan);
template int cuspread3d_blockgather_prop<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan);

} // namespace spreadinterp
} // namespace cufinufft
