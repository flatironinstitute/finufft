#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

#include "spreadinterp3d.cuh"

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cufinufft_spread3d(int nf1, int nf2, int nf3, cuda_complex<T> *d_fw, int M, T *d_kx, T *d_ky, T *d_kz,
                       cuda_complex<T> *d_c, cufinufft_plan_t<T> *d_plan)
/*
    This c function is written for only doing 3D spreading. See
    test/spread3d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    int ier;
    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->kz = d_kz;
    d_plan->c = d_c;
    d_plan->fw = d_fw;
    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->nf3 = nf3;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    ier = allocgpumem3d_plan<T>(d_plan);
    ier = allocgpumem3d_nupts<T>(d_plan);

    if (d_plan->opts.gpu_method == 1) {
        ier = cuspread3d_nuptsdriven_prop<T>(nf1, nf2, nf3, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread3d_nuptsdriven_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 2) {
        ier = cuspread3d_subprob_prop<T>(nf1, nf2, nf3, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread3d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 4) {
        ier = cuspread3d_blockgather_prop<T>(nf1, nf2, nf3, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread3d_blockgather_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    ier = cuspread3d<T>(d_plan, 1);
    freegpumemory3d<T>(d_plan);

    return ier;
}

template <typename T>
int cuspread3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem
    (4) Block gather

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int M = d_plan->M;

    int ier = 0;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuspread3d_nuptsdriven<T>(nf1, nf2, nf3, M, d_plan, blksize);

        if (ier != 0) {
            std::cout << "error: cnufftspread3d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        ier = cuspread3d_subprob<T>(nf1, nf2, nf3, M, d_plan, blksize);

        if (ier != 0) {
            std::cout << "error: cnufftspread3d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    case 4: {
        ier = cuspread3d_blockgather<T>(nf1, nf2, nf3, M, d_plan, blksize);

        if (ier != 0) {
            std::cout << "error: cnufftspread3d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cerr << "error: incorrect method, should be 1,2,4" << std::endl;
        return 2;
    }
    return ier;
}

template <typename T>
int cuspread3d_nuptsdriven_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    if (d_plan->opts.gpu_sort) {
        int bin_size_x = d_plan->opts.gpu_binsizex;
        int bin_size_y = d_plan->opts.gpu_binsizey;
        int bin_size_z = d_plan->opts.gpu_binsizez;
        if (bin_size_x < 0 || bin_size_y < 0 || bin_size_z < 0) {
            std::cout << "error: invalid binsize (binsizex, binsizey, binsizez) = (";
            std::cout << bin_size_x << "," << bin_size_y << "," << bin_size_z << ")" << std::endl;
            return 1;
        }

        int numbins[3];
        numbins[0] = ceil((T)nf1 / bin_size_x);
        numbins[1] = ceil((T)nf2 / bin_size_y);
        numbins[2] = ceil((T)nf3 / bin_size_z);

        T *d_kx = d_plan->kx;
        T *d_ky = d_plan->ky;
        T *d_kz = d_plan->kz;

        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;

        checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        calc_bin_size_noghost_3d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, nf3, bin_size_x, bin_size_y, bin_size_z,
                                                                  numbins[0], numbins[1], numbins[2], d_binsize, d_kx,
                                                                  d_ky, d_kz, d_sortidx, pirange);

        int n = numbins[0] * numbins[1] * numbins[2];
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);

        calc_inverse_of_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024>>>(
            M, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binstartpts, d_sortidx, d_kx,
            d_ky, d_kz, d_idxnupts, pirange, nf1, nf2, nf3);
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        trivial_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024>>>(M, d_idxnupts);
    }

    return 0;
}

template <typename T>
int cuspread3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;

    int *d_idxnupts = d_plan->idxnupts;
    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth == 1) {
        for (int t = 0; t < blksize; t++) {
            spread_3d_nupts_driven_horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_kz, d_c + t * M,
                                                                       d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3,
                                                                       sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            spread_3d_nupts_driven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_kz, d_c + t * M,
                                                                d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c,
                                                                es_beta, d_idxnupts, pirange);
        }
    }

    return 0;
}

template <typename T>
int cuspread3d_blockgather_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    dim3 threadsPerBlock;
    dim3 blocks;

    int pirange = d_plan->spopts.pirange;

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int o_bin_size_x = d_plan->opts.gpu_obinsizex;
    int o_bin_size_y = d_plan->opts.gpu_obinsizey;
    int o_bin_size_z = d_plan->opts.gpu_obinsizez;

    int numobins[3];
    if (nf1 % o_bin_size_x != 0 || nf2 % o_bin_size_y != 0 || nf3 % o_bin_size_z != 0) {
        std::cout << "error: mod(nf1, opts.gpu_obinsizex) != 0" << std::endl;
        std::cout << "       mod(nf2, opts.gpu_obinsizey) != 0" << std::endl;
        std::cout << "       mod(nf3, opts.gpu_obinsizez) != 0" << std::endl;
        std::cout << "error: (nf1, nf2, nf3) = (" << nf1 << ", " << nf2 << ", " << nf3 << ")" << std::endl;
        std::cout << "error: (obinsizex, obinsizey, obinsizez) = (" << o_bin_size_x << ", " << o_bin_size_y << ", "
                  << o_bin_size_z << ")" << std::endl;
        return 1;
    }

    numobins[0] = ceil((T)nf1 / o_bin_size_x);
    numobins[1] = ceil((T)nf2 / o_bin_size_y);
    numobins[2] = ceil((T)nf3 / o_bin_size_z);

    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    if (o_bin_size_x % bin_size_x != 0 || o_bin_size_y % bin_size_y != 0 || o_bin_size_z % bin_size_z != 0) {
        std::cout << "error: mod(ops.gpu_obinsizex, opts.gpu_binsizex) != 0" << std::endl;
        std::cout << "       mod(ops.gpu_obinsizey, opts.gpu_binsizey) != 0" << std::endl;
        std::cout << "       mod(ops.gpu_obinsizez, opts.gpu_binsizez) != 0" << std::endl;
        std::cout << "error: (binsizex, binsizey, binsizez) = (" << bin_size_x << ", " << bin_size_y << ", "
                  << bin_size_z << ")" << std::endl;
        std::cout << "error: (obinsizex, obinsizey, obinsizez) = (" << o_bin_size_x << ", " << o_bin_size_y << ", "
                  << o_bin_size_z << ")" << std::endl;
        return 1;
    }

    int binsperobinx, binsperobiny, binsperobinz;
    int numbins[3];
    binsperobinx = o_bin_size_x / bin_size_x + 2;
    binsperobiny = o_bin_size_y / bin_size_y + 2;
    binsperobinz = o_bin_size_z / bin_size_z + 2;
    numbins[0] = numobins[0] * (binsperobinx);
    numbins[1] = numobins[1] * (binsperobiny);
    numbins[2] = numobins[2] * (binsperobinz);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;

    int *d_binsize = d_plan->binsize;
    int *d_sortidx = d_plan->sortidx;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_idxnupts = NULL;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_subprob_to_bin = NULL;

    checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
    locate_nupts_to_bins_ghost<<<(M + 1024 - 1) / 1024, 1024>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numobins[0], numobins[1], numobins[2], binsperobinx, binsperobiny,
        binsperobinz, d_binsize, d_kx, d_ky, d_kz, d_sortidx, pirange, nf1, nf2, nf3);

    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    threadsPerBlock.z = 8;

    blocks.x = (threadsPerBlock.x + numbins[0] - 1) / threadsPerBlock.x;
    blocks.y = (threadsPerBlock.y + numbins[1] - 1) / threadsPerBlock.y;
    blocks.z = (threadsPerBlock.z + numbins[2] - 1) / threadsPerBlock.z;

    fill_ghost_bins<<<blocks, threadsPerBlock>>>(binsperobinx, binsperobiny, binsperobinz, numobins[0], numobins[1],
                                                 numobins[2], d_binsize);

    int n = numbins[0] * numbins[1] * numbins[2];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_binstartpts, 0, sizeof(int)));

    int totalNUpts;
    checkCudaErrors(hipMemcpy(&totalNUpts, &d_binstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_idxnupts, totalNUpts * sizeof(int)));

    calc_inverse_of_global_sort_index_ghost<<<(M + 1024 - 1) / 1024, 1024>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numobins[0], numobins[1], numobins[2], binsperobinx, binsperobiny,
        binsperobinz, d_binstartpts, d_sortidx, d_kx, d_ky, d_kz, d_idxnupts, pirange, nf1, nf2, nf3);

    threadsPerBlock.x = 2;
    threadsPerBlock.y = 2;
    threadsPerBlock.z = 2;

    blocks.x = (threadsPerBlock.x + numbins[0] - 1) / threadsPerBlock.x;
    blocks.y = (threadsPerBlock.y + numbins[1] - 1) / threadsPerBlock.y;
    blocks.z = (threadsPerBlock.z + numbins[2] - 1) / threadsPerBlock.z;

    ghost_bin_pts_index<<<blocks, threadsPerBlock>>>(binsperobinx, binsperobiny, binsperobinz, numobins[0], numobins[1],
                                                     numobins[2], d_binsize, d_idxnupts, d_binstartpts, M);
    if (d_plan->idxnupts != NULL)
        hipFree(d_plan->idxnupts);
    d_plan->idxnupts = d_idxnupts;

    /* --------------------------------------------- */
    //        Determining Subproblem properties      //
    /* --------------------------------------------- */
    n = numobins[0] * numobins[1] * numobins[2];
    calc_subprob_3d_v1<<<(n + 1024 - 1) / 1024, 1024>>>(binsperobinx, binsperobiny, binsperobinz, d_binsize,
                                                        d_numsubprob, maxsubprobsize,
                                                        numobins[0] * numobins[1] * numobins[2]);

    n = numobins[0] * numobins[1] * numobins[2];
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));

    int totalnumsubprob;
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    map_b_into_subprob_3d_v1<<<(n + 1024 - 1) / 1024, 1024>>>(d_subprob_to_bin, d_subprobstartpts, d_numsubprob, n);
    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    d_plan->totalnumsubprob = totalnumsubprob;

    return 0;
}

template <typename T>
int cuspread3d_blockgather(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    int ns = d_plan->spopts.nspread;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    int obin_size_x = d_plan->opts.gpu_obinsizex;
    int obin_size_y = d_plan->opts.gpu_obinsizey;
    int obin_size_z = d_plan->opts.gpu_obinsizez;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numobins[3];
    numobins[0] = ceil((T)nf1 / obin_size_x);
    numobins[1] = ceil((T)nf2 / obin_size_y);
    numobins[2] = ceil((T)nf3 / obin_size_z);

    int binsperobinx, binsperobiny, binsperobinz;
    binsperobinx = obin_size_x / bin_size_x + 2;
    binsperobiny = obin_size_y / bin_size_y + 2;
    binsperobinz = obin_size_z / bin_size_z + 2;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binstartpts = d_plan->binstartpts;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth == 1) {
            size_t sharedplanorysize = obin_size_x * obin_size_y * obin_size_z * sizeof(cuda_complex<T>);
            if (sharedplanorysize > 49152) {
                std::cout << "error: not enough shared memory" << std::endl;
                return 1;
            }
            spread_3d_block_gather_horner<<<totalnumsubprob, 64, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, obin_size_x, obin_size_y, obin_size_z, binsperobinx * binsperobiny * binsperobinz,
                d_subprob_to_bin, d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1], numobins[2], d_idxnupts,
                pirange);
        } else {
            size_t sharedplanorysize = obin_size_x * obin_size_y * obin_size_z * sizeof(cuda_complex<T>);
            if (sharedplanorysize > 49152) {
                std::cout << "error: not enough shared memory" << std::endl;
                return 1;
            }
            spread_3d_block_gather<<<totalnumsubprob, 64, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, obin_size_x, obin_size_y, obin_size_z, binsperobinx * binsperobiny * binsperobinz,
                d_subprob_to_bin, d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1], numobins[2], d_idxnupts,
                pirange);
        }
    }

    return 0;
}

template <typename T>
int cuspread3d_subprob_prop(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan) {
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    if (bin_size_x < 0 || bin_size_y < 0 || bin_size_z < 0) {
        std::cout << "error: invalid binsize (binsizex, binsizey, binsizez) = (";
        std::cout << bin_size_x << "," << bin_size_y << "," << bin_size_z << ")" << std::endl;
        return 1;
    }

    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;
    int pirange = d_plan->spopts.pirange;

    checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
    calc_bin_size_noghost_3d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, nf3, bin_size_x, bin_size_y, bin_size_z,
                                                              numbins[0], numbins[1], numbins[2], d_binsize, d_kx, d_ky,
                                                              d_kz, d_sortidx, pirange);

    int n = numbins[0] * numbins[1] * numbins[2];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);

    calc_inverse_of_global_sort_index_3d<<<(M + 1024 - 1) / 1024, 1024>>>(
        M, bin_size_x, bin_size_y, bin_size_z, numbins[0], numbins[1], numbins[2], d_binstartpts, d_sortidx, d_kx, d_ky,
        d_kz, d_idxnupts, pirange, nf1, nf2, nf3);
    /* --------------------------------------------- */
    //        Determining Subproblem properties      //
    /* --------------------------------------------- */
    calc_subprob_3d_v2<<<(M + 1024 - 1) / 1024, 1024>>>(d_binsize, d_numsubprob, maxsubprobsize,
                                                        numbins[0] * numbins[1] * numbins[2]);

    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));

    int totalnumsubprob;
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    map_b_into_subprob_3d_v2<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024>>>(
        d_subprob_to_bin, d_subprobstartpts, d_numsubprob, numbins[0] * numbins[1] * numbins[2]);
    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != NULL);
    d_plan->totalnumsubprob = totalnumsubprob;

    return 0;
}

template <typename T>
int cuspread3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) *
                               (bin_size_z + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory (" << sharedplanorysize << ")" << std::endl;
        return 1;
    }

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth) {
            spread_3d_subprob_horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts, d_numsubprob,
                maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
        } else {
            spread_3d_subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
        }
    }

    return 0;
}

template int cuspread3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuspread3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

template int cufinufft_spread3d<float>(int nf1, int nf2, int nf3, cuda_complex<float> *d_fw, int M, float *d_kx,
                                       float *d_ky, float *d_kz, cuda_complex<float> *d_c,
                                       cufinufft_plan_t<float> *d_plan);
template int cufinufft_spread3d<double>(int nf1, int nf2, int nf3, cuda_complex<double> *d_fw, int M, double *d_kx,
                                        double *d_ky, double *d_kz, cuda_complex<double> *d_c,
                                        cufinufft_plan_t<double> *d_plan);
} // namespace spreadinterp
} // namespace cufinufft
