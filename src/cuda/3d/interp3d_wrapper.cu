#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>

#include "spreadinterp3d.cuh"

using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cufinufft_interp3d(int nf1, int nf2, int nf3, cuda_complex<T> *d_fw, int M, T *d_kx, T *d_ky, T *d_kz,
                       cuda_complex<T> *d_c, cufinufft_plan_t<T> *d_plan)
/*
    This c function is written for only doing 3D interpolation. See
    test/interp3d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    int ier;
    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->kz = d_kz;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->nf3 = nf3;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    ier = allocgpumem3d_plan<T>(d_plan);
    ier = allocgpumem3d_nupts<T>(d_plan);

    if (d_plan->opts.gpu_method == 1) {
        ier = cuspread3d_nuptsdriven_prop<T>(nf1, nf2, nf3, M, d_plan);
        if (ier != 0) {
            printf("error: cuinterp3d_nuptsdriven_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 2) {
        ier = cuspread3d_subprob_prop<T>(nf1, nf2, nf3, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread3d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    ier = cuinterp3d<T>(d_plan, 1);

    freegpumemory<T>(d_plan);

    return ier;
}

template <typename T>
int cuinterp3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int M = d_plan->M;

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuinterp3d_nuptsdriven<T>(nf1, nf2, nf3, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread3d_gpu_nuptsdriven" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        ier = cuinterp3d_subprob<T>(nf1, nf2, nf3, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread3d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1,2" << std::endl;
        return 2;
    }

    return ier;
}

template <typename T>
int cuinterp3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;
    int pirange = d_plan->spopts.pirange;

    int *d_idxnupts = d_plan->idxnupts;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            interp_3d_nupts_driven<T, 1>
                <<<blocks, threadsPerBlock, 0, 0>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns,
                                                    nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            interp_3d_nupts_driven<T, 0>
                <<<blocks, threadsPerBlock, 0, 0>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns,
                                                    nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
        }
    }

    return 0;
}

template <typename T>
int cuinterp3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;
    int totalnumsubprob = d_plan->totalnumsubprob;

    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) *
                               (bin_size_z + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth == 1) {
            interp_3d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
        } else {
            interp_3d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
        }
    }

    return 0;
}

template int cufinufft_interp3d(int nf1, int nf2, int nf3, cuda_complex<float> *d_fw, int M, float *d_kx, float *d_ky,
                                float *d_kz, cuda_complex<float> *d_c, cufinufft_plan_t<float> *d_plan);
template int cufinufft_interp3d(int nf1, int nf2, int nf3, cuda_complex<double> *d_fw, int M, double *d_kx,
                                double *d_ky, double *d_kz, cuda_complex<double> *d_c,
                                cufinufft_plan_t<double> *d_plan);

template int cuinterp3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

template int cuinterp3d_nuptsdriven<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan,
                                           int blksize);
template int cuinterp3d_nuptsdriven<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan,
                                            int blksize);

template int cuinterp3d_subprob<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d_subprob<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan,
                                        int blksize);

} // namespace spreadinterp
} // namespace cufinufft
