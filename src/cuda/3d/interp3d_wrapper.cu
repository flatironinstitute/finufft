#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>

#include "spreadinterp3d.cuh"

using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cuinterp3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int M = d_plan->M;

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuinterp3d_nuptsdriven<T>(nf1, nf2, nf3, M, d_plan, blksize);
    } break;
    case 2: {
        ier = cuinterp3d_subprob<T>(nf1, nf2, nf3, M, d_plan, blksize);
    } break;
    default:
        std::cerr << "[cuinterp3d] error: incorrect method, should be 1,2\n";
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

    return ier;
}

template <typename T>
int cuinterp3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;
    int pirange = d_plan->spopts.pirange;

    int *d_idxnupts = d_plan->idxnupts;

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            interp_3d_nupts_driven<T, 1>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M,
                                                         ns, nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            interp_3d_nupts_driven<T, 0>
                <<<blocks, threadsPerBlock, 0, stream>>>(d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M,
                                                         ns, nf1, nf2, nf3, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template <typename T>
int cuinterp3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int bin_size_z = d_plan->opts.gpu_binsizez;
    int numbins[3];
    numbins[0] = ceil((T)nf1 / bin_size_x);
    numbins[1] = ceil((T)nf2 / bin_size_y);
    numbins[2] = ceil((T)nf3 / bin_size_z);

    T *d_kx = d_plan->kx;
    T *d_ky = d_plan->ky;
    T *d_kz = d_plan->kz;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;
    int totalnumsubprob = d_plan->totalnumsubprob;

    T sigma = d_plan->spopts.upsampfac;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int pirange = d_plan->spopts.pirange;
    size_t sharedplanorysize = (bin_size_x + 2 * ceil(ns / 2.0)) * (bin_size_y + 2 * ceil(ns / 2.0)) *
                               (bin_size_z + 2 * ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cerr << "[cuinterp3d_subprob] error: not enough shared memory\n";
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }

    for (int t = 0; t < blksize; t++) {
        if (d_plan->opts.gpu_kerevalmeth == 1) {
            interp_3d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        } else {
            interp_3d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
                d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3, es_c, es_beta, sigma,
                d_binstartpts, d_binsize, bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
                d_numsubprob, maxsubprobsize, numbins[0], numbins[1], numbins[2], d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template int cuinterp3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

template int cuinterp3d_nuptsdriven<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan,
                                           int blksize);
template int cuinterp3d_nuptsdriven<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan,
                                            int blksize);

template int cuinterp3d_subprob<float>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d_subprob<double>(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan,
                                        int blksize);

} // namespace spreadinterp
} // namespace cufinufft
