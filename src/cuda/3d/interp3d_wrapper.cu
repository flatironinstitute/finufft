#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>

#include "spreadinterp3d.cuh"
#include <cufinufft/common.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>

using namespace cufinufft::memtransfer;
using namespace cufinufft::common;

namespace cufinufft {
namespace spreadinterp {

template<typename T>
int cuinterp3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
  int nf1 = d_plan->nf1;
  int nf2 = d_plan->nf2;
  int nf3 = d_plan->nf3;
  int M   = d_plan->M;

  int ier;
  switch (d_plan->opts.gpu_method) {
  case 1: {
    ier = cuinterp3d_nuptsdriven<T>(nf1, nf2, nf3, M, d_plan, blksize);
  } break;
  case 2: {
    ier = cuinterp3d_subprob<T>(nf1, nf2, nf3, M, d_plan, blksize);
  } break;
  default:
    std::cerr << "[cuinterp3d] error: incorrect method, should be 1,2\n";
    ier = FINUFFT_ERR_METHOD_NOTVALID;
  }

  return ier;
}

template<typename T>
int cuinterp3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan,
                           int blksize) {
  auto &stream = d_plan->stream;

  dim3 threadsPerBlock;
  dim3 blocks;

  int ns    = d_plan->spopts.nspread; // psi's support in terms of number of cells
  T es_c    = d_plan->spopts.ES_c;
  T es_beta = d_plan->spopts.ES_beta;
  T sigma   = d_plan->spopts.upsampfac;

  int *d_idxnupts = d_plan->idxnupts;

  T *d_kx               = d_plan->kx;
  T *d_ky               = d_plan->ky;
  T *d_kz               = d_plan->kz;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  threadsPerBlock.x = 16;
  threadsPerBlock.y = 1;
  blocks.x          = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
  blocks.y          = 1;

  if (d_plan->opts.gpu_kerevalmeth) {
    for (int t = 0; t < blksize; t++) {
      interp_3d_nupts_driven<T, 1><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3,
          es_c, es_beta, sigma, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  } else {
    for (int t = 0; t < blksize; t++) {
      interp_3d_nupts_driven<T, 0><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3,
          es_c, es_beta, sigma, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }

  return 0;
}

template<typename T>
int cuinterp3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan_t<T> *d_plan,
                       int blksize) {
  auto &stream = d_plan->stream;

  int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
  int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

  // assume that bin_size_x > ns/2;
  int bin_size_x = d_plan->opts.gpu_binsizex;
  int bin_size_y = d_plan->opts.gpu_binsizey;
  int bin_size_z = d_plan->opts.gpu_binsizez;
  int numbins[3];
  numbins[0] = ceil((T)nf1 / bin_size_x);
  numbins[1] = ceil((T)nf2 / bin_size_y);
  numbins[2] = ceil((T)nf3 / bin_size_z);

  T *d_kx               = d_plan->kx;
  T *d_ky               = d_plan->ky;
  T *d_kz               = d_plan->kz;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  int *d_binsize         = d_plan->binsize;
  int *d_binstartpts     = d_plan->binstartpts;
  int *d_numsubprob      = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts        = d_plan->idxnupts;
  int *d_subprob_to_bin  = d_plan->subprob_to_bin;
  int totalnumsubprob    = d_plan->totalnumsubprob;

  T sigma   = d_plan->spopts.upsampfac;
  T es_c    = d_plan->spopts.ES_c;
  T es_beta = d_plan->spopts.ES_beta;
  const auto sharedplanorysize =
      shared_memory_required<T>(3, d_plan->spopts.nspread, d_plan->opts.gpu_binsizex,
                                d_plan->opts.gpu_binsizey, d_plan->opts.gpu_binsizez);

  for (int t = 0; t < blksize; t++) {
    if (d_plan->opts.gpu_kerevalmeth == 1) {
      cufinufft_set_shared_memory(interp_3d_subprob<T, 1>, 3, *d_plan);
      interp_3d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3,
          es_c, es_beta, sigma, d_binstartpts, d_binsize, bin_size_x, bin_size_y,
          bin_size_z, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
          numbins[0], numbins[1], numbins[2], d_idxnupts);
      RETURN_IF_CUDA_ERROR
    } else {
      cufinufft_set_shared_memory(interp_3d_subprob<T, 0>, 3, *d_plan);
      interp_3d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_ky, d_kz, d_c + t * M, d_fw + t * nf1 * nf2 * nf3, M, ns, nf1, nf2, nf3,
          es_c, es_beta, sigma, d_binstartpts, d_binsize, bin_size_x, bin_size_y,
          bin_size_z, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
          numbins[0], numbins[1], numbins[2], d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }

  return 0;
}

template int cuinterp3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

template int cuinterp3d_nuptsdriven<float>(int nf1, int nf2, int nf3, int M,
                                           cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d_nuptsdriven<double>(
    int nf1, int nf2, int nf3, int M, cufinufft_plan_t<double> *d_plan, int blksize);

template int cuinterp3d_subprob<float>(int nf1, int nf2, int nf3, int M,
                                       cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp3d_subprob<double>(int nf1, int nf2, int nf3, int M,
                                        cufinufft_plan_t<double> *d_plan, int blksize);

} // namespace spreadinterp
} // namespace cufinufft
