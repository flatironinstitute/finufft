#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>

#include <cufinufft/memtransfer.h>
#include <cufinufft/profile.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/types.h>

using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
inline int cufinufft_interp1d(int nf1, cuda_complex<T> *d_fw, int M, T *d_kx, cuda_complex<T> *d_c,
                              cufinufft_plan_template<T> *d_plan)
/*
    This c function is written for only doing 1D interpolation. See
    test/interp1d_test.cu for usage.

    note: not allocate,transfer and free memories on gpu.
    Melody Shih 11/21/21
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    d_plan->nf1 = nf1;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    d_plan->kx = d_kx;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    hipEventRecord(start);
    ier = allocgpumem1d_plan(d_plan);
    ier = allocgpumem1d_nupts(d_plan);
    if (d_plan->opts.gpu_method == 1) {
        ier = cuspread1d_nuptsdriven_prop(nf1, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread1d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
    if (d_plan->opts.gpu_method == 2) {
        ier = cuspread1d_subprob_prop(nf1, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread1d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }
#ifdef TIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Obtain Interp Prop\t %.3g ms\n", milliseconds);
#endif
    hipEventRecord(start);
    ier = cuinterp1d(d_plan, 1);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Interp (%d)\t\t %.3g ms\n", d_plan->opts.gpu_method, milliseconds);
#endif
    hipEventRecord(start);
    freegpumemory1d(d_plan);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
    return ier;
}

template int cufinufft_interp1d(int nf1, cuda_complex<float> *d_fw, int M, float *d_kx, cuda_complex<float> *d_c,
                                cufinufft_plan_template<float> *d_plan);
template int cufinufft_interp1d(int nf1, cuda_complex<double> *d_fw, int M, double *d_kx, cuda_complex<double> *d_c,
                                cufinufft_plan_template<double> *d_plan);

template <typename T>
int cuinterp1d(cufinufft_plan_template<T> d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 11/21/21
*/
{
    int nf1 = d_plan->nf1;
    int M = d_plan->M;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        hipEventRecord(start);
        {
            ier = CUINTERP1D_NUPTSDRIVEN(nf1, M, d_plan, blksize);
            if (ier != 0) {
                std::cout << "error: cnufftspread1d_gpu_nuptsdriven" << std::endl;
                return 1;
            }
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1" << std::endl;
        return 2;
    }
#ifdef SPREADTIME
    float milliseconds;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "[time  ]"
              << " Interp " << milliseconds << " ms" << std::endl;
#endif
    return ier;
}

template <typename T>
int cuinterp1d_nuptsdriven(int nf1, int M, cufinufft_plan_template<T> d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->opts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;

    T *d_kx = d_plan->kx;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    hipEventRecord(start);
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Interp_1d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1,
                                                                      sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Interp_1d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c,
                                                               es_beta, d_idxnupts, pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Interp_1d_NUptsdriven (%d)\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

} // namespace spreadinterp
} // namespace cufinufft
