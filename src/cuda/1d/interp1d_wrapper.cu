#include <hip/hip_complex.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>

#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/types.h>

using namespace cufinufft::memtransfer;

#include "spreadinterp1d.cuh"

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cuinterp1d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different interpolation methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 11/21/21
*/
{
    int nf1 = d_plan->nf1;
    int M = d_plan->M;

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        ier = cuinterp1d_nuptsdriven<T>(nf1, M, d_plan, blksize);
    } break;
    default:
        std::cerr << "[cuinterp1d] error: incorrect method, should be 1" << std::endl;
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

    return ier;
}

template <typename T>
int cuinterp1d_nuptsdriven(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
    auto &stream = d_plan->stream;
    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->opts.upsampfac;
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;

    T *d_kx = d_plan->kx;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            interp_1d_nuptsdriven<T, 1><<<blocks, threadsPerBlock, 0, stream>>>(
                d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            interp_1d_nuptsdriven<T, 0><<<blocks, threadsPerBlock, 0, stream>>>(
                d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma, d_idxnupts, pirange);
            RETURN_IF_CUDA_ERROR
        }
    }

    return 0;
}

template int cuinterp1d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuinterp1d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

} // namespace spreadinterp
} // namespace cufinufft
