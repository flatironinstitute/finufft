#include "hip/hip_runtime.h"
#include "cufinufft/types.h"
#include <complex>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <math.h>

#include <cufinufft/cudeconvolve.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft_eitherprec.h>

using namespace cufinufft::deconvolve;
using namespace cufinufft::spreadinterp;
using std::min;

template <typename T>
int cufinufft1d1_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_template<T> *d_plan)
/*
    1D Type-1 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: spread data to oversampled regular mesh using kernel
        Step 2: compute FFT on uniform mesh
        Step 3: deconvolve by division of each Fourier mode independently by the
                Fourier series coefficient of the kernel.

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int blksize;
    int ier;
    CUCPX *d_fkstart;
    CUCPX *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;
        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        checkCudaErrors(
            hipMemset(d_plan->fw, 0, d_plan->maxbatchsize * d_plan->nf1 * sizeof(CUCPX))); // this is needed
#ifdef TIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tInitialize fw to 0\t %.3g s\n", milliseconds / 1000);
#endif
        // Step 1: Spread
        hipEventRecord(start);
        ier = CUSPREAD1D(d_plan, blksize);
        if (ier != 0) {
            printf("error: cuspread1d, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tSpread (%d)\t\t %.3g s\n", milliseconds / 1000, d_plan->opts.gpu_method);
#endif
        // Step 2: FFT
        hipEventRecord(start);
        CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds / 1000);
#endif

        // Step 3: deconvolve and shuffle
        hipEventRecord(start);
        CUDECONVOLVE1D(d_plan, blksize);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds / 1000);
#endif
    }
    return ier;
}

template <typename T>
int cufinufft1d2_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_template<T> *d_plan)
/*
    1D Type-2 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel
                Fourier coeff
        Step 2: compute FFT on uniform mesh
        Step 3: interpolate data to regular mesh

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int blksize;
    int ier;
    cuda_complex<T> *d_fkstart;
    cuda_complex<T> *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;

        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        // Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
        hipEventRecord(start);
        CUDECONVOLVE1D(d_plan, blksize);
#ifdef TIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tAmplify & Copy fktofw\t %.3g s\n", milliseconds / 1000);
#endif
        // Step 2: FFT
        hipDeviceSynchronize();
        hipEventRecord(start);
        CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds / 1000);
#endif

        // Step 3: deconvolve and shuffle
        hipEventRecord(start);
        ier = CUINTERP1D(d_plan, blksize);
        if (ier != 0) {
            printf("error: cuinterp1d, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tUnspread (%d)\t\t %.3g s\n", milliseconds / 1000, d_plan->opts.gpu_method);
#endif
    }
    return ier;
}
