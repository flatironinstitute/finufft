#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <type_traits>

#include <assert.h>
#include <hipfft/hipfft.h>

#include <cufinufft/cudeconvolve.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/types.h>

using namespace cufinufft::deconvolve;
using namespace cufinufft::spreadinterp;

template <typename T>
int cufinufft1d1_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_t<T> *d_plan)
/*
    1D Type-1 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: spread data to oversampled regular mesh using kernel
        Step 2: compute FFT on uniform mesh
        Step 3: deconvolve by division of each Fourier mode independently by the
                Fourier series coefficient of the kernel.

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 1);
    auto &stream = d_plan->stream;

    int ier;
    cuda_complex<T> *d_fkstart;
    cuda_complex<T> *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        int blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;
        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        // this is needed
        if ((ier = checkCudaErrors(
                 hipMemsetAsync(d_plan->fw, 0, d_plan->maxbatchsize * d_plan->nf1 * sizeof(cuda_complex<T>), stream))))
            return ier;

        // Step 1: Spread
        if ((ier = cuspread1d<T>(d_plan, blksize)))
            return ier;

        // Step 2: FFT
        hipfftResult cufft_status = cufft_ex(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
        if (cufft_status != HIPFFT_SUCCESS)
            return FINUFFT_ERR_CUDA_FAILURE;

        // Step 3: deconvolve and shuffle
        if ((ier = cudeconvolve1d<T>(d_plan, blksize)))
            return ier;
    }

    return 0;
}

template <typename T>
int cufinufft1d2_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_t<T> *d_plan)
/*
    1D Type-2 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel
                Fourier coeff
        Step 2: compute FFT on uniform mesh
        Step 3: interpolate data to regular mesh

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 2);

    int ier;
    cuda_complex<T> *d_fkstart;
    cuda_complex<T> *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        int blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;

        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        // Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
        if ((ier = cudeconvolve1d<T>(d_plan, blksize)))
            return ier;

        // Step 2: FFT
        hipfftResult cufft_status = cufft_ex(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
        if (cufft_status != HIPFFT_SUCCESS)
            return FINUFFT_ERR_CUDA_FAILURE;

        // Step 3: deconvolve and shuffle
        if ((ier = cuinterp1d<T>(d_plan, blksize)))
            return ier;
    }

    return 0;
}

template int cufinufft1d1_exec<float>(cuda_complex<float> *d_c, cuda_complex<float> *d_fk,
                                      cufinufft_plan_t<float> *d_plan);
template int cufinufft1d1_exec<double>(cuda_complex<double> *d_c, cuda_complex<double> *d_fk,
                                       cufinufft_plan_t<double> *d_plan);
template int cufinufft1d2_exec<float>(cuda_complex<float> *d_c, cuda_complex<float> *d_fk,
                                      cufinufft_plan_t<float> *d_plan);
template int cufinufft1d2_exec<double>(cuda_complex<double> *d_c, cuda_complex<double> *d_fk,
                                       cufinufft_plan_t<double> *d_plan);
