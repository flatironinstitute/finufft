#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <type_traits>

#include <assert.h>
#include <hipfft/hipfft.h>

#include <cufinufft/cudeconvolve.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/types.h>

using namespace cufinufft::deconvolve;
using namespace cufinufft::spreadinterp;

template <typename T>
int cufinufft1d1_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_template<T> d_plan)
/*
    1D Type-1 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: spread data to oversampled regular mesh using kernel
        Step 2: compute FFT on uniform mesh
        Step 3: deconvolve by division of each Fourier mode independently by the
                Fourier series coefficient of the kernel.

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int blksize;
    int ier;
    cuda_complex<T> *d_fkstart;
    cuda_complex<T> *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;
        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        checkCudaErrors(
            hipMemset(d_plan->fw, 0, d_plan->maxbatchsize * d_plan->nf1 * sizeof(cuda_complex<T>))); // this is needed
#ifdef TIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tInitialize fw to 0\t %.3g s\n", milliseconds / 1000);
#endif
        // Step 1: Spread
        hipEventRecord(start);
        ier = cuspread1d<T>(d_plan, blksize);
        if (ier != 0) {
            printf("error: cuspread1d, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tSpread (%d)\t\t %.3g s\n", milliseconds / 1000, d_plan->opts.gpu_method);
#endif
        // Step 2: FFT
        hipEventRecord(start);
        cufft_ex(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);

#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds / 1000);
#endif

        // Step 3: deconvolve and shuffle
        hipEventRecord(start);
        cudeconvolve1d<T>(d_plan, blksize);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds / 1000);
#endif
    }
    return ier;
}

template <typename T>
int cufinufft1d2_exec(cuda_complex<T> *d_c, cuda_complex<T> *d_fk, cufinufft_plan_template<T> d_plan)
/*
    1D Type-2 NUFFT

    This function is called in "exec" stage (See ../cufinufft.cu).
    It includes (copied from doc in finufft library)
        Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel
                Fourier coeff
        Step 2: compute FFT on uniform mesh
        Step 3: interpolate data to regular mesh

    Melody Shih 11/21/21
*/
{
    assert(d_plan->spopts.spread_direction == 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int blksize;
    int ier;
    cuda_complex<T> *d_fkstart;
    cuda_complex<T> *d_cstart;
    for (int i = 0; i * d_plan->maxbatchsize < d_plan->ntransf; i++) {
        blksize = std::min(d_plan->ntransf - i * d_plan->maxbatchsize, d_plan->maxbatchsize);
        d_cstart = d_c + i * d_plan->maxbatchsize * d_plan->M;
        d_fkstart = d_fk + i * d_plan->maxbatchsize * d_plan->ms;

        d_plan->c = d_cstart;
        d_plan->fk = d_fkstart;

        // Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
        hipEventRecord(start);
        cudeconvolve1d<T>(d_plan, blksize);
#ifdef TIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tAmplify & Copy fktofw\t %.3g s\n", milliseconds / 1000);
#endif
        // Step 2: FFT
        hipDeviceSynchronize();
        hipEventRecord(start);

        cufft_ex(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds / 1000);
#endif

        // Step 3: deconvolve and shuffle
        hipEventRecord(start);
        ier = cuinterp1d<T>(d_plan, blksize);
        if (ier != 0) {
            printf("error: cuinterp1d, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tUnspread (%d)\t\t %.3g s\n", milliseconds / 1000, d_plan->opts.gpu_method);
#endif
    }
    return ier;
}

template int cufinufft1d1_exec<float>(cuda_complex<float> *d_c, cuda_complex<float> *d_fk,
                                      cufinufft_plan_template<float> d_plan);
template int cufinufft1d1_exec<double>(cuda_complex<double> *d_c, cuda_complex<double> *d_fk,
                                       cufinufft_plan_template<double> d_plan);
template int cufinufft1d2_exec<float>(cuda_complex<float> *d_c, cuda_complex<float> *d_fk,
                                      cufinufft_plan_template<float> d_plan);
template int cufinufft1d2_exec<double>(cuda_complex<double> *d_c, cuda_complex<double> *d_fk,
                                       cufinufft_plan_template<double> d_plan);
