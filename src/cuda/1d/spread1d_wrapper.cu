#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

template <typename T>
int cufinufft_spread1d(int nf1, cuda_complex<T> *d_fw, int M, T *d_kx, cuda_complex<T> *d_c, cufinufft_plan_template<T> *d_plan)
/*
    This c function is written for only doing 1D spreading. See
    test/spread1d_test.cu for usage.

    note: not allocate,transfer and free memories on gpu.
    Melody Shih 11/21/21
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    d_plan->kx = d_kx;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    d_plan->nf1 = nf1;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    hipEventRecord(start);
    ier = ALLOCGPUMEM1D_PLAN(d_plan);
    ier = ALLOCGPUMEM1D_NUPTS(d_plan);

    if (d_plan->opts.gpu_method == 1) {
        ier = CUSPREAD1D_NUPTSDRIVEN_PROP(nf1, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread1d_nuptsdriven_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    if (d_plan->opts.gpu_method == 2) {
        ier = CUSPREAD1D_SUBPROB_PROP(nf1, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread1d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

#ifdef TIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Obtain Spread Prop\t %.3g ms\n", milliseconds);
#endif

    hipEventRecord(start);
    ier = CUSPREAD1D(d_plan, 1);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Spread (%d)\t\t %5.3f ms\n", d_plan->opts.gpu_method, milliseconds);
#endif

    hipEventRecord(start);
    FREEGPUMEMORY1D(d_plan);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
    return ier;
}

template <typename T>
int cuspread1d(cufinufft_plan_template<T> *d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 11/21/21
*/
{
    int nf1 = d_plan->nf1;
    int M = d_plan->M;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        hipEventRecord(start);
        ier = CUSPREAD1D_NUPTSDRIVEN(nf1, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread1d_gpu_nuptsdriven" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        hipEventRecord(start);
        ier = CUSPREAD1D_SUBPROB(nf1, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread1d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1,2" << std::endl;
        return 2;
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "[time  ]"
              << " Spread " << milliseconds << " ms" << std::endl;
#endif
    return ier;
}

template <typename T>
int cuspread1d_nuptsdriven_prop(int nf1, int M, cufinufft_plan_template<T> *d_plan) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (d_plan->opts.gpu_sort) {

        int bin_size_x = d_plan->opts.gpu_binsizex;
        if (bin_size_x < 0) {
            std::cout << "error: invalid binsize (binsizex) = (" << bin_size_x << ")" << std::endl;
            return 1;
        }

        int numbins = ceil((T)nf1 / bin_size_x);
#ifdef DEBUG
        std::cout << "[debug ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "]"
                  << std::endl;
        std::cout << "[debug ] numbins = [" << numbins << "]" << std::endl;
#endif

        T *d_kx = d_plan->kx;
#ifdef DEBUG
        T *h_kx;
        h_kx = (T *)malloc(M * sizeof(T));

        checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(T), hipMemcpyDeviceToHost));
        for (int i = 0; i < M; i++) {
            std::cout << "[debug ] ";
            std::cout << "(" << setw(3) << h_kx[i] << ")" << std::endl;
        }
#endif
        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;

        hipEventRecord(start);
        checkCudaErrors(hipMemset(d_binsize, 0, numbins * sizeof(int)));
        CalcBinSize_noghost_1d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx,
                                                                pirange);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcBinSize_noghost_1d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binsize; // For debug
        h_binsize = (int *)malloc(numbins * sizeof(int));
        checkCudaErrors(hipMemcpy(h_binsize, d_binsize, numbins * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] bin size:" << std::endl;
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "bin[" << setw(1) << i << "]=" << h_binsize[i];
        }
        std::cout << std::endl;
        free(h_binsize);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;

        int *h_sortidx;
        h_sortidx = (int *)malloc(M * sizeof(int));

        checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));

        for (int i = 0; i < M; i++) {
            if (h_sortidx[i] < 0) {
                std::cout << "[debug ] ";
                std::cout << "point[" << setw(3) << i << "]=" << setw(3) << h_sortidx[i] << std::endl;
                std::cout << "[debug ] ";
                printf("(%10.10f) ", RESCALE(h_kx[i], nf1, pirange));
                printf("(%10.10f) ", RESCALE(h_kx[i], nf1, pirange) / 32);
                printf("(%f)\n", floor(RESCALE(h_kx[i], nf1, pirange) / 32));
            }
        }
#endif
        hipEventRecord(start);
        int n = numbins;
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel BinStartPts_1d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binstartpts;
        h_binstartpts = (int *)malloc((numbins) * sizeof(int));
        checkCudaErrors(hipMemcpy(h_binstartpts, d_binstartpts, (numbins) * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "bin[" << setw(1) << i << "]=" << h_binstartpts[i];
        }
        std::cout << std::endl;
        free(h_binstartpts);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;
#endif
        hipEventRecord(start);
        CalcInvertofGlobalSortIdx_1d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, numbins, d_binstartpts, d_sortidx,
                                                                      d_kx, d_idxnupts, pirange, nf1);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_1d \t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_idxnupts;
        h_idxnupts = (int *)malloc(M * sizeof(int));
        checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 0; i < M; i++) {
            std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
        }
        free(h_idxnupts);
#endif
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        hipEventRecord(start);
        TrivialGlobalSortIdx_1d<<<(M + 1024 - 1) / 1024, 1024>>>(M, d_idxnupts);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel TrivialGlobalSortIDx_1d \t\t%.3g ms\n", milliseconds);
#endif
    }
    return 0;
}

template <typename T>
int cuspread1d_nuptsdriven(int nf1, int M, cufinufft_plan_template<T> *d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    T sigma = d_plan->spopts.upsampfac;

    T *d_kx = d_plan->kx;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;
    hipEventRecord(start);
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_1d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1,
                                                                      sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_1d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c,
                                                               es_beta, d_idxnupts, pirange);
        }
    }

#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_1d_NUptsdriven (%d)\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

template <typename T>
int cuspread1d_subprob_prop(int nf1, int M, cufinufft_plan_template<T> *d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    if (bin_size_x < 0) {
        std::cout << "error: invalid binsize (binsizex) = (";
        std::cout << bin_size_x << ")" << std::endl;
        return 1;
    }
    int numbins = ceil((T)nf1 / bin_size_x);
#ifdef DEBUG
    std::cout << "[debug  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "]" << std::endl;
    std::cout << "[debug  ] numbins = [" << numbins << "]" << std::endl;
#endif

    T *d_kx = d_plan->kx;

#ifdef DEBUG
    T *h_kx;
    h_kx = (T *)malloc(M * sizeof(T));

    checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(T), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ]";
        std::cout << "(" << setw(3) << h_kx[i] << ")" << std::endl;
    }
#endif
    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;

    int pirange = d_plan->spopts.pirange;

    hipEventRecord(start);
    checkCudaErrors(hipMemset(d_binsize, 0, numbins * sizeof(int)));
    CalcBinSize_noghost_1d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx,
                                                            pirange);
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcBinSize_noghost_1d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binsize; // For debug
    h_binsize = (int *)malloc(numbins * sizeof(int));
    checkCudaErrors(hipMemcpy(h_binsize, d_binsize, numbins * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] bin size:" << std::endl;
    std::cout << "[debug ] ";
    for (int i = 0; i < numbins; i++) {
        if (i != 0)
            std::cout << " ";
        std::cout << "bin[" << setw(3) << i << "]=" << h_binsize[i];
    }
    free(h_binsize);
    std::cout << "[debug ] ----------------------------------------------------" << std::endl;
#endif
#ifdef DEBUG
    int *h_sortidx;
    h_sortidx = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ]";
    for (int i = 0; i < M; i++) {
        std::cout << "[debug] point[" << setw(3) << i << "]=" << setw(3) << h_sortidx[i] << std::endl;
    }

#endif

    hipEventRecord(start);
    int n = numbins;
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel BinStartPts_1d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binstartpts;
    h_binstartpts = (int *)malloc(numbins * sizeof(int));
    checkCudaErrors(hipMemcpy(h_binstartpts, d_binstartpts, numbins * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
    std::cout << "[debug ] ";
    for (int i = 0; i < numbins; i++) {
        if (i != 0)
            std::cout << " ";
        std::cout << "bin[" << setw(3) << i << "] = " << setw(2) << h_binstartpts[i];
    }
    free(h_binstartpts);
    std::cout << "[debug ] ---------------------------------------------------" << std::endl;
#endif
    hipEventRecord(start);
    CalcInvertofGlobalSortIdx_1d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, numbins, d_binstartpts, d_sortidx,
                                                                  d_kx, d_idxnupts, pirange, nf1);
#ifdef DEBUG
    int *h_idxnupts;
    h_idxnupts = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
    }
    free(h_idxnupts);
#endif
    hipEventRecord(start);
    CalcSubProb_1d<<<(M + 1024 - 1) / 1024, 1024>>>(d_binsize, d_numsubprob, maxsubprobsize, numbins);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcSubProb_1d\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_numsubprob;
    h_numsubprob = (int *)malloc(n * sizeof(int));
    checkCudaErrors(hipMemcpy(h_numsubprob, d_numsubprob, numbins * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] ";
    for (int i = 0; i < numbins; i++) {
        if (i != 0)
            std::cout << " ";
        std::cout << "nsub[" << setw(3) << i << "] = " << setw(2) << h_numsubprob[i];
    }
    std::cout << std::endl;
    free(h_numsubprob);
#endif
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Scan Subprob array\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
    printf("[debug ] Subproblem start points\n");
    int *h_subprobstartpts;
    h_subprobstartpts = (int *)malloc((n + 1) * sizeof(int));
    checkCudaErrors(hipMemcpy(h_subprobstartpts, d_subprobstartpts, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] ";
    for (int i = 0; i < numbins; i++) {
        if (i != 0)
            std::cout << " ";
        std::cout << "nsub[" << setw(3) << i << "] = " << setw(2) << h_subprobstartpts[i];
    }
    std::cout << std::endl;
    printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
    free(h_subprobstartpts);
#endif
    hipEventRecord(start);
    int totalnumsubprob;
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    MapBintoSubProb_1d<<<(numbins + 1024 - 1) / 1024, 1024>>>(d_subprob_to_bin, d_subprobstartpts, d_numsubprob,
                                                              numbins);
    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != NULL);
    d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
    printf("[debug ] Map Subproblem to Bins\n");
    int *h_subprob_to_bin;
    h_subprob_to_bin = (int *)malloc((totalnumsubprob) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_subprob_to_bin, d_subprob_to_bin, (totalnumsubprob) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < totalnumsubprob; j++) {
        std::cout << "[debug ] ";
        std::cout << "nsub[" << j << "] = " << setw(2) << h_subprob_to_bin[j];
        std::cout << std::endl;
    }
    free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
    return 0;
}

template <typename T>
int cuspread1d_subprob(int nf1, int M, cufinufft_plan_template<T> *d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    T es_c = d_plan->spopts.ES_c;
    T es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int numbins = ceil((T)nf1 / bin_size_x);

    T *d_kx = d_plan->kx;
    cuda_complex<T> *d_c = d_plan->c;
    cuda_complex<T> *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    int pirange = d_plan->spopts.pirange;

    T sigma = d_plan->opts.upsampfac;
    hipEventRecord(start);

    size_t sharedplanorysize = (bin_size_x + 2 * (int)ceil(ns / 2.0)) * sizeof(cuda_complex<T>);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_1d_Subprob_Horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, sigma, d_binstartpts, d_binsize, bin_size_x,
                d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_1d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma, d_binstartpts, d_binsize,
                bin_size_x, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins, d_idxnupts,
                pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_1d_Subprob (%d)\t\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

} // namespace spreadinterp
} // namespace cufinufft
