#include "hip/hip_runtime.h"
#include <cassert>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <iostream>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/common.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>

using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

#include "spreadinterp1d.cuh"
#include <thrust/sort.h>

namespace cufinufft {
namespace spreadinterp {

// Functor to handle function selection (nuptsdriven vs subprob)
struct Spread1DDispatcher {
  template<int ns, typename T>
  int operator()(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) const {
    switch (d_plan->opts.gpu_method) {
    case 1:
      return cuspread1d_nuptsdriven<T, ns>(nf1, M, d_plan, blksize);
    case 2:
      return cuspread1d_subprob<T, ns>(nf1, M, d_plan, blksize);
    default:
      std::cerr << "[cuspread1d] error: incorrect method, should be 1 or 2\n";
      return FINUFFT_ERR_METHOD_NOTVALID;
    }
  }
};

// Updated cuspread1d using generic dispatch
template<typename T> int cuspread1d(cufinufft_plan_t<T> *d_plan, int blksize) {
  /*
    A wrapper for different spreading methods.

    Methods available:
        (1) Non-uniform points driven

    Melody Shih 11/21/21

    Now the function is updated to dispatch based on ns. This is to avoid alloca which
    it seems slower according to the MRI community.
    Marco Barbone 01/30/25
 */
  return launch_dispatch_ns<Spread1DDispatcher, T>(Spread1DDispatcher(),
                                                   d_plan->spopts.nspread, d_plan->nf1,
                                                   d_plan->M, d_plan, blksize);
}

template<typename T> struct cmp : public thrust::binary_function<int, int, bool> {

  cmp(const T *kx) : kx(kx) {}

  __host__ __device__ bool operator()(const int a, const int b) const {
    return fold_rescale(kx[a], 1) < fold_rescale(kx[b], 1);
  }

private:
  const T *kx;
};

template<typename T>
int cuspread1d_nuptsdriven_prop(int nf1, int M, cufinufft_plan_t<T> *d_plan) {
  auto &stream = d_plan->stream;
  if (d_plan->opts.gpu_sort && d_plan->opts.gpu_method == 1) {
    int *d_idxnupts = d_plan->idxnupts;
    thrust::sequence(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M);
    RETURN_IF_CUDA_ERROR
    thrust::sort(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M,
                 cmp{d_plan->kx});
    RETURN_IF_CUDA_ERROR
    return 0;
  }
  if (d_plan->opts.gpu_sort) {
    int bin_size_x = d_plan->opts.gpu_binsizex;
    if (bin_size_x < 0) {
      std::cerr << "[cuspread1d_nuptsdriven_prop] error: invalid binsize (binsizex) = ("
                << bin_size_x << ")\n";
      return FINUFFT_ERR_BINSIZE_NOTVALID;
    }

    int numbins = ceil((T)nf1 / bin_size_x);

    T *d_kx = d_plan->kx;

    int *d_binsize     = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx     = d_plan->sortidx;
    int *d_idxnupts    = d_plan->idxnupts;

    int ier;
    if ((ier = checkCudaErrors(
             hipMemsetAsync(d_binsize, 0, numbins * sizeof(int), stream))))
      return ier;
    calc_bin_size_noghost_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx);
    RETURN_IF_CUDA_ERROR

    int n = numbins;
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);
    RETURN_IF_CUDA_ERROR

    calc_inverse_of_global_sort_idx_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, numbins, d_binstartpts, d_sortidx, d_kx, d_idxnupts, nf1);
    RETURN_IF_CUDA_ERROR
  } else {
    int *d_idxnupts = d_plan->idxnupts;
    thrust::sequence(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M);
    RETURN_IF_CUDA_ERROR
  }
  return 0;
}

template<typename T, int ns>
int cuspread1d_nuptsdriven(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
  auto &stream = d_plan->stream;
  dim3 threadsPerBlock;
  dim3 blocks;

  int *d_idxnupts = d_plan->idxnupts;
  T es_c          = d_plan->spopts.ES_c;
  T es_beta       = d_plan->spopts.ES_beta;
  T sigma         = d_plan->spopts.upsampfac;

  T *d_kx               = d_plan->kx;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  threadsPerBlock.x = 16;
  threadsPerBlock.y = 1;
  blocks.x          = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
  blocks.y          = 1;

  if (d_plan->opts.gpu_kerevalmeth) {
    for (int t = 0; t < blksize; t++) {
      spread_1d_nuptsdriven<T, 1, ns><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, nf1, es_c, es_beta, sigma, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  } else {
    for (int t = 0; t < blksize; t++) {
      spread_1d_nuptsdriven<T, 0, ns><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, nf1, es_c, es_beta, sigma, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }
  return 0;
}

template<typename T>
int cuspread1d_subprob_prop(int nf1, int M, cufinufft_plan_t<T> *d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{

  const auto maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
  const auto bin_size_x     = d_plan->opts.gpu_binsizex;
  if (bin_size_x < 0) {
    std::cerr << "[cuspread1d_subprob_prop] error: invalid binsize (binsizex) = ("
              << bin_size_x << ")\n";
    return FINUFFT_ERR_BINSIZE_NOTVALID;
  }

  const auto numbins           = (nf1 + bin_size_x - 1) / bin_size_x;
  const auto d_kx              = d_plan->kx;
  const auto d_binsize         = d_plan->binsize;
  const auto d_binstartpts     = d_plan->binstartpts;
  const auto d_sortidx         = d_plan->sortidx;
  const auto d_numsubprob      = d_plan->numsubprob;
  const auto d_subprobstartpts = d_plan->subprobstartpts;
  const auto d_idxnupts        = d_plan->idxnupts;
  const auto stream            = d_plan->stream;

  int *d_subprob_to_bin = nullptr;

  hipMemsetAsync(d_binsize, 0, numbins * sizeof(int), stream);
  RETURN_IF_CUDA_ERROR
  calc_bin_size_noghost_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
      M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx);
  RETURN_IF_CUDA_ERROR

  int n = numbins;
  thrust::device_ptr<int> d_ptr(d_binsize);
  thrust::device_ptr<int> d_result(d_binstartpts);
  thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

  calc_inverse_of_global_sort_idx_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
      M, bin_size_x, numbins, d_binstartpts, d_sortidx, d_kx, d_idxnupts, nf1);
  RETURN_IF_CUDA_ERROR

  calc_subprob_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(d_binsize, d_numsubprob,
                                                              maxsubprobsize, numbins);
  RETURN_IF_CUDA_ERROR

  d_ptr    = thrust::device_pointer_cast(d_numsubprob);
  d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
  thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);
  RETURN_IF_CUDA_ERROR

  hipMemsetAsync(d_subprobstartpts, 0, sizeof(int), stream);
  RETURN_IF_CUDA_ERROR

  int totalnumsubprob{};
  hipMemcpyAsync(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int),
                  hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);
  RETURN_IF_CUDA_ERROR

  cudaMallocWrapper(&d_subprob_to_bin, totalnumsubprob * sizeof(int), stream,
                    d_plan->supports_pools);
  RETURN_IF_CUDA_ERROR

  map_b_into_subprob_1d<<<(numbins + 1024 - 1) / 1024, 1024, 0, stream>>>(
      d_subprob_to_bin, d_subprobstartpts, d_numsubprob, numbins);
  RETURN_IF_CUDA_ERROR
  assert(d_subprob_to_bin != nullptr);
  cudaFreeWrapper(d_plan->subprob_to_bin, stream, d_plan->supports_pools);
  d_plan->subprob_to_bin  = d_subprob_to_bin;
  d_plan->totalnumsubprob = totalnumsubprob;

  return 0;
}

template<typename T, int ns>
int cuspread1d_subprob(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
  auto &stream       = d_plan->stream;
  T es_c             = d_plan->spopts.ES_c;
  T es_beta          = d_plan->spopts.ES_beta;
  int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

  // assume that bin_size_x > ns/2;
  int bin_size_x = d_plan->opts.gpu_binsizex;
  int numbins    = ceil((T)nf1 / bin_size_x);

  T *d_kx               = d_plan->kx;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  int *d_binsize         = d_plan->binsize;
  int *d_binstartpts     = d_plan->binstartpts;
  int *d_numsubprob      = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts        = d_plan->idxnupts;

  int totalnumsubprob   = d_plan->totalnumsubprob;
  int *d_subprob_to_bin = d_plan->subprob_to_bin;

  T sigma = d_plan->opts.upsampfac;

  const auto sharedplanorysize =
      shared_memory_required<T>(1, d_plan->spopts.nspread, d_plan->opts.gpu_binsizex,
                                d_plan->opts.gpu_binsizey, d_plan->opts.gpu_binsizez);

  if (d_plan->opts.gpu_kerevalmeth) {
    if (const auto finufft_err =
            cufinufft_set_shared_memory(spread_1d_subprob<T, 1, ns>, 1, *d_plan) != 0) {
      return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }
    for (int t = 0; t < blksize; t++) {
      spread_1d_subprob<T, 1, ns><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, nf1, es_c, es_beta, sigma, d_binstartpts,
          d_binsize, bin_size_x, d_subprob_to_bin, d_subprobstartpts, d_numsubprob,
          maxsubprobsize, numbins, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  } else {
    if (const auto finufft_err =
            cufinufft_set_shared_memory(spread_1d_subprob<T, 0, ns>, 1, *d_plan) != 0) {
      return FINUFFT_ERR_INSUFFICIENT_SHMEM;
    }
    for (int t = 0; t < blksize; t++) {
      spread_1d_subprob<T, 0, ns><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, nf1, es_c, es_beta, sigma, d_binstartpts,
          d_binsize, bin_size_x, d_subprob_to_bin, d_subprobstartpts, d_numsubprob,
          maxsubprobsize, numbins, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }

  return 0;
}

template int cuspread1d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuspread1d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cuspread1d_nuptsdriven_prop<float>(int nf1, int M,
                                                cufinufft_plan_t<float> *d_plan);
template int cuspread1d_nuptsdriven_prop<double>(int nf1, int M,
                                                 cufinufft_plan_t<double> *d_plan);
template int cuspread1d_subprob_prop<float>(int nf1, int M,
                                            cufinufft_plan_t<float> *d_plan);
template int cuspread1d_subprob_prop<double>(int nf1, int M,
                                             cufinufft_plan_t<double> *d_plan);

} // namespace spreadinterp
} // namespace cufinufft
