#include "hip/hip_runtime.h"
#include <cassert>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <iostream>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/common.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>

using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

#include "spreadinterp1d.cuh"
#include <thrust/sort.h>

namespace cufinufft {
namespace spreadinterp {

template<typename T>
int cuspread1d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 11/21/21
*/
{
  int nf1 = d_plan->nf1;
  int M   = d_plan->M;

  int ier;
  switch (d_plan->opts.gpu_method) {
  case 1: {
    ier = cuspread1d_nuptsdriven<T>(nf1, M, d_plan, blksize);
  } break;
  case 2: {
    ier = cuspread1d_subprob<T>(nf1, M, d_plan, blksize);
  } break;
  default:
    std::cerr << "[cuspread1d] error: incorrect method, should be 1 or 2\n";
    ier = FINUFFT_ERR_METHOD_NOTVALID;
  }

  return ier;
}

template<typename T> struct cmp : public thrust::binary_function<int, int, bool> {

  cmp(const T *kx) : kx(kx) {}

  __host__ __device__ bool operator()(const int a, const int b) const {
    return fold_rescale(kx[a], 1) < fold_rescale(kx[b], 1);
  }

private:
  const T *kx;
};

template<typename T>
int cuspread1d_nuptsdriven_prop(int nf1, int M, cufinufft_plan_t<T> *d_plan) {
  auto &stream = d_plan->stream;
  if (d_plan->opts.gpu_sort && d_plan->opts.gpu_method == 1) {
    int *d_idxnupts = d_plan->idxnupts;
    thrust::sequence(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M);
    RETURN_IF_CUDA_ERROR
    thrust::sort(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M,
                 cmp{d_plan->kx});
    RETURN_IF_CUDA_ERROR
    return 0;
  }
  if (d_plan->opts.gpu_sort) {
    int bin_size_x = d_plan->opts.gpu_binsizex;
    if (bin_size_x < 0) {
      std::cerr << "[cuspread1d_nuptsdriven_prop] error: invalid binsize (binsizex) = ("
                << bin_size_x << ")\n";
      return FINUFFT_ERR_BINSIZE_NOTVALID;
    }

    int numbins = ceil((T)nf1 / bin_size_x);

    T *d_kx = d_plan->kx;

    int *d_binsize     = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx     = d_plan->sortidx;
    int *d_idxnupts    = d_plan->idxnupts;

    int ier;
    if ((ier = checkCudaErrors(
             hipMemsetAsync(d_binsize, 0, numbins * sizeof(int), stream))))
      return ier;
    calc_bin_size_noghost_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx);
    RETURN_IF_CUDA_ERROR

    int n = numbins;
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);
    RETURN_IF_CUDA_ERROR

    calc_inverse_of_global_sort_idx_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
        M, bin_size_x, numbins, d_binstartpts, d_sortidx, d_kx, d_idxnupts, nf1);
    RETURN_IF_CUDA_ERROR
  } else {
    int *d_idxnupts = d_plan->idxnupts;
    thrust::sequence(thrust::cuda::par.on(stream), d_idxnupts, d_idxnupts + M);
    RETURN_IF_CUDA_ERROR
  }
  return 0;
}

template<typename T>
int cuspread1d_nuptsdriven(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
  auto &stream = d_plan->stream;
  dim3 threadsPerBlock;
  dim3 blocks;

  int ns          = d_plan->spopts.nspread; // psi's support in terms of number of cells
  int *d_idxnupts = d_plan->idxnupts;
  T es_c          = d_plan->spopts.ES_c;
  T es_beta       = d_plan->spopts.ES_beta;
  T sigma         = d_plan->spopts.upsampfac;

  T *d_kx               = d_plan->kx;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  threadsPerBlock.x = 16;
  threadsPerBlock.y = 1;
  blocks.x          = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
  blocks.y          = 1;

  if (d_plan->opts.gpu_kerevalmeth) {
    for (int t = 0; t < blksize; t++) {
      spread_1d_nuptsdriven<T, 1><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma,
          d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  } else {
    for (int t = 0; t < blksize; t++) {
      spread_1d_nuptsdriven<T, 0><<<blocks, threadsPerBlock, 0, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma,
          d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }
  return 0;
}

template<typename T>
int cuspread1d_subprob_prop(int nf1, int M, cufinufft_plan_t<T> *d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{

  const auto maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
  const auto bin_size_x     = d_plan->opts.gpu_binsizex;
  if (bin_size_x < 0) {
    std::cerr << "[cuspread1d_subprob_prop] error: invalid binsize (binsizex) = ("
              << bin_size_x << ")\n";
    return FINUFFT_ERR_BINSIZE_NOTVALID;
  }

  const auto numbins           = (nf1 + bin_size_x - 1) / bin_size_x;
  const auto d_kx              = d_plan->kx;
  const auto d_binsize         = d_plan->binsize;
  const auto d_binstartpts     = d_plan->binstartpts;
  const auto d_sortidx         = d_plan->sortidx;
  const auto d_numsubprob      = d_plan->numsubprob;
  const auto d_subprobstartpts = d_plan->subprobstartpts;
  const auto d_idxnupts        = d_plan->idxnupts;
  const auto stream            = d_plan->stream;

  int *d_subprob_to_bin = nullptr;

  hipMemsetAsync(d_binsize, 0, numbins * sizeof(int), stream);
  RETURN_IF_CUDA_ERROR
  calc_bin_size_noghost_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
      M, nf1, bin_size_x, numbins, d_binsize, d_kx, d_sortidx);
  RETURN_IF_CUDA_ERROR

  int n = numbins;
  thrust::device_ptr<int> d_ptr(d_binsize);
  thrust::device_ptr<int> d_result(d_binstartpts);
  thrust::exclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);

  calc_inverse_of_global_sort_idx_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(
      M, bin_size_x, numbins, d_binstartpts, d_sortidx, d_kx, d_idxnupts, nf1);
  RETURN_IF_CUDA_ERROR

  calc_subprob_1d<<<(M + 1024 - 1) / 1024, 1024, 0, stream>>>(d_binsize, d_numsubprob,
                                                              maxsubprobsize, numbins);
  RETURN_IF_CUDA_ERROR

  d_ptr    = thrust::device_pointer_cast(d_numsubprob);
  d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
  thrust::inclusive_scan(thrust::cuda::par.on(stream), d_ptr, d_ptr + n, d_result);
  RETURN_IF_CUDA_ERROR

  hipMemsetAsync(d_subprobstartpts, 0, sizeof(int), stream);
  RETURN_IF_CUDA_ERROR

  int totalnumsubprob{};
  hipMemcpyAsync(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int),
                  hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);
  RETURN_IF_CUDA_ERROR

  cudaMallocWrapper(&d_subprob_to_bin, totalnumsubprob * sizeof(int), stream,
                    d_plan->supports_pools);
  RETURN_IF_CUDA_ERROR

  map_b_into_subprob_1d<<<(numbins + 1024 - 1) / 1024, 1024, 0, stream>>>(
      d_subprob_to_bin, d_subprobstartpts, d_numsubprob, numbins);
  RETURN_IF_CUDA_ERROR
  assert(d_subprob_to_bin != nullptr);
  cudaFreeWrapper(d_plan->subprob_to_bin, stream, d_plan->supports_pools);
  d_plan->subprob_to_bin  = d_subprob_to_bin;
  d_plan->totalnumsubprob = totalnumsubprob;

  return 0;
}

template<typename T>
int cuspread1d_subprob(int nf1, int M, cufinufft_plan_t<T> *d_plan, int blksize) {
  auto &stream = d_plan->stream;

  int ns    = d_plan->spopts.nspread; // psi's support in terms of number of cells
  T es_c    = d_plan->spopts.ES_c;
  T es_beta = d_plan->spopts.ES_beta;
  int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

  // assume that bin_size_x > ns/2;
  int bin_size_x = d_plan->opts.gpu_binsizex;
  int numbins    = ceil((T)nf1 / bin_size_x);

  T *d_kx               = d_plan->kx;
  cuda_complex<T> *d_c  = d_plan->c;
  cuda_complex<T> *d_fw = d_plan->fw;

  int *d_binsize         = d_plan->binsize;
  int *d_binstartpts     = d_plan->binstartpts;
  int *d_numsubprob      = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts        = d_plan->idxnupts;

  int totalnumsubprob   = d_plan->totalnumsubprob;
  int *d_subprob_to_bin = d_plan->subprob_to_bin;

  T sigma = d_plan->opts.upsampfac;

  const auto sharedplanorysize =
      shared_memory_required<T>(1, d_plan->spopts.nspread, d_plan->opts.gpu_binsizex,
                                d_plan->opts.gpu_binsizey, d_plan->opts.gpu_binsizez);

  if (d_plan->opts.gpu_kerevalmeth) {
    for (int t = 0; t < blksize; t++) {

      if (const auto finufft_err =
              cufinufft_set_shared_memory(spread_1d_subprob<T, 1>, 1, *d_plan) != 0) {
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
      }
      RETURN_IF_CUDA_ERROR
      spread_1d_subprob<T, 1><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma,
          d_binstartpts, d_binsize, bin_size_x, d_subprob_to_bin, d_subprobstartpts,
          d_numsubprob, maxsubprobsize, numbins, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  } else {
    for (int t = 0; t < blksize; t++) {
      if (const auto finufft_err =
              cufinufft_set_shared_memory(spread_1d_subprob<T, 0>, 1, *d_plan) != 0) {
        return FINUFFT_ERR_INSUFFICIENT_SHMEM;
      }
      RETURN_IF_CUDA_ERROR
      spread_1d_subprob<T, 0><<<totalnumsubprob, 256, sharedplanorysize, stream>>>(
          d_kx, d_c + t * M, d_fw + t * nf1, M, ns, nf1, es_c, es_beta, sigma,
          d_binstartpts, d_binsize, bin_size_x, d_subprob_to_bin, d_subprobstartpts,
          d_numsubprob, maxsubprobsize, numbins, d_idxnupts);
      RETURN_IF_CUDA_ERROR
    }
  }

  return 0;
}

template int cuspread1d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cuspread1d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cuspread1d_nuptsdriven_prop<float>(int nf1, int M,
                                                cufinufft_plan_t<float> *d_plan);
template int cuspread1d_nuptsdriven_prop<double>(int nf1, int M,
                                                 cufinufft_plan_t<double> *d_plan);
template int cuspread1d_subprob_prop<float>(int nf1, int M,
                                            cufinufft_plan_t<float> *d_plan);
template int cuspread1d_subprob_prop<double>(int nf1, int M,
                                             cufinufft_plan_t<double> *d_plan);

} // namespace spreadinterp
} // namespace cufinufft
