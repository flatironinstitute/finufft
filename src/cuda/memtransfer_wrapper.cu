#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/types.h>
#include <cufinufft/utils.h>

#include <cufinufft/contrib/hip/hip_runtime_api.h>

namespace cufinufft {
namespace memtransfer {

template <typename T>
int allocgpumem1d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;

    int ier;
    int nf1 = d_plan->nf1;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, numbins * sizeof(int), stream))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, numbins * sizeof(int), stream))))
                goto finalize;
        }
    } break;
    case 2: {
        int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->numsubprob, numbins * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, numbins * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, numbins * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->subprobstartpts, (numbins + 1) * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fw, maxbatchsize * nf1 * sizeof(cuda_complex<T>), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem1d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;
    int ier;

    int M = d_plan->M;
    CUDA_FREE_AND_NULL(d_plan->sortidx, stream);
    CUDA_FREE_AND_NULL(d_plan->idxnupts, stream);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort &&
            (ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem1d_nupts] error: invalid method\n";
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem2d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;
    int ier;

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[2];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            if ((ier =
                     checkCudaErrors(hipMallocAsync(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int), stream))))
                goto finalize;
            if ((ier = checkCudaErrors(
                     hipMallocAsync(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int), stream))))
                goto finalize;
        }
    } break;
    case 2: {
        int64_t numbins[2];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        if ((ier =
                 checkCudaErrors(hipMallocAsync(&d_plan->numsubprob, numbins[0] * numbins[1] * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int), stream))))
            goto finalize;
        if ((ier =
                 checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(
                 hipMallocAsync(&d_plan->subprobstartpts, (numbins[0] * numbins[1] + 1) * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem2d_plan] error: invalid method\n";
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(
                 hipMallocAsync(&d_plan->fw, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem2d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;
    int ier;

    const int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx, stream);
    CUDA_FREE_AND_NULL(d_plan->idxnupts, stream);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort &&
            (ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem2d_nupts] error: invalid method\n";
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem3d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;
    int ier;

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            const int64_t nbins_tot = ceil((T)nf1 / d_plan->opts.gpu_binsizex) *
                                      ceil((T)nf2 / d_plan->opts.gpu_binsizey) *
                                      ceil((T)nf3 / d_plan->opts.gpu_binsizez);
            if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, nbins_tot * sizeof(int), stream))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, nbins_tot * sizeof(int), stream))))
                goto finalize;
        }
    } break;
    case 2: {
        const int64_t nbins_tot = ceil((T)nf1 / d_plan->opts.gpu_binsizex) * ceil((T)nf2 / d_plan->opts.gpu_binsizey) *
                                  ceil((T)nf3 / d_plan->opts.gpu_binsizez);

        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->numsubprob, nbins_tot * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, nbins_tot * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, nbins_tot * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->subprobstartpts, (nbins_tot + 1) * sizeof(int), stream))))
            goto finalize;
    } break;
    case 4: {
        const int numobins[3] = {(int)ceil((T)nf1 / d_plan->opts.gpu_obinsizex),
                                 (int)ceil((T)nf2 / d_plan->opts.gpu_obinsizey),
                                 (int)ceil((T)nf3 / d_plan->opts.gpu_obinsizez)};

        const int binsperobins[3] = {d_plan->opts.gpu_obinsizex / d_plan->opts.gpu_binsizex,
                                     d_plan->opts.gpu_obinsizey / d_plan->opts.gpu_binsizey,
                                     d_plan->opts.gpu_obinsizez / d_plan->opts.gpu_binsizez};

        const int numbins[3] = {numobins[0] * (binsperobins[0] + 2), numobins[1] * (binsperobins[1] + 2),
                                numobins[2] * (binsperobins[2] + 2)};

        const int64_t numobins_tot = numobins[0] * numobins[1] * numobins[2];
        const int64_t numbins_tot = numbins[0] * numbins[1] * numbins[2];

        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->numsubprob, numobins_tot * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binsize, numbins_tot * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->binstartpts, (numbins_tot + 1) * sizeof(int), stream))))
            goto finalize;
        if ((ier =
                 checkCudaErrors(hipMallocAsync(&d_plan->subprobstartpts, (numobins_tot + 1) * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem3d_plan] error: invalid method\n";
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(
                 hipMallocAsync(&d_plan->fw, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->fwkerhalf3, (nf3 / 2 + 1) * sizeof(T), stream))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem3d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;
    int ier;
    int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx, stream);
    CUDA_FREE_AND_NULL(d_plan->idxnupts, stream)

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort &&
            ((ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->idxnupts, M * sizeof(int), stream))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
    } break;
    case 4: {
        if ((ier = checkCudaErrors(hipMallocAsync(&d_plan->sortidx, M * sizeof(int), stream))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem3d_nupts] error: invalid method\n";
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
void freegpumemory(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    auto &stream = d_plan->stream;

    CUDA_FREE_AND_NULL(d_plan->fw, stream);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf1, stream);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf2, stream);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf3, stream);

    CUDA_FREE_AND_NULL(d_plan->idxnupts, stream);
    CUDA_FREE_AND_NULL(d_plan->sortidx, stream);
    CUDA_FREE_AND_NULL(d_plan->numsubprob, stream);
    CUDA_FREE_AND_NULL(d_plan->binsize, stream);
    CUDA_FREE_AND_NULL(d_plan->binstartpts, stream);
    CUDA_FREE_AND_NULL(d_plan->subprob_to_bin, stream);
    CUDA_FREE_AND_NULL(d_plan->subprobstartpts, stream);

    CUDA_FREE_AND_NULL(d_plan->numnupts, stream);
    CUDA_FREE_AND_NULL(d_plan->numsubprob, stream);
}

template int allocgpumem1d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem1d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template void freegpumemory<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem2d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem2d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem3d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem3d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_nupts<double>(cufinufft_plan_t<double> *d_plan);

} // namespace memtransfer
} // namespace cufinufft
