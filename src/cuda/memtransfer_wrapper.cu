#include "hip/hip_runtime.h"
#include <cufinufft/types.h>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/memtransfer.h>
#include <hip/hip_runtime_api.h>

namespace cufinufft {
namespace memtransfer {

template <typename T>
int allocgpumem1d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 11/21/21
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int maxbatchsize = d_plan->maxbatchsize;

    d_plan->byte_now = 0;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)));
            checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)));
        }
    } break;
    case 2: {
        int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins + 1) * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * sizeof(cuda_complex<T>)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)));
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
    return 0;
}

template <typename T>
int allocgpumem1d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 11/21/21
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int M = d_plan->M;

    if (d_plan->sortidx) {
        checkCudaErrors(hipFree(d_plan->sortidx));
        d_plan->sortidx = nullptr;
    }
    if (d_plan->idxnupts) {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        d_plan->idxnupts = nullptr;
    }

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort)
            checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
    } break;
    case 2:
    case 3: {
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
void freegpumemory1d(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 11/21/21
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    if (!d_plan->opts.gpu_spreadinterponly) {
        checkCudaErrors(hipFree(d_plan->fw));
        checkCudaErrors(hipFree(d_plan->fwkerhalf1));
    }
    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            checkCudaErrors(hipFree(d_plan->idxnupts));
            checkCudaErrors(hipFree(d_plan->sortidx));
            checkCudaErrors(hipFree(d_plan->binsize));
            checkCudaErrors(hipFree(d_plan->binstartpts));
        } else {
            checkCudaErrors(hipFree(d_plan->idxnupts));
        }
    } break;
    case 2: {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        checkCudaErrors(hipFree(d_plan->sortidx));
        checkCudaErrors(hipFree(d_plan->numsubprob));
        checkCudaErrors(hipFree(d_plan->binsize));
        checkCudaErrors(hipFree(d_plan->binstartpts));
        checkCudaErrors(hipFree(d_plan->subprobstartpts));
        checkCudaErrors(hipFree(d_plan->subprob_to_bin));
    } break;
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
}

template <typename T>
int allocgpumem2d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int maxbatchsize = d_plan->maxbatchsize;

    d_plan->byte_now = 0;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[2];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)));
            checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)));
        }
    } break;
    case 2: {
        int numbins[2];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins[0] * numbins[1] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins[0] * numbins[1] + 1) * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)));
    }

    hipStream_t *streams = (hipStream_t *)malloc(d_plan->opts.gpu_nstreams * sizeof(hipStream_t));
    for (int i = 0; i < d_plan->opts.gpu_nstreams; i++)
        checkCudaErrors(hipStreamCreate(&streams[i]));
    d_plan->streams = streams;

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
    return 0;
}

template <typename T>
int allocgpumem2d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int M = d_plan->M;

    if (d_plan->sortidx) {
        checkCudaErrors(hipFree(d_plan->sortidx));
        d_plan->sortidx = nullptr;
    }
    if (d_plan->idxnupts) {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        d_plan->idxnupts = nullptr;
    }

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort)
            checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
    } break;
    case 2: {
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
void freegpumemory2d(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    if (!d_plan->opts.gpu_spreadinterponly) {
        checkCudaErrors(hipFree(d_plan->fw));
        checkCudaErrors(hipFree(d_plan->fwkerhalf1));
        checkCudaErrors(hipFree(d_plan->fwkerhalf2));
    }
    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            checkCudaErrors(hipFree(d_plan->idxnupts));
            checkCudaErrors(hipFree(d_plan->sortidx));
            checkCudaErrors(hipFree(d_plan->binsize));
            checkCudaErrors(hipFree(d_plan->binstartpts));
        } else {
            checkCudaErrors(hipFree(d_plan->idxnupts));
        }
    } break;
    case 2: {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        checkCudaErrors(hipFree(d_plan->sortidx));
        checkCudaErrors(hipFree(d_plan->numsubprob));
        checkCudaErrors(hipFree(d_plan->binsize));
        checkCudaErrors(hipFree(d_plan->binstartpts));
        checkCudaErrors(hipFree(d_plan->subprobstartpts));
        checkCudaErrors(hipFree(d_plan->subprob_to_bin));
    } break;
    }

    for (int i = 0; i < d_plan->opts.gpu_nstreams; i++)
        checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
}

template <typename T>
int allocgpumem3d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int maxbatchsize = d_plan->maxbatchsize;

    d_plan->byte_now = 0;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[3];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            numbins[2] = ceil((T)nf3 / d_plan->opts.gpu_binsizez);
            checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
            checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        }
    } break;
    case 2: {
        int numbins[3];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        numbins[2] = ceil((T)nf3 / d_plan->opts.gpu_binsizez);
        checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins[0] * numbins[1] * numbins[2] + 1) * sizeof(int)));
    } break;
    case 4: {
        int numobins[3], numbins[3];
        int binsperobins[3];
        numobins[0] = ceil((T)nf1 / d_plan->opts.gpu_obinsizex);
        numobins[1] = ceil((T)nf2 / d_plan->opts.gpu_obinsizey);
        numobins[2] = ceil((T)nf3 / d_plan->opts.gpu_obinsizez);

        binsperobins[0] = d_plan->opts.gpu_obinsizex / d_plan->opts.gpu_binsizex;
        binsperobins[1] = d_plan->opts.gpu_obinsizey / d_plan->opts.gpu_binsizey;
        binsperobins[2] = d_plan->opts.gpu_obinsizez / d_plan->opts.gpu_binsizez;

        numbins[0] = numobins[0] * (binsperobins[0] + 2);
        numbins[1] = numobins[1] * (binsperobins[1] + 2);
        numbins[2] = numobins[2] * (binsperobins[2] + 2);

        checkCudaErrors(hipMalloc(&d_plan->numsubprob, numobins[0] * numobins[1] * numobins[2] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->binstartpts, (numbins[0] * numbins[1] * numbins[2] + 1) * sizeof(int)));
        checkCudaErrors(
            hipMalloc(&d_plan->subprobstartpts, (numobins[0] * numobins[1] * numobins[2] + 1) * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3, (nf3 / 2 + 1) * sizeof(T)));
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
int allocgpumem3d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int M = d_plan->M;

    d_plan->byte_now = 0;

    if (d_plan->sortidx) {
        checkCudaErrors(hipFree(d_plan->sortidx));
        d_plan->sortidx = nullptr;
    }
    if (d_plan->idxnupts) {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        d_plan->idxnupts = nullptr;
    }

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort)
            checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
    } break;
    case 2: {
        checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)));
        checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
    } break;
    case 4: {
        checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)));
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
void freegpumemory3d(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    if (!d_plan->opts.gpu_spreadinterponly) {
        hipFree(d_plan->fw);
        hipFree(d_plan->fwkerhalf1);
        hipFree(d_plan->fwkerhalf2);
        hipFree(d_plan->fwkerhalf3);
    }

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            checkCudaErrors(hipFree(d_plan->idxnupts));
            checkCudaErrors(hipFree(d_plan->sortidx));
            checkCudaErrors(hipFree(d_plan->binsize));
            checkCudaErrors(hipFree(d_plan->binstartpts));
        } else {
            checkCudaErrors(hipFree(d_plan->idxnupts));
        }
    } break;
    case 2: {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        checkCudaErrors(hipFree(d_plan->sortidx));
        checkCudaErrors(hipFree(d_plan->numsubprob));
        checkCudaErrors(hipFree(d_plan->binsize));
        checkCudaErrors(hipFree(d_plan->binstartpts));
        checkCudaErrors(hipFree(d_plan->subprobstartpts));
        checkCudaErrors(hipFree(d_plan->subprob_to_bin));
    } break;
    case 4: {
        checkCudaErrors(hipFree(d_plan->idxnupts));
        checkCudaErrors(hipFree(d_plan->sortidx));
        checkCudaErrors(hipFree(d_plan->numsubprob));
        checkCudaErrors(hipFree(d_plan->binsize));
        checkCudaErrors(hipFree(d_plan->binstartpts));
        checkCudaErrors(hipFree(d_plan->subprobstartpts));
        checkCudaErrors(hipFree(d_plan->subprob_to_bin));
    } break;
    }

    for (int i = 0; i < d_plan->opts.gpu_nstreams; i++)
        checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
}

template int allocgpumem1d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem1d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_nupts<double>(cufinufft_plan_t<double> *d_plan);
template void freegpumemory1d<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory1d<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem2d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem2d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_nupts<double>(cufinufft_plan_t<double> *d_plan);
template void freegpumemory2d<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory2d<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem3d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem3d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_nupts<double>(cufinufft_plan_t<double> *d_plan);
template void freegpumemory3d<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory3d<double>(cufinufft_plan_t<double> *d_plan);

} // namespace memtransfer
} // namespace cufinufft
