#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/memtransfer.h>
#include <cufinufft/types.h>
#include <cufinufft/utils.h>

#include <cufinufft/contrib/hip/hip_runtime_api.h>

namespace cufinufft {
namespace memtransfer {

template <typename T>
int allocgpumem1d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);

    int ier;
    int nf1 = d_plan->nf1;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem1d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    int ier;

    int M = d_plan->M;
    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && (ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem1d_nupts] error: invalid method\n";
        ier = FINUFFT_ERR_METHOD_NOTVALID;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem2d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    int ier;

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[2];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        int64_t numbins[2];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins[0] * numbins[1] + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem2d_plan] error: invalid method\n";
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem2d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    int ier;

    const int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && (ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem2d_nupts] error: invalid method\n";
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem3d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    int ier;

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            const int64_t nbins_tot = ceil((T)nf1 / d_plan->opts.gpu_binsizex) *
                                      ceil((T)nf2 / d_plan->opts.gpu_binsizey) *
                                      ceil((T)nf3 / d_plan->opts.gpu_binsizez);
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, nbins_tot * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, nbins_tot * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        const int64_t nbins_tot = ceil((T)nf1 / d_plan->opts.gpu_binsizex) * ceil((T)nf2 / d_plan->opts.gpu_binsizey) *
                                  ceil((T)nf3 / d_plan->opts.gpu_binsizez);

        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, nbins_tot * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, nbins_tot * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, nbins_tot * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (nbins_tot + 1) * sizeof(int)))))
            goto finalize;
    } break;
    case 4: {
        const int numobins[3] = {(int)ceil((T)nf1 / d_plan->opts.gpu_obinsizex),
                                 (int)ceil((T)nf2 / d_plan->opts.gpu_obinsizey),
                                 (int)ceil((T)nf3 / d_plan->opts.gpu_obinsizez)};

        const int binsperobins[3] = {d_plan->opts.gpu_obinsizex / d_plan->opts.gpu_binsizex,
                                     d_plan->opts.gpu_obinsizey / d_plan->opts.gpu_binsizey,
                                     d_plan->opts.gpu_obinsizez / d_plan->opts.gpu_binsizez};

        const int numbins[3] = {numobins[0] * (binsperobins[0] + 2), numobins[1] * (binsperobins[1] + 2),
                                numobins[2] * (binsperobins[2] + 2)};

        const int64_t numobins_tot = numobins[0] * numobins[1] * numobins[2];
        const int64_t numbins_tot = numbins[0] * numbins[1] * numbins[2];

        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, numobins_tot * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins_tot * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, (numbins_tot + 1) * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numobins_tot + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem3d_plan] error: invalid method\n";
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3, (nf3 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
int allocgpumem3d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);
    int ier;
    int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts)

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int))))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    case 4: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "[allocgpumem3d_nupts] error: invalid method\n";
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    return ier;
}

template <typename T>
void freegpumemory(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 11/21/21
*/
{
    utils::WithCudaDevice device_swapper(d_plan->opts.gpu_device_id);

    CUDA_FREE_AND_NULL(d_plan->fw);
    CUDA_FREE_AND_NULL(d_plan->fw);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf1);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf2);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf3);

    CUDA_FREE_AND_NULL(d_plan->idxnupts);
    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->numsubprob);
    CUDA_FREE_AND_NULL(d_plan->binsize);
    CUDA_FREE_AND_NULL(d_plan->binstartpts);
    CUDA_FREE_AND_NULL(d_plan->subprob_to_bin);
    CUDA_FREE_AND_NULL(d_plan->subprobstartpts);

    CUDA_FREE_AND_NULL(d_plan->numnupts);
    CUDA_FREE_AND_NULL(d_plan->numsubprob);
}

template int allocgpumem1d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem1d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template void freegpumemory<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem2d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem2d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem3d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem3d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_nupts<double>(cufinufft_plan_t<double> *d_plan);

} // namespace memtransfer
} // namespace cufinufft
