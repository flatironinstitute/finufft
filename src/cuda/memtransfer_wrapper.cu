#include "hip/hip_runtime.h"
#include <cufinufft/types.h>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <cufinufft/memtransfer.h>
#include <hip/hip_runtime_api.h>

namespace cufinufft {
namespace memtransfer {

template <typename T>
int allocgpumem1d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 11/21/21
*/
{
    // Multi-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    int ier = 0;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        int numbins = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return ier;
}

template <typename T>
int allocgpumem1d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 11/21/21
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id, ier;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int M = d_plan->M;
    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && (ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2:
    case 3: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return ier;
}

template <typename T>
int allocgpumem2d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id, ier;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[2];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        int64_t numbins[2];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->subprobstartpts, (numbins[0] * numbins[1] + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method " << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return ier;
}

template <typename T>
int allocgpumem2d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id, ier;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    const int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts);

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && (ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return ier;
}

template <typename T>
int allocgpumem3d_plan(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "plan" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id, ier;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int maxbatchsize = d_plan->maxbatchsize;

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort) {
            int numbins[3];
            numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
            numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
            numbins[2] = ceil((T)nf3 / d_plan->opts.gpu_binsizez);
            if ((ier =
                     checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
                goto finalize;
            if ((ier = checkCudaErrors(
                     hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
                goto finalize;
        }
    } break;
    case 2: {
        int numbins[3];
        numbins[0] = ceil((T)nf1 / d_plan->opts.gpu_binsizex);
        numbins[1] = ceil((T)nf2 / d_plan->opts.gpu_binsizey);
        numbins[2] = ceil((T)nf3 / d_plan->opts.gpu_binsizez);
        if ((ier =
                 checkCudaErrors(hipMalloc(&d_plan->numsubprob, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
            goto finalize;
        if ((ier =
                 checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(
                 hipMalloc(&d_plan->subprobstartpts, (numbins[0] * numbins[1] * numbins[2] + 1) * sizeof(int)))))
            goto finalize;
    } break;
    case 4: {
        int numobins[3], numbins[3];
        int binsperobins[3];
        numobins[0] = ceil((T)nf1 / d_plan->opts.gpu_obinsizex);
        numobins[1] = ceil((T)nf2 / d_plan->opts.gpu_obinsizey);
        numobins[2] = ceil((T)nf3 / d_plan->opts.gpu_obinsizez);

        binsperobins[0] = d_plan->opts.gpu_obinsizex / d_plan->opts.gpu_binsizex;
        binsperobins[1] = d_plan->opts.gpu_obinsizey / d_plan->opts.gpu_binsizey;
        binsperobins[2] = d_plan->opts.gpu_obinsizez / d_plan->opts.gpu_binsizez;

        numbins[0] = numobins[0] * (binsperobins[0] + 2);
        numbins[1] = numobins[1] * (binsperobins[1] + 2);
        numbins[2] = numobins[2] * (binsperobins[2] + 2);

        if ((ier = checkCudaErrors(
                 hipMalloc(&d_plan->numsubprob, numobins[0] * numobins[1] * numobins[2] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->binsize, numbins[0] * numbins[1] * numbins[2] * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(
                 hipMalloc(&d_plan->binstartpts, (numbins[0] * numbins[1] * numbins[2] + 1) * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(
                 hipMalloc(&d_plan->subprobstartpts, (numobins[0] * numobins[1] * numobins[2] + 1) * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

    if (!d_plan->opts.gpu_spreadinterponly) {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1, (nf1 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2, (nf2 / 2 + 1) * sizeof(T)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3, (nf3 / 2 + 1) * sizeof(T)))))
            goto finalize;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
int allocgpumem3d_nupts(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for gpu memory allocation in "setNUpts" stage.

    Melody Shih 07/25/19
*/
{
    // Mult-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id, ier;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    int M = d_plan->M;

    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->idxnupts)

    switch (d_plan->opts.gpu_method) {
    case 1: {
        if (d_plan->opts.gpu_sort && ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int))))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
    } break;
    case 2: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->idxnupts, M * sizeof(int)))))
            goto finalize;
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    case 4: {
        if ((ier = checkCudaErrors(hipMalloc(&d_plan->sortidx, M * sizeof(int)))))
            goto finalize;
    } break;
    default:
        std::cerr << "err: invalid method" << std::endl;
    }

finalize:
    if (ier)
        freegpumemory(d_plan);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);

    return 0;
}

template <typename T>
void freegpumemory(cufinufft_plan_t<T> *d_plan)
/*
    wrapper for freeing gpu memory.

    Melody Shih 11/21/21
*/
{
    // Multi-GPU support: set the CUDA Device ID:
    int orig_gpu_device_id;
    hipGetDevice(&orig_gpu_device_id);
    hipSetDevice(d_plan->opts.gpu_device_id);

    CUDA_FREE_AND_NULL(d_plan->fw);
    CUDA_FREE_AND_NULL(d_plan->fw);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf1);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf2);
    CUDA_FREE_AND_NULL(d_plan->fwkerhalf3);

    CUDA_FREE_AND_NULL(d_plan->idxnupts);
    CUDA_FREE_AND_NULL(d_plan->sortidx);
    CUDA_FREE_AND_NULL(d_plan->numsubprob);
    CUDA_FREE_AND_NULL(d_plan->binsize);
    CUDA_FREE_AND_NULL(d_plan->binstartpts);
    CUDA_FREE_AND_NULL(d_plan->subprob_to_bin);
    CUDA_FREE_AND_NULL(d_plan->subprobstartpts);

    CUDA_FREE_AND_NULL(d_plan->numnupts);
    CUDA_FREE_AND_NULL(d_plan->numsubprob);

    // Multi-GPU support: reset the device ID
    hipSetDevice(orig_gpu_device_id);
}

template int allocgpumem1d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem1d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem1d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template void freegpumemory<float>(cufinufft_plan_t<float> *d_plan);
template void freegpumemory<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem2d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem2d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem2d_nupts<double>(cufinufft_plan_t<double> *d_plan);

template int allocgpumem3d_plan<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_plan<double>(cufinufft_plan_t<double> *d_plan);
template int allocgpumem3d_nupts<float>(cufinufft_plan_t<float> *d_plan);
template int allocgpumem3d_nupts<double>(cufinufft_plan_t<double> *d_plan);

} // namespace memtransfer
} // namespace cufinufft
