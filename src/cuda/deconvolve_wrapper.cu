#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>

#include <cufinufft/cudeconvolve.h>

namespace cufinufft {
namespace deconvolve {
/* Kernel for copying fw to fk with amplication by prefac/ker */
// Note: assume modeord=0: CMCL-compatible mode ordering in fk (from -N/2 up
// to N/2-1), modeord=1: FFT-compatible mode ordering in fk (from 0 to N/2-1, then -N/2 up
// to -1).
template<typename T, int modeord>
__global__ void deconvolve_1d(int ms, int nf1, cuda_complex<T> *fw, cuda_complex<T> *fk,
                              T *fwkerhalf1) {
  int pivot1, w1, fwkerind1;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms;
       i += blockDim.x * gridDim.x) {
    if (modeord == 0) {
      pivot1    = i - ms / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      fwkerind1 = abs(pivot1);
    } else {
      pivot1    = i - ms + ms / 2;
      w1        = (pivot1 >= 0) ? nf1 + i - ms : i;
      fwkerind1 = (pivot1 >= 0) ? ms - i : i;
    }

    kervalue = fwkerhalf1[fwkerind1];
    fk[i].x  = fw[w1].x / kervalue;
    fk[i].y  = fw[w1].y / kervalue;
  }
}

template<typename T, int modeord>
__global__ void deconvolve_2d(int ms, int mt, int nf1, int nf2, cuda_complex<T> *fw,
                              cuda_complex<T> *fk, T *fwkerhalf1, T *fwkerhalf2) {
  int pivot1, pivot2, w1, w2, fwkerind1, fwkerind2;
  int k1, k2, inidx, outidx;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt;
       i += blockDim.x * gridDim.x) {
    k1     = i % ms;
    k2     = i / ms;
    outidx = k1 + k2 * ms;

    if (modeord == 0) {
      pivot1    = k1 - ms / 2;
      pivot2    = k2 - mt / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      w2        = (pivot2 >= 0) ? pivot2 : nf2 + pivot2;
      fwkerind1 = abs(pivot1);
      fwkerind2 = abs(pivot2);
    } else {
      pivot1    = k1 - ms + ms / 2;
      pivot2    = k2 - mt + mt / 2;
      w1        = (pivot1 >= 0) ? nf1 + k1 - ms : k1;
      w2        = (pivot2 >= 0) ? nf2 + k2 - mt : k2;
      fwkerind1 = (pivot1 >= 0) ? ms - k1 : k1;
      fwkerind2 = (pivot2 >= 0) ? mt - k2 : k2;
    }

    inidx        = w1 + w2 * nf1;
    kervalue     = fwkerhalf1[fwkerind1] * fwkerhalf2[fwkerind2];
    fk[outidx].x = fw[inidx].x / kervalue;
    fk[outidx].y = fw[inidx].y / kervalue;
  }
}

template<typename T, int modeord>
__global__ void deconvolve_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3,
                              cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1,
                              T *fwkerhalf2, T *fwkerhalf3) {
  int pivot1, pivot2, pivot3, w1, w2, w3, fwkerind1, fwkerind2, fwkerind3;
  int k1, k2, k3, inidx, outidx;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt * mu;
       i += blockDim.x * gridDim.x) {
    k1     = i % ms;
    k2     = (i / ms) % mt;
    k3     = (i / ms / mt);
    outidx = k1 + k2 * ms + k3 * ms * mt;

    if (modeord == 0) {
      pivot1    = k1 - ms / 2;
      pivot2    = k2 - mt / 2;
      pivot3    = k3 - mu / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      w2        = (pivot2 >= 0) ? pivot2 : nf2 + pivot2;
      w3        = (pivot3 >= 0) ? pivot3 : nf3 + pivot3;
      fwkerind1 = abs(pivot1);
      fwkerind2 = abs(pivot2);
      fwkerind3 = abs(pivot3);
    } else {
      pivot1    = k1 - ms + ms / 2;
      pivot2    = k2 - mt + mt / 2;
      pivot3    = k3 - mu + mu / 2;
      w1        = (pivot1 >= 0) ? nf1 + k1 - ms : k1;
      w2        = (pivot2 >= 0) ? nf2 + k2 - mt : k2;
      w3        = (pivot3 >= 0) ? nf3 + k3 - mu : k3;
      fwkerind1 = (pivot1 >= 0) ? ms - k1 : k1;
      fwkerind2 = (pivot2 >= 0) ? mt - k2 : k2;
      fwkerind3 = (pivot3 >= 0) ? mu - k3 : k3;
    }

    inidx        = w1 + w2 * nf1 + w3 * nf1 * nf2;
    kervalue     = fwkerhalf1[fwkerind1] * fwkerhalf2[fwkerind2] * fwkerhalf3[fwkerind3];
    fk[outidx].x = fw[inidx].x / kervalue;
    fk[outidx].y = fw[inidx].y / kervalue;
  }
}

/* Kernel for copying fk to fw with same amplication */
template<typename T, int modeord>
__global__ void amplify_1d(int ms, int nf1, cuda_complex<T> *fw, cuda_complex<T> *fk,
                           T *fwkerhalf1) {
  int pivot1, w1, fwkerind1;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms;
       i += blockDim.x * gridDim.x) {
    if (modeord == 0) {
      pivot1    = i - ms / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      fwkerind1 = abs(pivot1);
    } else {
      pivot1    = i - ms + ms / 2;
      w1        = (pivot1 >= 0) ? nf1 + i - ms : i;
      fwkerind1 = (pivot1 >= 0) ? ms - i : i;
    }

    kervalue = fwkerhalf1[fwkerind1];
    fw[w1].x = fk[i].x / kervalue;
    fw[w1].y = fk[i].y / kervalue;
  }
}

template<typename T, int modeord>
__global__ void amplify_2d(int ms, int mt, int nf1, int nf2, cuda_complex<T> *fw,
                           cuda_complex<T> *fk, T *fwkerhalf1, T *fwkerhalf2) {
  int pivot1, pivot2, w1, w2, fwkerind1, fwkerind2;
  int k1, k2, inidx, outidx;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt;
       i += blockDim.x * gridDim.x) {
    k1    = i % ms;
    k2    = i / ms;
    inidx = k1 + k2 * ms;

    if (modeord == 0) {
      pivot1    = k1 - ms / 2;
      pivot2    = k2 - mt / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      w2        = (pivot2 >= 0) ? pivot2 : nf2 + pivot2;
      fwkerind1 = abs(pivot1);
      fwkerind2 = abs(pivot2);
    } else {
      pivot1    = k1 - ms + ms / 2;
      pivot2    = k2 - mt + mt / 2;
      w1        = (pivot1 >= 0) ? nf1 + k1 - ms : k1;
      w2        = (pivot2 >= 0) ? nf2 + k2 - mt : k2;
      fwkerind1 = (pivot1 >= 0) ? ms - k1 : k1;
      fwkerind2 = (pivot2 >= 0) ? mt - k2 : k2;
    }

    outidx       = w1 + w2 * nf1;
    kervalue     = fwkerhalf1[fwkerind1] * fwkerhalf2[fwkerind2];
    fw[outidx].x = fk[inidx].x / kervalue;
    fw[outidx].y = fk[inidx].y / kervalue;
  }
}

template<typename T, int modeord>
__global__ void amplify_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3,
                           cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1,
                           T *fwkerhalf2, T *fwkerhalf3) {
  int pivot1, pivot2, pivot3, w1, w2, w3, fwkerind1, fwkerind2, fwkerind3;
  int k1, k2, k3, inidx, outidx;
  T kervalue;

  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt * mu;
       i += blockDim.x * gridDim.x) {
    k1    = i % ms;
    k2    = (i / ms) % mt;
    k3    = (i / ms / mt);
    inidx = k1 + k2 * ms + k3 * ms * mt;

    if (modeord == 0) {
      pivot1    = k1 - ms / 2;
      pivot2    = k2 - mt / 2;
      pivot3    = k3 - mu / 2;
      w1        = (pivot1 >= 0) ? pivot1 : nf1 + pivot1;
      w2        = (pivot2 >= 0) ? pivot2 : nf2 + pivot2;
      w3        = (pivot3 >= 0) ? pivot3 : nf3 + pivot3;
      fwkerind1 = abs(pivot1);
      fwkerind2 = abs(pivot2);
      fwkerind3 = abs(pivot3);
    } else {
      pivot1    = k1 - ms + ms / 2;
      pivot2    = k2 - mt + mt / 2;
      pivot3    = k3 - mu + mu / 2;
      w1        = (pivot1 >= 0) ? nf1 + k1 - ms : k1;
      w2        = (pivot2 >= 0) ? nf2 + k2 - mt : k2;
      w3        = (pivot3 >= 0) ? nf3 + k3 - mu : k3;
      fwkerind1 = (pivot1 >= 0) ? ms - k1 : k1;
      fwkerind2 = (pivot2 >= 0) ? mt - k2 : k2;
      fwkerind3 = (pivot3 >= 0) ? mu - k3 : k3;
    }

    outidx       = w1 + w2 * nf1 + w3 * nf1 * nf2;
    kervalue     = fwkerhalf1[fwkerind1] * fwkerhalf2[fwkerind2] * fwkerhalf3[fwkerind3];
    fw[outidx].x = fk[inidx].x / kervalue;
    fw[outidx].y = fk[inidx].y / kervalue;
  }
}

template<typename T, int modeord>
int cudeconvolve1d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 1D.

    Melody Shih 11/21/21
*/
{
  auto &stream = d_plan->stream;

  int ms           = d_plan->ms;
  int nf1          = d_plan->nf1;
  int nmodes       = ms;
  int maxbatchsize = d_plan->batchsize;

  if (d_plan->spopts.spread_direction == 1) {
    for (int t = 0; t < blksize; t++) {
      deconvolve_1d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, nf1, d_plan->fw + t * nf1, d_plan->fk + t * nmodes, d_plan->fwkerhalf1);
    }
  } else {
    checkCudaErrors(hipMemsetAsync(
        d_plan->fw, 0, maxbatchsize * nf1 * sizeof(cuda_complex<T>), stream));
    for (int t = 0; t < blksize; t++) {
      amplify_1d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, nf1, d_plan->fw + t * nf1, d_plan->fk + t * nmodes, d_plan->fwkerhalf1);
    }
  }
  return 0;
}

template<typename T, int modeord>
int cudeconvolve2d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 2D.

    Melody Shih 07/25/19
*/
{
  auto &stream = d_plan->stream;

  int ms           = d_plan->ms;
  int mt           = d_plan->mt;
  int nf1          = d_plan->nf1;
  int nf2          = d_plan->nf2;
  int nmodes       = ms * mt;
  int maxbatchsize = d_plan->batchsize;

  if (d_plan->spopts.spread_direction == 1) {
    for (int t = 0; t < blksize; t++) {
      deconvolve_2d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, mt, nf1, nf2, d_plan->fw + t * nf1 * nf2, d_plan->fk + t * nmodes,
          d_plan->fwkerhalf1, d_plan->fwkerhalf2);
    }
  } else {
    checkCudaErrors(hipMemsetAsync(
        d_plan->fw, 0, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>), stream));
    for (int t = 0; t < blksize; t++) {
      amplify_2d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, mt, nf1, nf2, d_plan->fw + t * nf1 * nf2, d_plan->fk + t * nmodes,
          d_plan->fwkerhalf1, d_plan->fwkerhalf2);
    }
  }
  return 0;
}

template<typename T, int modeord>
int cudeconvolve3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 3D.

    Melody Shih 07/25/19
*/
{
  auto &stream = d_plan->stream;

  int ms           = d_plan->ms;
  int mt           = d_plan->mt;
  int mu           = d_plan->mu;
  int nf1          = d_plan->nf1;
  int nf2          = d_plan->nf2;
  int nf3          = d_plan->nf3;
  int nmodes       = ms * mt * mu;
  int maxbatchsize = d_plan->batchsize;
  if (d_plan->spopts.spread_direction == 1) {
    for (int t = 0; t < blksize; t++) {
      deconvolve_3d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, mt, mu, nf1, nf2, nf3, d_plan->fw + t * nf1 * nf2 * nf3,
          d_plan->fk + t * nmodes, d_plan->fwkerhalf1, d_plan->fwkerhalf2,
          d_plan->fwkerhalf3);
    }
  } else {
    checkCudaErrors(hipMemsetAsync(
        d_plan->fw, 0, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>), stream));
    for (int t = 0; t < blksize; t++) {
      amplify_3d<T, modeord><<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
          ms, mt, mu, nf1, nf2, nf3, d_plan->fw + t * nf1 * nf2 * nf3,
          d_plan->fk + t * nmodes, d_plan->fwkerhalf1, d_plan->fwkerhalf2,
          d_plan->fwkerhalf3);
    }
  }
  return 0;
}

template int cudeconvolve1d<float, 0>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve1d<float, 1>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve1d<double, 0>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve1d<double, 1>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve2d<float, 0>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve2d<float, 1>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve2d<double, 0>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve2d<double, 1>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve3d<float, 0>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve3d<float, 1>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve3d<double, 0>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve3d<double, 1>(cufinufft_plan_t<double> *d_plan, int blksize);

} // namespace deconvolve
} // namespace cufinufft
