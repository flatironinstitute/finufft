#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cufinufft/contrib/hip/hip_runtime_api.h>

#include <cufinufft/cudeconvolve.h>

namespace cufinufft {
namespace deconvolve {
/* Kernel for copying fw to fk with amplication by prefac/ker */
// Note: assume modeord=0: CMCL-compatible mode ordering in fk (from -N/2 up
// to N/2-1)
template <typename T>
__global__ void deconvolve_1d(int ms, int nf1, cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms; i += blockDim.x * gridDim.x) {
        int w1 = i - ms / 2 >= 0 ? i - ms / 2 : nf1 + i - ms / 2;

        T kervalue = fwkerhalf1[abs(i - ms / 2)];
        fk[i].x = fw[w1].x / kervalue;
        fk[i].y = fw[w1].y / kervalue;
    }
}

template <typename T>
__global__ void deconvolve_2d(int ms, int mt, int nf1, int nf2, cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1,
                              T *fwkerhalf2) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt; i += blockDim.x * gridDim.x) {
        int k1 = i % ms;
        int k2 = i / ms;
        int outidx = k1 + k2 * ms;
        int w1 = k1 - ms / 2 >= 0 ? k1 - ms / 2 : nf1 + k1 - ms / 2;
        int w2 = k2 - mt / 2 >= 0 ? k2 - mt / 2 : nf2 + k2 - mt / 2;
        int inidx = w1 + w2 * nf1;

        T kervalue = fwkerhalf1[abs(k1 - ms / 2)] * fwkerhalf2[abs(k2 - mt / 2)];
        fk[outidx].x = fw[inidx].x / kervalue;
        fk[outidx].y = fw[inidx].y / kervalue;
    }
}

template <typename T>
__global__ void deconvolve_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, cuda_complex<T> *fw,
                              cuda_complex<T> *fk, T *fwkerhalf1, T *fwkerhalf2, T *fwkerhalf3) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt * mu; i += blockDim.x * gridDim.x) {
        int k1 = i % ms;
        int k2 = (i / ms) % mt;
        int k3 = (i / ms / mt);
        int outidx = k1 + k2 * ms + k3 * ms * mt;
        int w1 = k1 - ms / 2 >= 0 ? k1 - ms / 2 : nf1 + k1 - ms / 2;
        int w2 = k2 - mt / 2 >= 0 ? k2 - mt / 2 : nf2 + k2 - mt / 2;
        int w3 = k3 - mu / 2 >= 0 ? k3 - mu / 2 : nf3 + k3 - mu / 2;
        int inidx = w1 + w2 * nf1 + w3 * nf1 * nf2;

        T kervalue = fwkerhalf1[abs(k1 - ms / 2)] * fwkerhalf2[abs(k2 - mt / 2)] * fwkerhalf3[abs(k3 - mu / 2)];
        fk[outidx].x = fw[inidx].x / kervalue;
        fk[outidx].y = fw[inidx].y / kervalue;
    }
}

/* Kernel for copying fk to fw with same amplication */
template <typename T>
__global__ void amplify_1d(int ms, int nf1, cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms; i += blockDim.x * gridDim.x) {
        int w1 = i - ms / 2 >= 0 ? i - ms / 2 : nf1 + i - ms / 2;

        T kervalue = fwkerhalf1[abs(i - ms / 2)];
        fw[w1].x = fk[i].x / kervalue;
        fw[w1].y = fk[i].y / kervalue;
    }
}

template <typename T>
__global__ void amplify_2d(int ms, int mt, int nf1, int nf2, cuda_complex<T> *fw, cuda_complex<T> *fk, T *fwkerhalf1,
                           T *fwkerhalf2) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt; i += blockDim.x * gridDim.x) {
        int k1 = i % ms;
        int k2 = i / ms;
        int inidx = k1 + k2 * ms;
        int w1 = k1 - ms / 2 >= 0 ? k1 - ms / 2 : nf1 + k1 - ms / 2;
        int w2 = k2 - mt / 2 >= 0 ? k2 - mt / 2 : nf2 + k2 - mt / 2;
        int outidx = w1 + w2 * nf1;

        T kervalue = fwkerhalf1[abs(k1 - ms / 2)] * fwkerhalf2[abs(k2 - mt / 2)];
        fw[outidx].x = fk[inidx].x / kervalue;
        fw[outidx].y = fk[inidx].y / kervalue;
    }
}

template <typename T>
__global__ void amplify_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, cuda_complex<T> *fw, cuda_complex<T> *fk,
                           T *fwkerhalf1, T *fwkerhalf2, T *fwkerhalf3) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < ms * mt * mu; i += blockDim.x * gridDim.x) {
        int k1 = i % ms;
        int k2 = (i / ms) % mt;
        int k3 = (i / ms / mt);
        int inidx = k1 + k2 * ms + k3 * ms * mt;
        int w1 = k1 - ms / 2 >= 0 ? k1 - ms / 2 : nf1 + k1 - ms / 2;
        int w2 = k2 - mt / 2 >= 0 ? k2 - mt / 2 : nf2 + k2 - mt / 2;
        int w3 = k3 - mu / 2 >= 0 ? k3 - mu / 2 : nf3 + k3 - mu / 2;
        int outidx = w1 + w2 * nf1 + w3 * nf1 * nf2;

        T kervalue = fwkerhalf1[abs(k1 - ms / 2)] * fwkerhalf2[abs(k2 - mt / 2)] * fwkerhalf3[abs(k3 - mu / 2)];
        fw[outidx].x = fk[inidx].x / kervalue;
        fw[outidx].y = fk[inidx].y / kervalue;
    }
}

template <typename T>
int cudeconvolve1d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 1D.

    Melody Shih 11/21/21
*/
{
    auto &stream = d_plan->stream;

    int ms = d_plan->ms;
    int nf1 = d_plan->nf1;
    int nmodes = ms;
    int maxbatchsize = d_plan->maxbatchsize;

    if (d_plan->spopts.spread_direction == 1) {
        for (int t = 0; t < blksize; t++) {
            deconvolve_1d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(ms, nf1, d_plan->fw + t * nf1,
                                                                        d_plan->fk + t * nmodes, d_plan->fwkerhalf1);
        }
    } else {
        checkCudaErrors(hipMemsetAsync(d_plan->fw, 0, maxbatchsize * nf1 * sizeof(cuda_complex<T>), stream));
        for (int t = 0; t < blksize; t++) {
            amplify_1d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(ms, nf1, d_plan->fw + t * nf1,
                                                                     d_plan->fk + t * nmodes, d_plan->fwkerhalf1);
        }
    }
    return 0;
}

template <typename T>
int cudeconvolve2d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 2D.

    Melody Shih 07/25/19
*/
{
    auto &stream = d_plan->stream;

    int ms = d_plan->ms;
    int mt = d_plan->mt;
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nmodes = ms * mt;
    int maxbatchsize = d_plan->maxbatchsize;

    if (d_plan->spopts.spread_direction == 1) {
        for (int t = 0; t < blksize; t++) {
            deconvolve_2d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(ms, mt, nf1, nf2, d_plan->fw + t * nf1 * nf2,
                                                                        d_plan->fk + t * nmodes, d_plan->fwkerhalf1,
                                                                        d_plan->fwkerhalf2);
        }
    } else {
        checkCudaErrors(hipMemsetAsync(d_plan->fw, 0, maxbatchsize * nf1 * nf2 * sizeof(cuda_complex<T>), stream));
        for (int t = 0; t < blksize; t++) {
            amplify_2d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(ms, mt, nf1, nf2, d_plan->fw + t * nf1 * nf2,
                                                                     d_plan->fk + t * nmodes, d_plan->fwkerhalf1,
                                                                     d_plan->fwkerhalf2);
        }
    }
    return 0;
}

template <typename T>
int cudeconvolve3d(cufinufft_plan_t<T> *d_plan, int blksize)
/*
    wrapper for deconvolution & amplication in 3D.

    Melody Shih 07/25/19
*/
{
    auto &stream = d_plan->stream;

    int ms = d_plan->ms;
    int mt = d_plan->mt;
    int mu = d_plan->mu;
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int nf3 = d_plan->nf3;
    int nmodes = ms * mt * mu;
    int maxbatchsize = d_plan->maxbatchsize;
    if (d_plan->spopts.spread_direction == 1) {
        for (int t = 0; t < blksize; t++) {
            deconvolve_3d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
                ms, mt, mu, nf1, nf2, nf3, d_plan->fw + t * nf1 * nf2 * nf3, d_plan->fk + t * nmodes,
                d_plan->fwkerhalf1, d_plan->fwkerhalf2, d_plan->fwkerhalf3);
        }
    } else {
        checkCudaErrors(
            hipMemsetAsync(d_plan->fw, 0, maxbatchsize * nf1 * nf2 * nf3 * sizeof(cuda_complex<T>), stream));
        for (int t = 0; t < blksize; t++) {
            amplify_3d<<<(nmodes + 256 - 1) / 256, 256, 0, stream>>>(
                ms, mt, mu, nf1, nf2, nf3, d_plan->fw + t * nf1 * nf2 * nf3, d_plan->fk + t * nmodes,
                d_plan->fwkerhalf1, d_plan->fwkerhalf2, d_plan->fwkerhalf3);
        }
    }
    return 0;
}

template int cudeconvolve1d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve1d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve2d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve2d<double>(cufinufft_plan_t<double> *d_plan, int blksize);
template int cudeconvolve3d<float>(cufinufft_plan_t<float> *d_plan, int blksize);
template int cudeconvolve3d<double>(cufinufft_plan_t<double> *d_plan, int blksize);

} // namespace deconvolve
} // namespace cufinufft
