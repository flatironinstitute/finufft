#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <limits>
#include <random>

// Include the custom operators for hipComplex
#include <cufinufft/contrib/hip/hip_vector_types.h>
#include <cufinufft/types.h>

// Helper function to create hipComplex
template<typename T> cuda_complex<T> make_cuda_complex(T real, T imag) {
  return cuda_complex<T>{real, imag};
}

// Helper function to compare hipComplex with std::complex<T> using 1 - ratio as error
template<typename T>
bool compareComplex(const cuda_complex<T> a, const std::complex<T> b,
                    const std::string &operation,
                    T epsilon = std::numeric_limits<T>::epsilon()) {
  const auto std_a = std::complex<T>(a.x, a.y);
  const auto err   = std::abs(std_a - b) / std::abs(std_a);
  if (err > epsilon) {
    std::cout << "Comparison failed in operation: " << operation << "\n";
    std::cout << "hipComplex: (" << a.x << ", " << a.y << ")\n";
    std::cout << "std::complex: (" << b.real() << ", " << b.imag() << ")\n";
    std::cout << "Error: " << err << "\n";
  }
  return err <= epsilon;
}

template<typename T> int testRandomOperations() {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<T> dis(-100.0, 100.0);

  for (int i = 0; i < 1000; ++i) {
    T real1  = dis(gen);
    T imag1  = dis(gen);
    T real2  = dis(gen);
    T imag2  = dis(gen);
    T scalar = dis(gen);

    cuda_complex<T> a = make_cuda_complex(real1, imag1);
    cuda_complex<T> b = make_cuda_complex(real2, imag2);
    std::complex<T> std_a(real1, imag1);
    std::complex<T> std_b(real2, imag2);

    // Test addition
    cuda_complex<T> result_add   = a + b;
    std::complex<T> expected_add = std_a + std_b;
    if (!compareComplex(result_add, expected_add,
                        "add complex<" + std::string(typeid(T).name()) + "> complex<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    // Test subtraction
    cuda_complex<T> result_sub   = a - b;
    std::complex<T> expected_sub = std_a - std_b;
    if (!compareComplex(result_sub, expected_sub,
                        "sub complex<" + std::string(typeid(T).name()) + "> complex<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    // Test multiplication
    cuda_complex<T> result_mul   = a * b;
    std::complex<T> expected_mul = std_a * std_b;
    if (!compareComplex(result_mul, expected_mul,
                        "mul complex<" + std::string(typeid(T).name()) + "> complex<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    // Test division
    // Avoid division by small numbers as the implementation is slightly different
    // Maybe there is a better way to test it
    if (real2 < 1.0 || imag2 < 1.0) { // Avoid division by zero
      cuda_complex<T> result_div   = a / b;
      std::complex<T> expected_div = std_a / std_b;
      if (!compareComplex(result_div, expected_div,
                          "div complex<" + std::string(typeid(T).name()) + "> complex<" +
                              std::string(typeid(T).name()) + ">",
                          std::numeric_limits<T>::epsilon() * 1000))
        return 1;
    }

    // Test addition with scalar
    cuda_complex<T> result_add_scalar   = a + scalar;
    std::complex<T> expected_add_scalar = std_a + scalar;
    if (!compareComplex(result_add_scalar, expected_add_scalar,
                        "add complex<" + std::string(typeid(T).name()) + "> scalar<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    // Test subtraction with scalar
    cuda_complex<T> result_sub_scalar   = a - scalar;
    std::complex<T> expected_sub_scalar = std_a - scalar;
    if (!compareComplex(result_sub_scalar, expected_sub_scalar,
                        "sub complex<" + std::string(typeid(T).name()) + "> scalar<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    // Test multiplication with scalar
    cuda_complex<T> result_mul_scalar   = a * scalar;
    std::complex<T> expected_mul_scalar = std_a * scalar;
    if (!compareComplex(result_mul_scalar, expected_mul_scalar,
                        "mul complex<" + std::string(typeid(T).name()) + "> scalar<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;

    cuda_complex<T> result_div_scalar   = a / scalar;
    std::complex<T> expected_div_scalar = std_a / scalar;
    if (!compareComplex(result_div_scalar, expected_div_scalar,
                        "div complex<" + std::string(typeid(T).name()) + "> scalar<" +
                            std::string(typeid(T).name()) + ">"))
      return 1;
  }
  return 0;
}

int main() {
  if (testRandomOperations<float>()) return 1;
  if (testRandomOperations<double>()) return 1;

  std::cout << "All tests passed!" << std::endl;
  return 0;
}
