#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <cufinufft/contrib/hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <random>

#include <cufinufft.h>

#include <cufinufft/impl.h>
#include <cufinufft/utils.h>

#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using cufinufft::utils::infnorm;

template <typename T>
int run_test(int method, int type, int N1, int M, T tol, T checktol, int iflag) {
    std::cout << std::scientific << std::setprecision(3);
    int ier;

    thrust::host_vector<T> x(M);
    thrust::host_vector<thrust::complex<T>> c(M);
    thrust::host_vector<thrust::complex<T>> fk(N1);

    thrust::device_vector<T> d_x(M);
    thrust::device_vector<thrust::complex<T>> d_c(M);
    thrust::device_vector<thrust::complex<T>> d_fk(N1);

    std::default_random_engine eng(1);
    std::uniform_real_distribution<T> dist11(-1, 1);
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    // Making data
    for (int i = 0; i < M; i++) {
        x[i] = M_PI * randm11(); // x in [-pi,pi)
    }
    if (type == 1) {
        for (int i = 0; i < M; i++) {
            c[i].real(randm11());
            c[i].imag(randm11());
        }
    } else if (type == 2) {
        for (int i = 0; i < N1; i++) {
            fk[i].real(randm11());
            fk[i].imag(randm11());
        }
    } else {
        std::cerr << "Invalid type " << type << " supplied\n";
        return 1;
    }

    d_x = x;
    if (type == 1)
        d_c = c;
    else if (type == 2)
        d_fk = fk;

    hipEvent_t start, stop;
    float milliseconds = 0;
    float totaltime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warm up CUFFT (is slow, takes around 0.2 sec... )
    hipEventRecord(start);
    {
        int nf1 = 1;
        hipfftHandle fftplan;
        hipfftPlan1d(&fftplan, nf1, cufft_type<T>(), 1);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] dummy warmup call to CUFFT\t %.3g s\n", milliseconds / 1000);

    // now to the test...
    cufinufft_plan_t<T> *dplan;
    const int dim = 1;

    // Here we setup our own opts, for gpu_method.
    cufinufft_opts opts;
    cufinufft_default_opts(&opts);

    opts.gpu_method = method;
    opts.gpu_maxbatchsize = 1;

    int nmodes[3] = {N1, 1, 1};
    int ntransf = 1;
    hipEventRecord(start);

    ier = cufinufft_makeplan_impl<T>(type, dim, nmodes, iflag, ntransf, tol, &dplan, &opts);
    if (ier != 0) {
        printf("err: cufinufft1d_plan\n");
        return ier;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = cufinufft_setpts_impl<T>(M, d_x.data().get(), NULL, NULL, 0, NULL, NULL, NULL, dplan);

    if (ier != 0) {
        printf("err: cufinufft_setpts\n");
        return ier;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = cufinufft_execute_impl<T>((cuda_complex<T> *)d_c.data().get(), (cuda_complex<T> *)d_fk.data().get(), dplan);

    if (ier != 0) {
        printf("err: cufinufft1d_exec\n");
        return ier;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    float exec_ms = milliseconds;
    printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = cufinufft_destroy_impl<T>(dplan);
    if (ier != 0) {
        printf("err %d: cufinufft1d_destroy\n", ier);
        return ier;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds / 1000);

    printf("[Method %d] %d U pts to %d NU pts in %.3g s:      %.3g NU pts/s\n", opts.gpu_method, N1, M,
           totaltime / 1000, M / totaltime * 1000);
    printf("\t\t\t\t\t(exec-only thoughput: %.3g NU pts/s)\n", M / exec_ms * 1000);

    T rel_error = std::numeric_limits<T>::max();
    if (type == 1) {
        fk = d_fk;
        int nt1 = 0.37 * N1; // choose some mode index to check
        thrust::complex<T> Ft = thrust::complex<T>(0, 0), J = thrust::complex<T>(0.0, iflag);
        for (int j = 0; j < M; ++j)
            Ft += c[j] * exp(J * (nt1 * x[j])); // crude direct
        int it = N1 / 2 + nt1;                  // index in complex F as 1d array

        rel_error = abs(Ft - fk[it]) / infnorm(N1, (std::complex<T> *)fk.data());
        printf("[gpu   ] one mode: rel err in F[%d] is %.3g\n", nt1, rel_error);
    } else if (type == 2) {
        c = d_c;

        int jt = M / 2; // check arbitrary choice of one targ pt
        thrust::complex<T> J = thrust::complex<T>(0, iflag);
        thrust::complex<T> ct = thrust::complex<T>(0, 0);
        int m = 0;
        for (int m1 = -(N1 / 2); m1 <= (N1 - 1) / 2; ++m1)
            ct += fk[m++] * exp(J * (m1 * x[jt])); // crude direct
        rel_error = abs(c[jt] - ct) / infnorm(M, (std::complex<T> *)c.data());
        printf("[gpu   ] one targ: rel err in c[%d] is %.3g\n", jt, rel_error);
    }

    return std::isnan(rel_error) || rel_error > checktol;
}

int main(int argc, char *argv[]) {
    if (argc != 8) {
        fprintf(stderr, "Usage: cufinufft1d_test method type N1 M tol checktol prec\n"
                        "Arguments:\n"
                        "  method: One of\n"
                        "    1: nupts driven\n"
                        "  type: Type of transform (1, 2)\n"
                        "  N1: Number of fourier modes\n"
                        "  M: The number of non-uniform points\n"
                        "  tol: NUFFT tolerance\n"
                        "  checktol:  relative error to pass test\n"
                        "  precision: f or d\n");
        return 1;
    }
    const int method = atoi(argv[1]);
    const int type = atoi(argv[2]);
    const int N1 = atof(argv[3]);
    const int M = atof(argv[4]);
    const double tol = atof(argv[5]);
    const double checktol = atof(argv[6]);
    const int iflag = 1;
    const char prec = argv[7][0];
    if (prec == 'f')
        return run_test<float>(method, type, N1, M, tol, checktol, iflag);
    else if (prec == 'd')
        return run_test<double>(method, type, N1, M, tol, checktol, iflag);
    else
        return -1;
}
