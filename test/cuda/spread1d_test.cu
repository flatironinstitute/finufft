#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <complex>
#include <iomanip>
#include <iostream>
#include <random>

#include <hip/hip_runtime_api.h>

#include <cufinufft.h>
#include <cufinufft/common.h>
#include <cufinufft/spreadinterp.h>
#include <cufinufft/types.h>
#include <cufinufft/utils.h>

using namespace cufinufft::common;
using namespace cufinufft::spreadinterp;
using namespace cufinufft::utils;

template <typename T>
int run_test(int method, int nupts_distribute, int nf1, int maxsubprobsize, int M, T tol, int kerevalmeth) {
    using real_t = T;
    using complex_t = cuda_complex<T>;

    int ier;
    int dim = 1;

    cufinufft_plan_template<real_t> dplan;
    dplan = (cufinufft_plan_template<real_t>)malloc(sizeof(*dplan));
    // Zero out your struct, (sets all pointers to NULL, crucial)
    memset(dplan, 0, sizeof(*dplan));

    ier = cufinufft_default_opts(2, dim, &(dplan->opts));
    dplan->opts.gpu_method = method;
    dplan->opts.gpu_maxsubprobsize = maxsubprobsize;
    dplan->opts.gpu_kerevalmeth = kerevalmeth;
    dplan->opts.gpu_sort = 1; // ahb changed from 0
    dplan->opts.gpu_spreadinterponly = 1;
    dplan->opts.gpu_binsizex = 1024; // binsize needs to be set here, since
                                     // SETUP_BINSIZE() is not called in
                                     // spread, interp only wrappers.
    ier = setup_spreader_for_nufft(dplan->spopts, tol, dplan->opts);

    std::cout << std::scientific << std::setprecision(3);

    real_t *x;
    complex_t *c, *fw;
    hipHostMalloc(&x, M * sizeof(real_t));
    hipHostMalloc(&c, M * sizeof(complex_t));
    hipHostMalloc(&fw, nf1 * sizeof(complex_t));

    real_t *d_x;
    complex_t *d_c, *d_fw;
    checkCudaErrors(hipMalloc(&d_x, M * sizeof(real_t)));
    checkCudaErrors(hipMalloc(&d_c, M * sizeof(complex_t)));
    checkCudaErrors(hipMalloc(&d_fw, nf1 * sizeof(complex_t)));

    std::default_random_engine eng(1);
    std::uniform_real_distribution<double> dist01(0, 1);
    std::uniform_real_distribution<double> dist11(-1, 1);
    auto rand01 = [&eng, &dist01]() { return dist01(eng); };
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    switch (nupts_distribute) {
    // Making data
    case 0: // uniform
    {
        for (int i = 0; i < M; i++) {
            x[i] = M_PI * randm11(); // x in [-pi,pi)
            c[i].x = randm11();
            c[i].y = randm11();
        }
    } break;
    case 1: // concentrate on a small region
    {
        for (int i = 0; i < M; i++) {
            x[i] = M_PI * rand01() / (nf1 * 2 / 32);
            c[i].x = randm11();
            c[i].y = randm11();
        }
    } break;
    default:
        std::cerr << "not valid nupts distr" << std::endl;
    }

    checkCudaErrors(hipMemcpy(d_x, x, M * sizeof(real_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_c, c, M * sizeof(complex_t), hipMemcpyHostToDevice));

    CNTime timer;
    timer.restart();
    ier = cufinufft_spread1d<real_t>(nf1, d_fw, M, d_x, d_c, dplan);
    if (ier != 0) {
        std::cout << "error: cnufftspread2d" << std::endl;
        return 0;
    }
    real_t t = timer.elapsedsec();
    printf("[Method %d] %d NU pts to #%d U pts in %.3g s (%.3g NU pts/s)\n", dplan->opts.gpu_method, M, nf1, t, M / t);

    checkCudaErrors(hipMemcpy(fw, d_fw, nf1 * sizeof(complex_t), hipMemcpyDeviceToHost));
    std::cout << "[result-input]" << std::endl;

    for (int i = std::max(nf1 / 2 - 5, 0); i < std::min(nf1 / 2 + 5, nf1 - 1); i++) {
        if (i % dplan->opts.gpu_binsizex == 0 && i != 0)
            printf(" |");
        printf(" (%2.3g,%2.3g)", fw[i].x, fw[i].y);
    }
    printf("\n");

    hipHostFree(x);
    hipHostFree(c);
    hipHostFree(fw);
    hipFree(d_x);
    hipFree(d_c);
    hipFree(d_fw);

    return 0;
}

int main(int argc, char *argv[]) {
    int nf1, N1, M;
    double upsampfac = 2.0;
    if (argc < 4) {
        fprintf(stderr, "Usage: spread1d_test method nupts_distr nf1 [maxsubprobsize [M [tol [kerevalmeth]]]]\n"
                        "Arguments:\n"
                        "  method: One of\n"
                        "    1: nupts driven, or\n"
                        "    2: sub-problem\n"
                        "  nupts_distr: The distribution of the points; one of\n"
                        "    0: uniform, or\n"
                        "    1: concentrated in a small region.\n"
                        "  nf1: The size of the 1D array.\n"
                        "  maxsubprobsize: Maximum size of subproblems (default 65536).\n"
                        "  M: The number of non-uniform points (default nf1 / 2).\n"
                        "  tol: NUFFT tolerance (default 1e-6).\n"
                        "  kerevalmeth: Kernel evaluation method; one of\n"
                        "     0: Exponential of square root (default), or\n"
                        "     1: Horner evaluation.\n");
        return 1;
    }
    double w;
    int method;
    sscanf(argv[1], "%d", &method);

    int nupts_distribute;
    sscanf(argv[2], "%d", &nupts_distribute);
    sscanf(argv[3], "%lf", &w);
    nf1 = (int)w; // so can read 1e6 right!

    int maxsubprobsize = 65536;
    if (argc > 4) {
        sscanf(argv[4], "%d", &maxsubprobsize);
    }

    N1 = (int)nf1 / upsampfac;
    M = N1;
    if (argc > 5) {
        sscanf(argv[5], "%lf", &w);
        M = w; // so can read 1e6 right!
    }

    double tol = 1e-6;
    if (argc > 6) {
        sscanf(argv[6], "%lf", &w);
        tol = w; // so can read 1e6 right!
    }

    int kerevalmeth = 0;
    if (argc > 7) {
        sscanf(argv[7], "%d", &kerevalmeth);
    }

    printf("float test\n");
    run_test<float>(method, nupts_distribute, nf1, maxsubprobsize, M, tol, kerevalmeth);
    printf("double test\n");
    run_test<double>(method, nupts_distribute, nf1, maxsubprobsize, M, tol, kerevalmeth);

    return 0;
}
