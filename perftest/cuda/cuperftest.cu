#include <getopt.h>

#include <cstdlib>
#include <iostream>
#include <random>
#include <string>
#include <unordered_map>

#include <cufinufft.h>
#include <cufinufft/impl.h>

#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

struct timespec get_wtime() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts;
}

double get_wtime_diff(const struct timespec &ts, const struct timespec &tf) {
    return (tf.tv_sec - ts.tv_sec) + (tf.tv_nsec - ts.tv_nsec) * 1E-9;
}

std::string get_or(const std::unordered_map<std::string, std::string> &m, const std::string &key,
                   const std::string &default_value) {
    auto it = m.find(key);
    if (it == m.end()) {
        return default_value;
    }
    return it->second;
}

struct test_options_t {
    char prec;
    int type;
    int n_runs;
    int N[3];
    int M;
    int kerevalmethod;
    int method;
    int sort;
    double tol;

    test_options_t(int argc, char *argv[]) {
        std::unordered_map<std::string, std::string> options_map;

        while (true) {
            int option_index = 0;

            // clang-format off
            static struct option long_options[] {
                {"prec", required_argument, 0, 0},
                {"type", required_argument, 0, 0},
                {"n_runs", required_argument, 0, 0},
                {"N1", required_argument, 0, 0},
                {"N2", required_argument, 0, 0},
                {"N3", required_argument, 0, 0},
                {"M", required_argument, 0, 0},
                {"tol", required_argument, 0, 0},
                {"method", required_argument, 0, 0},
                {"kerevalmethod", required_argument, 0, 0},
                {"sort", required_argument, 0, 0},
                {0, 0, 0, 0},
            };
            // clang-format on

            int c = getopt_long(argc, argv, "", long_options, &option_index);
            if (c == -1)
                break;

            switch (c) {
            case 0:
                options_map[long_options[option_index].name] = optarg;
                break;

            default:
                break;
            }
        }

        prec = get_or(options_map, "prec", "f")[0];
        type = std::stoi(get_or(options_map, "type", "1"));
        n_runs = std::stoi(get_or(options_map, "n_runs", "10"));
        N[0] = std::stof(get_or(options_map, "N1", "1E6"));
        N[1] = std::stof(get_or(options_map, "N2", "1"));
        N[2] = std::stof(get_or(options_map, "N3", "1"));
        M = std::stof(get_or(options_map, "M", "2E6"));
        method = std::stoi(get_or(options_map, "method", "1"));
        kerevalmethod = std::stoi(get_or(options_map, "kerevalmethod", "1"));
        sort = std::stoi(get_or(options_map, "sort", "1"));
        tol = std::stof(get_or(options_map, "tol", "1E-5"));
    }

    friend std::ostream &operator<<(std::ostream &outs, const test_options_t &opts) {
        return outs << "prec = " << opts.prec << "\n"
                    << "type = " << opts.type << "\n"
                    << "n_runs = " << opts.n_runs << "\n"
                    << "N1 = " << opts.N[0] << "\n"
                    << "N2 = " << opts.N[1] << "\n"
                    << "N3 = " << opts.N[2] << "\n"
                    << "M = " << opts.M << "\n"
                    << "method = " << opts.method << "\n"
                    << "kerevalmethod = " << opts.kerevalmethod << "\n"
                    << "sort = " << opts.sort << "\n"
                    << "tol = " << opts.tol << "\n";
    }
};

template <class F, class... Args>
inline double timeit(F f, Args... args) {
    auto st = get_wtime();
    f(args...);
    hipDeviceSynchronize();
    auto ft = get_wtime();
    return get_wtime_diff(st, ft);
}

void gpu_warmup() {
    int nf1 = 1;
    hipfftHandle fftplan;
    hipfftPlan1d(&fftplan, nf1, HIPFFT_Z2Z, 1);
}

template <typename T>
void run_test(test_options_t &test_opts) {
    std::cout << test_opts;
    const int M = test_opts.M;
    const int N = test_opts.N[0] * test_opts.N[1] * test_opts.N[2];
    const int type = test_opts.type;
    constexpr int iflag = 1;

    thrust::host_vector<T> x(M), y(M), z(M);
    thrust::host_vector<thrust::complex<T>> c(M), fk(N);

    thrust::device_vector<T> d_x(M), d_y(M), d_z(M);
    thrust::device_vector<thrust::complex<T>> d_c(M), d_fk(N);

    std::default_random_engine eng(1);
    std::uniform_real_distribution<T> dist11(-1, 1);
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    // Making data
    for (int i = 0; i < M; i++) {
        x[i] = M_PI * randm11(); // x in [-pi,pi)
        y[i] = M_PI * randm11();
        z[i] = M_PI * randm11();
    }
    d_x = x, d_y = y, d_z = z;

    if (type == 1) {
        for (int i = 0; i < M; i++) {
            c[i].real(randm11());
            c[i].imag(randm11());
        }
        d_c = c;
    } else if (type == 2) {
        for (int i = 0; i < N; i++) {
            fk[i].real(randm11());
            fk[i].imag(randm11());
        }
        d_fk = fk;
    } else {
        std::cerr << "Invalid type " << type << " supplied\n";
        return;
    }

    gpu_warmup();

    cufinufft_opts opts;
    int dim = 0;
    for (int i = 0; i < 3; ++i)
        dim = test_opts.N[i] > 1 ? i + 1 : dim;

    cufinufft_default_opts(&opts);
    opts.gpu_method = test_opts.method;
    opts.gpu_sort = test_opts.sort;
    opts.gpu_kerevalmeth = test_opts.kerevalmethod;

    cufinufft_plan_t<T> *dplan;
    double makeplan_time{0}, setpts_time{0}, execute_time{0};
    makeplan_time =
        timeit(cufinufft_makeplan_impl<T>, test_opts.type, dim, test_opts.N, iflag, 1, test_opts.tol, &dplan, &opts);

    T *d_x_p = dim >= 1 ? d_x.data().get() : nullptr;
    T *d_y_p = dim >= 2 ? d_y.data().get() : nullptr;
    T *d_z_p = dim == 3 ? d_z.data().get() : nullptr;
    cuda_complex<T> *d_c_p = (cuda_complex<T> *)d_c.data().get();
    cuda_complex<T> *d_fk_p = (cuda_complex<T> *)d_fk.data().get();
    for (int i = 0; i < test_opts.n_runs; ++i) {
        setpts_time += timeit(cufinufft_setpts_impl<T>, M, d_x_p, d_y_p, d_z_p, 0, nullptr, nullptr, nullptr, dplan);
        execute_time += timeit(cufinufft_execute_impl<T>, d_c_p, d_fk_p, dplan);
    }

    setpts_time /= test_opts.n_runs;
    execute_time /= test_opts.n_runs;

    std::cout << std::endl;
    std::cout << "makeplan: " << makeplan_time << std::endl;
    std::cout << "setpts  : " << setpts_time << std::endl;
    std::cout << "execute : " << execute_time << std::endl;
    std::cout << "total   : " << makeplan_time + setpts_time + execute_time << std::endl;
}

int main(int argc, char *argv[]) {
    if (argc == 2 && (std::string(argv[1]) == "--help" || std::string(argv[1]) == "-h")) {
        test_options_t default_opts(0, nullptr);
        // clang-format off
        std::cout << "Valid options:\n"
                     "    --prec <char>\n"
                     "           float or double precision. i.e. 'f' or 'd'\n"
                     "           default: " << default_opts.prec << "\n" << 
                     "    --type <int>\n"
                     "           type of transform. 1 or 2\n"
                     "           default: " << default_opts.type << "\n" << 
                     "    --n_runs <int>\n"
                     "           number of runs to average performance over\n"
                     "           default: " << default_opts.n_runs << "\n" <<
                     "    --N1 <int>\n"
                     "           number of modes in first dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[0] << "\n" <<
                     "    --N2 <int>\n"
                     "           number of modes in second dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[1] << "\n" <<
                     "    --N3 <int>\n"
                     "           number of modes in third dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[2] << "\n" <<
                     "    --M <int>\n"
                     "           number of non-uniform points. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.M << "\n" <<
                     "    --tol <float>\n"
                     "           NUFFT tolerance. Scientific notation accepted (i.e. 1.2E-7)\n"
                     "           default: " << default_opts.tol << "\n" <<
                     "    --method <int>\n"
                     "           NUFFT method\n"
                     "               1: nupts driven\n"
                     "               2: sub-problem\n"
                     "               4: block-gather\n"
                     "           Note that not all methods are compatible with all dim/type combinations\n"
                     "           default: " << default_opts.method << "\n" <<
                     "    --kerevalmeth <int>\n"
                     "           kernel evaluation method\n"
                     "               0: Exponential of square root\n"
                     "               1: Horner evaluation\n"
                     "           default: " << default_opts.kerevalmethod << "\n" <<
                     "    --sort: <int>\n"
                     "           sort strategy\n"
                     "               0: do not sort the points\n"
                     "               1: sort the points\n"
                     "           default: " << default_opts.sort << "\n";
        // clang-format on
        return 0;
    }
    test_options_t opts(argc, argv);

    if (opts.prec == 'f')
        run_test<float>(opts);
    else if (opts.prec == 'd')
        run_test<double>(opts);

    return 0;
}
