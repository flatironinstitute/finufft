#include "hip/hip_runtime.h"
#include <cstdint>
#include <getopt.h>

#include <cstdlib>
#include <iostream>
#include <random>
#include <string>
#include <unordered_map>

#include <cufinufft.h>
#include <cufinufft/impl.h>

#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

std::string get_or(const std::unordered_map<std::string, std::string> &m, const std::string &key,
                   const std::string &default_value) {
    auto it = m.find(key);
    if (it == m.end()) {
        return default_value;
    }
    return it->second;
}

struct test_options_t {
    char prec;
    int type;
    int n_runs;
    int N[3];
    int M;
    int ntransf;
    int kerevalmethod;
    int method;
    int sort;
    double tol;

    test_options_t(int argc, char *argv[]) {
        std::unordered_map<std::string, std::string> options_map;

        while (true) {
            int option_index = 0;

            // clang-format off
            static struct option long_options[] {
                {"prec", required_argument, 0, 0},
                {"type", required_argument, 0, 0},
                {"n_runs", required_argument, 0, 0},
                {"N1", required_argument, 0, 0},
                {"N2", required_argument, 0, 0},
                {"N3", required_argument, 0, 0},
                {"M", required_argument, 0, 0},
                {"ntransf", required_argument, 0, 0},
                {"tol", required_argument, 0, 0},
                {"method", required_argument, 0, 0},
                {"kerevalmethod", required_argument, 0, 0},
                {"sort", required_argument, 0, 0},
                {0, 0, 0, 0},
            };
            // clang-format on

            int c = getopt_long(argc, argv, "", long_options, &option_index);
            if (c == -1)
                break;

            switch (c) {
            case 0:
                options_map[long_options[option_index].name] = optarg;
                break;

            default:
                break;
            }
        }

        prec = get_or(options_map, "prec", "f")[0];
        type = std::stoi(get_or(options_map, "type", "1"));
        n_runs = std::stoi(get_or(options_map, "n_runs", "10"));
        N[0] = std::stof(get_or(options_map, "N1", "1E6"));
        N[1] = std::stof(get_or(options_map, "N2", "1"));
        N[2] = std::stof(get_or(options_map, "N3", "1"));
        M = std::stof(get_or(options_map, "M", "2E6"));
        ntransf = std::stoi(get_or(options_map, "ntransf", "1"));
        method = std::stoi(get_or(options_map, "method", "1"));
        kerevalmethod = std::stoi(get_or(options_map, "kerevalmethod", "1"));
        sort = std::stoi(get_or(options_map, "sort", "1"));
        tol = std::stof(get_or(options_map, "tol", "1E-5"));
    }

    friend std::ostream &operator<<(std::ostream &outs, const test_options_t &opts) {
        return outs << "# prec = " << opts.prec << "\n"
                    << "# type = " << opts.type << "\n"
                    << "# n_runs = " << opts.n_runs << "\n"
                    << "# N1 = " << opts.N[0] << "\n"
                    << "# N2 = " << opts.N[1] << "\n"
                    << "# N3 = " << opts.N[2] << "\n"
                    << "# M = " << opts.M << "\n"
                    << "# ntransf = " << opts.ntransf << "\n"
                    << "# method = " << opts.method << "\n"
                    << "# kerevalmethod = " << opts.kerevalmethod << "\n"
                    << "# sort = " << opts.sort << "\n"
                    << "# tol = " << opts.tol << "\n";
    }
};

struct CudaTimer {
    CudaTimer() {}

    ~CudaTimer() {
        for (auto &event : start_)
            hipEventDestroy(event);
        for (auto &event : stop_)
            hipEventDestroy(event);
    }

    void start() {
        start_.push_back(hipEvent_t{});
        stop_.push_back(hipEvent_t{});

        hipEventCreate(&start_.back());
        hipEventCreate(&stop_.back());

        hipEventRecord(start_.back());
    }

    void stop() { hipEventRecord(stop_.back()); }

    void sync() {
        for (auto &event : stop_)
            hipEventSynchronize(event);
    }

    float mean() { return this->tot() / start_.size(); }

    float std() {
        float avg = this->mean();

        double var = 0.0;
        for (int i = 0; i < start_.size(); ++i) {
            float dt;
            hipEventElapsedTime(&dt, start_[i], stop_[i]);
            var += (dt - avg) * (dt - avg);
        }
        var /= start_.size();

        return sqrt(var);
    }

    float tot() {
        float dt_tot = 0.;
        for (int i = 0; i < start_.size(); ++i) {
            float dt;
            hipEventElapsedTime(&dt, start_[i], stop_[i]);
            dt_tot += dt;
        }

        return dt_tot;
    }

    int count() { return start_.size(); }

    std::vector<hipEvent_t> start_;
    std::vector<hipEvent_t> stop_;
};

template <class F, class... Args>
inline void timeit(F f, CudaTimer &timer, Args... args) {
    timer.start();
    f(args...);
    timer.stop();
}

void gpu_warmup() {
    int nf1 = 100;
    hipfftHandle fftplan;
    hipfftPlan1d(&fftplan, nf1, HIPFFT_Z2Z, 1);
    thrust::device_vector<hipfftDoubleComplex> in(nf1), out(nf1);
    hipfftExecZ2Z(fftplan, in.data().get(), out.data().get(), 1);
    hipDeviceSynchronize();
}

template <typename T>
void run_test(test_options_t &test_opts) {
    std::cout << test_opts;
    const int ntransf = test_opts.ntransf;
    const int64_t M = test_opts.M;
    const int N = test_opts.N[0] * test_opts.N[1] * test_opts.N[2];
    const int type = test_opts.type;
    constexpr int iflag = 1;

    thrust::host_vector<T> x(M * ntransf), y(M * ntransf), z(M * ntransf);
    thrust::host_vector<thrust::complex<T>> c(M * ntransf), fk(N * ntransf);

    thrust::device_vector<T> d_x(M * ntransf), d_y(M * ntransf), d_z(M * ntransf);
    thrust::device_vector<thrust::complex<T>> d_c(M * ntransf), d_fk(N * ntransf);

    std::default_random_engine eng(1);
    std::uniform_real_distribution<T> dist11(-1, 1);
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    // Making data
    for (int64_t i = 0; i < M; i++) {
        x[i] = M_PI * randm11(); // x in [-pi,pi)
        y[i] = M_PI * randm11();
        z[i] = M_PI * randm11();
    }
    for (int64_t i = M; i < M * ntransf; ++i) {
        int64_t j = i % M;
        x[i] = x[j];
        y[i] = y[j];
        z[i] = z[j];
    }

    if (type == 1) {
        for (int i = 0; i < M * ntransf; i++) {
            c[i].real(randm11());
            c[i].imag(randm11());
        }

    } else if (type == 2) {
        for (int i = 0; i < N * ntransf; i++) {
            fk[i].real(randm11());
            fk[i].imag(randm11());
        }
    } else {
        std::cerr << "Invalid type " << type << " supplied\n";
        return;
    }

    gpu_warmup();

    cufinufft_opts opts;
    int dim = 0;
    for (int i = 0; i < 3; ++i)
        dim = test_opts.N[i] > 1 ? i + 1 : dim;

    cufinufft_default_opts(&opts);
    opts.gpu_method = test_opts.method;
    opts.gpu_sort = test_opts.sort;
    opts.gpu_kerevalmeth = test_opts.kerevalmethod;

    cufinufft_plan_t<T> *dplan;
    CudaTimer h2d_timer, makeplan_timer, setpts_timer, execute_timer, d2h_timer, amortized_timer;
    {
        amortized_timer.start();
        h2d_timer.start();
        d_x = x, d_y = y, d_z = z;
        if (type == 1)
            d_c = c;
        if (type == 2)
            d_fk = fk;
        h2d_timer.stop();

        T *d_x_p = dim >= 1 ? d_x.data().get() : nullptr;
        T *d_y_p = dim >= 2 ? d_y.data().get() : nullptr;
        T *d_z_p = dim == 3 ? d_z.data().get() : nullptr;
        cuda_complex<T> *d_c_p = (cuda_complex<T> *)d_c.data().get();
        cuda_complex<T> *d_fk_p = (cuda_complex<T> *)d_fk.data().get();

        timeit(cufinufft_makeplan_impl<T>, makeplan_timer, test_opts.type, dim, test_opts.N, iflag, ntransf,
               test_opts.tol, &dplan, &opts);
        for (int i = 0; i < test_opts.n_runs; ++i) {
            timeit(cufinufft_setpts_impl<T>, setpts_timer, M, d_x_p, d_y_p, d_z_p, 0, nullptr, nullptr, nullptr, dplan);
            timeit(cufinufft_execute_impl<T>, execute_timer, d_c_p, d_fk_p, dplan);
        }

        d2h_timer.start();
        if (type == 1)
            fk = d_fk;
        if (type == 2)
            c = d_c;
        d2h_timer.stop();
        
        amortized_timer.stop();

        h2d_timer.sync();
        makeplan_timer.sync();
        setpts_timer.sync();
        execute_timer.sync();
        d2h_timer.sync();
        amortized_timer.sync();
    }

    const int64_t nupts_tot = M * test_opts.n_runs * ntransf;

    printf("event,count,tot(ms),mean(ms),std(ms),nupts/s,ns/nupt\n");
    printf("host_to_device,%d,%f,%f,%f,0.0,0.0\n", h2d_timer.count(), h2d_timer.tot(),
           h2d_timer.mean(), h2d_timer.std());
    printf("makeplan,%d,%f,%f,%f,0.0,0.0\n", makeplan_timer.count(), makeplan_timer.tot(), makeplan_timer.mean(),
           makeplan_timer.std());
    printf("setpts,%d,%f,%f,%f,%g,%f\n", test_opts.n_runs, setpts_timer.tot(), setpts_timer.mean(), setpts_timer.std(),
           nupts_tot * 1000 / setpts_timer.tot(), setpts_timer.tot() * 1E6 / nupts_tot);
    printf("execute,%d,%f,%f,%f,%g,%f\n", test_opts.n_runs, execute_timer.tot(), execute_timer.mean(),
           execute_timer.std(), nupts_tot * 1000 / execute_timer.tot(), execute_timer.tot() * 1E6 / nupts_tot);
    printf("device_to_host,%d,%f,%f,%f,0.0,0.0\n", d2h_timer.count(), d2h_timer.tot(),
           d2h_timer.mean(), d2h_timer.std());
    printf("amortized,%d,%f,%f,%f,%g,%f\n", 1, amortized_timer.tot(), amortized_timer.mean(), amortized_timer.std(),
           nupts_tot * 1000 / amortized_timer.tot(), amortized_timer.tot() * 1E6 / nupts_tot);
}

int main(int argc, char *argv[]) {
    if (argc == 2 && (std::string(argv[1]) == "--help" || std::string(argv[1]) == "-h")) {
        test_options_t default_opts(0, nullptr);
        // clang-format off
        std::cout << "Valid options:\n"
                     "    --prec <char>\n"
                     "           float or double precision. i.e. 'f' or 'd'\n"
                     "           default: " << default_opts.prec << "\n" << 
                     "    --type <int>\n"
                     "           type of transform. 1 or 2\n"
                     "           default: " << default_opts.type << "\n" << 
                     "    --n_runs <int>\n"
                     "           number of runs to average performance over\n"
                     "           default: " << default_opts.n_runs << "\n" <<
                     "    --N1 <int>\n"
                     "           number of modes in first dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[0] << "\n" <<
                     "    --N2 <int>\n"
                     "           number of modes in second dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[1] << "\n" <<
                     "    --N3 <int>\n"
                     "           number of modes in third dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[2] << "\n" <<
                     "    --M <int>\n"
                     "           number of non-uniform points. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.M << "\n" <<
                     "    --ntransf <int>\n"
                     "           number of transforms to do simultaneously\n"
                     "           default: " << default_opts.ntransf << "\n" <<
                     "    --tol <float>\n"
                     "           NUFFT tolerance. Scientific notation accepted (i.e. 1.2E-7)\n"
                     "           default: " << default_opts.tol << "\n" <<
                     "    --method <int>\n"
                     "           NUFFT method\n"
                     "               1: nupts driven\n"
                     "               2: sub-problem\n"
                     "               4: block-gather\n"
                     "           Note that not all methods are compatible with all dim/type combinations\n"
                     "           default: " << default_opts.method << "\n" <<
                     "    --kerevalmeth <int>\n"
                     "           kernel evaluation method\n"
                     "               0: Exponential of square root\n"
                     "               1: Horner evaluation\n"
                     "           default: " << default_opts.kerevalmethod << "\n" <<
                     "    --sort: <int>\n"
                     "           sort strategy\n"
                     "               0: do not sort the points\n"
                     "               1: sort the points\n"
                     "           default: " << default_opts.sort << "\n";
        // clang-format on
        return 0;
    }
    test_options_t opts(argc, argv);

    if (opts.prec == 'f')
        run_test<float>(opts);
    else if (opts.prec == 'd')
        run_test<double>(opts);

    return 0;
}
